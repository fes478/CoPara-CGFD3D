#include "hip/hip_runtime.h"
#include "typedata.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<helper_functions.h>
#include<unistd.h>

using namespace std;
using namespace defstruct;
using namespace flatstruct;
using namespace constant;

const Real ChildProcs::RK4A[3] = {0.5, 0.5, 1.0};
const Real ChildProcs::RK4B[4] = {1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0};

#define errprt(...) com.errorprint(__FILE__, __LINE__, __VA_ARGS__)

#define CC(call) CheckCuda(__FILE__, __LINE__, call)
inline void CheckCuda(const char *file, const int line, hipError_t errS)
{//check at special cuda calls
	hipError_t err = hipGetLastError();
	if ( hipSuccess != errS )
	{
		char errstr[256];
		sprintf(errstr,"CUDA Error at %s line %d, error string is %s, error definition and numberis %s <%d> \n",
				file, line, errS, hipGetErrorString(errS), (int)err);
		hipDeviceReset();
		MpiErrorPrint(file,line,Fail2Cuda,errstr);
	}
}

#define Kcheck(call) KernelCheck(__FILE__,__LINE__,call)
inline void KernelCheck(const char *file, const int line, const char* call)
{//check at special position
	hipError_t err = hipGetLastError();
	if(hipSuccess!=err)
	{
		char errstr[256];
		sprintf(errstr,"%s behind line %d in file %s, error information is %s\n",call,line,file,hipGetErrorString(err));
		hipDeviceReset();
		MpiErrorPrint(file,line,Fail2Kernel,errstr);
	}
}

//---------------------------------------device kernel and function declaration---------------
hipError_t errmessage;
__constant__ int ipam[11];
__device__ void matinv(Real *A);//3*3 matrix invertion
__device__ void matmul(Real *A, Real *B, Real *C);//3*3 matrix mutiply

__global__ void perform();//display ipam
__global__ void generatewave(defstruct::wfield, int, int);//check wave and index

__global__ void WavefieldPick(defstruct::wfield, defstruct::wfield, flatstruct::PointIndexBufferF, int, int, int);
__global__ void SnapWavefieldPick(defstruct::wfield, defstruct::wfield, flatstruct::SnapIndexBufferF, int, int, int);//Abandoned

__global__ void VelPDcoeff(flatstruct::derivF, flatstruct::mdparF, defstruct::apara, Real*, Real*);// velocity partial derivative conversion coeffients

__global__ void CalDiff(int, int, int, int, Real, Real*, Real*, defstruct::wfield, flatstruct::PartialD);// space-domain stress and velocity partial derivative
__global__ void CalWave(int, flatstruct::derivF, flatstruct::mdparF, flatstruct::PartialD, defstruct::apara, Real*, Real*, defstruct::wfield,
			  defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield);
__global__ void CalDiffCL(int, int, int, int, Real, Real*, Real*, defstruct::wfield, flatstruct::PartialD);// space-domain stress and velocity partial derivative
__global__ void CalWaveCL(int, flatstruct::derivF, flatstruct::mdparF, flatstruct::PartialD, defstruct::apara, Real*, Real*, defstruct::wfield,
			  defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield);
__global__ void CalDiffSL(int, int, int, int, Real, Real*, Real*, defstruct::wfield, flatstruct::PartialD);// space-domain stress and velocity partial derivative
__global__ void CalWaveSL(int, flatstruct::derivF, flatstruct::mdparF, defstruct::apara, flatstruct::PartialD, defstruct::wfield,
			  defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield);
__global__ void CalTIMG(int, int, int, Real, Real*, flatstruct::derivF, defstruct::wfield, defstruct::wfield,
		        defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::apara);// Z-direction traction image free surface condition
__global__ void CalVUCD(int, int, int, Real, Real*, Real*, flatstruct::mdparF, flatstruct::derivF, defstruct::wfield, defstruct::wfield,
			defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::apara);// velocity unilater compact difference
__global__ void LoadForce(int, defstruct::cindx, Real, int, int, flatstruct::forceF, Real*, Real*, defstruct::wfield);//load force
__global__ void LoadMoment(int, defstruct::cindx, Real, int, int, flatstruct::momentF, Real*, defstruct::wfield);//load moment
__global__ void LoadRmom(defstruct::cindx, Real, int, flatstruct::RmomF, Real*, defstruct::wfield);//load focus
	
__global__ void IterationBegin(Real, Real, Real, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       int*, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield);//RK iteration begin
__global__ void IterationInner(Real, Real, Real, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       int*, defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield, defstruct::wfield);//RK iteration inner, excute twice
__global__ void IterationFinal(Real, Real, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       int*, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield);//RK iteration final
__global__ void IterationFinalPV(Real, Real, defstruct::PeakVel, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       int*, defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield,
			       defstruct::wfield, defstruct::wfield, defstruct::wfield);//RK iteration final
__global__ void ErrorSta(defstruct::wfield , int*);
__global__ void AbsExp(Real*, Real*, Real*, int*, defstruct::wfield);//expotional absorbing condition


//--------------------------------public--------------------------------------------
ChildProcs::ChildProcs(const char *filename, cindx i_cdx, Real i_steph, Real i_stept, 
			int i_nfrc, int i_nmnt, int i_nstf, int sepsize, int cxn, int cstart, 
			int i_ConIndex, int i_HyGrid, int *i_nabs,
			const int Cppn, const int i_nt, int *i_CSpn, int i_nsnap, const int Cfpn, const int Cfnt, Real Cfdt, int PVflag,
			const int restart, const int myid, const int cpn)
{
	int i;
	//pars init
	HostMpiRank = myid;
	cdx = i_cdx;
	steph = i_steph;
	stept = i_stept;
	nfrc = i_nfrc;
	nmnt = i_nmnt;
	nstf = i_nstf;
	Csize = sepsize;//valid size with 2 bounds
	Cxn = cxn;//valid size
	Cstart = cstart;//start index in absolute location
	ConIndex = i_ConIndex;//default at bottom
	HyGrid = i_HyGrid;
	ppn = Cppn;
	nt = i_nt;
	nsnap = i_nsnap;
	fpn = Cfpn;
	FNT = Cfnt;
	FDT = Cfdt;
	InterpTime = -1.0;//time to interp focus
	PVF = PVflag;
	
	CSpn = new int[nsnap]();
	for(i=0;i<nsnap;i++)
		CSpn[i] = i_CSpn[i];
	
	//host side boundary GS buffer, X-dir two bounds length
	fullsize = 2*LenFD*i_cdx.ny*i_cdx.nz;
	if(!myid) fullsize = (cpn+1)*fullsize;
	IraB.Txx = new Real [fullsize]();
	IraB.Tyy = new Real [fullsize]();
	IraB.Tzz = new Real [fullsize]();
	IraB.Txy = new Real [fullsize]();
	IraB.Txz = new Real [fullsize]();
	IraB.Tyz = new Real [fullsize]();
	IraB.Vx = new Real  [fullsize]();
	IraB.Vy = new Real  [fullsize]();
	IraB.Vz = new Real  [fullsize]();
	
	if(!myid)
	{
		Mflag = true;//master procs   //only malloc needed variables // 	W     for boundary
		printf("---accomplished GPU device boundary allocation work at Process[%d]\n",myid);
		return;
	}
	else
		Mflag = false;//child procs
	
	if(restart==1)
		Rwork = true;//restart work, reading the exists
	else
		Rwork = false;
	
	GpuAbility(filename);
	
	Cid.xdim = idxcom(Cxn, cdx.nj, Cid.DNum, Cid.ydim, Cid.xl, Cid.xr, Cid.yd, Cid.yu);
	
	printf("\n\n***Start to do GPU device initialization, parameter transfer, data array allocation and prepare\n");
	printf("On Rank %d node, full_X_size is %d, vaild_X_size = %d, start at %d, data index seires is :\n",myid,Csize,Cxn,Cstart);
	for(i=0;i<Cid.DNum;i++)
		printf("in device ID[%d]: xl=%d xr=%d yd=%d yu=%d\n",Cid.Rank[i],Cid.xl[i],Cid.xr[i],Cid.yd[i],Cid.yu[i]);
	printf("\n");

	//struct malloc
	int *ipamcache;
	ipamcache = new int[Cid.DNum*11];
	loadfixedarray(ipamcache, HostMpiRank, Cstart, Cxn, cdx.ni, cdx.nj, cdx.nk, Cid);
	for(i=0;i<Cid.DNum;i++)
	{
		printf("at PCS[%d], ipamcache[No.%d]=",HostMpiRank,i);
		for(int j=0;j<11;j++)
			printf("%d ",ipamcache[i*11+j]);
		printf("\n");
	}

	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		hipMemcpyToSymbol(HIP_SYMBOL(ipam),ipamcache+i*11,11*sizeof(int),0,hipMemcpyHostToDevice);
		//perform<<<1,1>>>();
	}
	delete [] ipamcache;

	//data struct allocation
	HSW = new wfield[nsnap]();	DSW = new wfield*[nsnap];
	HSpt = new SnapIndexBufferF[nsnap]();
	DSpt = new SnapIndexBufferF*[nsnap];	D_DSpt = new SnapIndexBufferF*[nsnap];
	for(i=0;i<nsnap;i++)
	{	
		DSpt[i] = new SnapIndexBufferF[Cid.DNum]();	D_DSpt[i] = new SnapIndexBufferF[Cid.DNum]();
		DSW[i] = new wfield[Cid.DNum]();
	}

	if(PVF) Dpv = new PeakVel[Cid.DNum]();
	if(fpn) DFpt = new FocalIndexBufferF[Cid.DNum]();//only valid under focal source
	Dpt = new PointIndexBufferF[Cid.DNum]();	D_Dpt = new PointIndexBufferF[Cid.DNum]();
	DPW = new wfield[Cid.DNum]();
	FW = new wfield[Cid.DNum]();	//h_FW = new wfield[Cid.DNum]();	
	W = new wfield[Cid.DNum]();	mW = new wfield[Cid.DNum]();	hW = new wfield[Cid.DNum]();	tW = new wfield[Cid.DNum]();
	pd = new PartialD[Cid.DNum]();

	drv = new derivF[Cid.DNum]();	mpa = new mdparF[Cid.DNum](); 
	matVx2Vz = new Real*[Cid.DNum];	matVy2Vz = new Real*[Cid.DNum];

	apr = new apara[Cid.DNum]();	frc = new forceF[Cid.DNum]();	mnt = new momentF[Cid.DNum]();	Rmnt = new RmomF[Cid.DNum]();	IM = new InterpMom[Cid.DNum]();

#ifdef CFSPML
	Ax = new wfield[Cid.DNum]();	mAx = new wfield[Cid.DNum]();	hAx = new wfield[Cid.DNum]();	tAx = new wfield[Cid.DNum]();	FAx = new wfield[Cid.DNum]();
	Ay = new wfield[Cid.DNum]();	mAy = new wfield[Cid.DNum]();	hAy = new wfield[Cid.DNum]();	tAy = new wfield[Cid.DNum]();	FAy = new wfield[Cid.DNum]();
	Az = new wfield[Cid.DNum]();	mAz = new wfield[Cid.DNum]();	hAz = new wfield[Cid.DNum]();	tAz = new wfield[Cid.DNum]();	FAz = new wfield[Cid.DNum]();
#endif

	fullsize = Csize*cdx.ny*cdx.nz;//seperate size, Csize = Cxn + 2*LenFD
	
	//host side PV allocation
	if(PVF)
	{
		Hpv.Vx = new Real[Csize*cdx.ny](); Hpv.Vy = new Real[Csize*cdx.ny](); Hpv.Vz = new Real[Csize*cdx.ny]();
	}

	//host side HSpt allocation
	for(i=0;i<this->nsnap;i++)
	{
		HSpt[i].Rsn = new int[CSpn[i]]();	HSpt[i].Gsn = new int[CSpn[i]]();
		HSpt[i].locx = new int[CSpn[i]]();	HSpt[i].locy = new int[CSpn[i]]();	HSpt[i].locz = new int[CSpn[i]]();
	}

	//host side Hpt and HPW allocation
	Hpt.Rsn = new int[ppn]();	Hpt.Gsn = new int[ppn]();
	Hpt.locx = new int[ppn]();	Hpt.locy = new int[ppn]();	Hpt.locz = new int[ppn]();

	HPW.Vx = new Real[nt*ppn]();	HPW.Vy = new Real[nt*ppn]();	HPW.Vz = new Real[nt*ppn]();
	HPW.Txx = new Real[nt*ppn]();	HPW.Tyy = new Real[nt*ppn]();	HPW.Tzz = new Real[nt*ppn]();
	HPW.Txy = new Real[nt*ppn]();	HPW.Txz = new Real[nt*ppn]();	HPW.Tyz = new Real[nt*ppn]();

	//host side, node-size gather buffer
	GD.Txx = new Real [fullsize](); GD.Tyy = new Real [fullsize](); GD.Tzz = new Real [fullsize]();
	GD.Txy = new Real [fullsize](); GD.Txz = new Real [fullsize](); GD.Tyz = new Real [fullsize]();
	GD.Vx = new Real  [fullsize](); GD.Vy = new Real  [fullsize](); GD.Vz = new Real  [fullsize]();
	
	//host side, node-size, par buffer, free after deliver
	H_drv.xix  = new Real [ fullsize ](); H_drv.xiy  = new Real [ fullsize ](); H_drv.xiz  = new Real [ fullsize ](); 
	H_drv.etax = new Real [ fullsize ](); H_drv.etay = new Real [ fullsize ](); H_drv.etaz = new Real [ fullsize ](); 
	H_drv.zetax= new Real [ fullsize ](); H_drv.zetay= new Real [ fullsize ](); H_drv.zetaz= new Real [ fullsize ](); 
	H_drv.jac  = new Real [ fullsize ](); 

	H_mpa.alpha = new Real [ fullsize ](); H_mpa.beta = new Real [ fullsize ](); H_mpa.rho = new Real [ fullsize ]();

	if(nfrc)
	{
		H_frc.locx = new int [ nfrc ]();	H_frc.locy = new int [ nfrc ]();	H_frc.locz = new int [ nfrc ]();
		H_frc.fx = new Real [ nfrc ]();		H_frc.fy = new Real [ nfrc ]();		H_frc.fz = new Real [ nfrc ]();
		H_frc.stf = new Real [ nfrc*nstf ]();
#ifdef SrcSmooth
		H_frc.dnorm = new Real [ nfrc*LenNorm*LenNorm*LenNorm ]();
#endif
	}

	if(nmnt)
	{
		H_mnt.locx = new int [ nmnt ](); H_mnt.locy = new int [ nmnt ](); H_mnt.locz = new int [ nmnt ]();
		H_mnt.mxx = new Real [ nmnt ](); H_mnt.myy = new Real [ nmnt ](); H_mnt.mzz = new Real [ nmnt ]();
		H_mnt.mxy = new Real [ nmnt ](); H_mnt.mxz = new Real [ nmnt ](); H_mnt.myz = new Real [ nmnt ]();
		H_mnt.stf = new Real [ nmnt*nstf ]();
#ifdef SrcSmooth
		H_mnt.dnorm = new Real [ nmnt*LenNorm*LenNorm*LenNorm ]();
#endif
	}
	
	if(fpn)
	{
		//host side HFpt allocation
		HFpt.Rsn = new int[fpn]();	HFpt.Gsn = new int[fpn]();
		HFpt.locx = new int[fpn]();	HFpt.locy = new int[fpn]();	HFpt.locz = new int[fpn]();
		
		//host side focal data allocation
		H_Rmnt.locx = new int [ fpn ](); H_Rmnt.locy = new int [ fpn ](); H_Rmnt.locz = new int [ fpn ]();
		H_Rmnt.mxx = new Real [ fpn*FNT ](); H_Rmnt.myy = new Real [ fpn*FNT ](); H_Rmnt.mzz = new Real [ fpn*FNT ]();
		H_Rmnt.mxy = new Real [ fpn*FNT ](); H_Rmnt.mxz = new Real [ fpn*FNT ](); H_Rmnt.myz = new Real [ fpn*FNT ]();
		// fpn->focus point,  FNT->time point
#ifdef SrcSmooth
		H_Rmnt.dnorm = new Real [ fpn*LenNorm*LenNorm*LenNorm ]();
#endif
	}
	
	H_apr.nabs = new int [ SeisGeo*2 ]();
#ifdef CFSPML	
	//CFS PML's par
	H_apr.APDx = new Real [ Csize ]();	H_apr.APDy = new Real [ cdx.ny ]();	H_apr.APDz = new Real [ cdx.nz ]();
	H_apr.Bx = new Real [ Csize ]();	H_apr.By = new Real [ cdx.ny ]();	H_apr.Bz = new Real [ cdx.nz ]();
	H_apr.DBx = new Real [ Csize ]();	H_apr.DBy = new Real [ cdx.ny ]();	H_apr.DBz = new Real [ cdx.nz ]();
	H_apr.CLoc = new int [ 26*6 ]();
#else	
	//Sponge Layer's par
	H_apr.Ex = new Real [ Csize ]();	H_apr.Ey = new Real [ cdx.ny ]();	H_apr.Ez = new Real [ cdx.nz ]();
	H_apr.ELoc = new int [ 6*6 ]();
#endif

	hipError_t err;

	//device side allocation
	for(i=0;i<Cid.DNum;i++)
	{
		err = hipSetDevice( Cid.Rank[i] );
		if(err != 0) printf("err = %d, errS=%s, error may occur at setdev\n",err, hipGetErrorString(err) );

		fullsize = (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz;//with boundary device-size
		hysize = (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*(cdx.nk2-ConIndex);
			//3<=ConIndex<=idz<cdx.nk2,should change array size I/O index
		axsize = (i_nabs[0]+i_nabs[1])*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz;
		aysize = (i_nabs[2]+i_nabs[3])*(Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*cdx.nz;
		azsize = (i_nabs[4]+i_nabs[5])*(Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD);

                //-------------------------------------lanuch pars ------------------------------------------------------
		(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD) <= BlockPerGrid.x ? BPG[i].x = Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD : BPG[i].x = BlockPerGrid.x;
		(Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD) <= BlockPerGrid.y ? BPG[i].y = Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD : BPG[i].y = BlockPerGrid.y;
		BPG[i].z = 1;

		//-------------------------------------wavefield variables-----------------------------------------------
		//h_FW[i].Txx = new Real [fullsize](); h_FW[i].Tyy = new Real [fullsize](); h_FW[i].Tzz = new Real [fullsize]();
		//h_FW[i].Txy = new Real [fullsize](); h_FW[i].Txz = new Real [fullsize](); h_FW[i].Tyz = new Real [fullsize]();
		//h_FW[i].Vx = new Real  [fullsize](); h_FW[i].Vy = new Real  [fullsize](); h_FW[i].Vz = new Real  [fullsize]();
		
		hipMalloc( (Real**)&FW[i].Txx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Tyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Tzz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Txy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Txz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Tyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Vx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&FW[i].Vy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&FW[i].Vz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc FW\n",err, hipGetErrorString(err) );

		hipMemset(FW[i].Txx, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Tyy, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Tzz, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Txy, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Txz, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Tyz, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Vx, 0, fullsize*sizeof(Real));
		hipMemset(FW[i].Vy, 0, fullsize*sizeof(Real));
		err = hipMemset(FW[i].Vz, 0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset FW\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&W[i].Txx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Tyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Tzz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Txy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Txz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Tyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Vx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&W[i].Vy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&W[i].Vz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc W\n",err, hipGetErrorString(err) );

		hipMemset(W[i].Txx, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Tyy, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Tzz, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Txy, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Txz, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Tyz, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Vx, 0, fullsize*sizeof(Real));
		hipMemset(W[i].Vy, 0, fullsize*sizeof(Real));
		err = hipMemset(W[i].Vz, 0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset W\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&mW[i].Txx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Tyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Tzz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Txy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Txz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Tyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Vx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mW[i].Vy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&mW[i].Vz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc mW\n",err, hipGetErrorString(err) );

		hipMemset(mW[i].Txx, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Tyy, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Tzz, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Txy, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Txz, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Tyz, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Vx, 0, fullsize*sizeof(Real));
		hipMemset(mW[i].Vy, 0, fullsize*sizeof(Real));
		err = hipMemset(mW[i].Vz, 0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset mW\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&hW[i].Txx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Tyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Tzz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Txy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Txz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Tyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Vx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&hW[i].Vy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&hW[i].Vz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc hW\n",err, hipGetErrorString(err) );

		hipMemset(hW[i].Txx, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Tyy, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Tzz, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Txy, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Txz, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Tyz, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Vx, 0, fullsize*sizeof(Real));
		hipMemset(hW[i].Vy, 0, fullsize*sizeof(Real));
		err = hipMemset(hW[i].Vz, 0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset hW\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&tW[i].Txx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Tyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Tzz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Txy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Txz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Tyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Vx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&tW[i].Vy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&tW[i].Vz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc tW\n",err, hipGetErrorString(err) );

		hipMemset(tW[i].Txx, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Tyy, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Tzz, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Txy, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Txz, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Tyz, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Vx, 0, fullsize*sizeof(Real));
		hipMemset(tW[i].Vy, 0, fullsize*sizeof(Real));
		err = hipMemset(tW[i].Vz, 0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset tW\n",err, hipGetErrorString(err) );
		//-------------------------------------wavefield variables-----------------------------------------------
		
		//-------------------------------------wavefield partial derivative--------------------------------------
#ifdef HYindex		
		hipMalloc( (Real**)&pd[i].DxTyy, hysize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DxTzz, hysize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DxTyz, hysize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DyTxx, hysize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DyTzz, hysize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DyTxz, hysize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DzTxx, hysize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzTyy, hysize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzTxy, hysize*sizeof(Real) );
		hipMemset(pd[i].DxTyy, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DxTzz, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DxTyz, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DyTxx, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DyTzz, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DyTxz, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DzTxx, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DzTyy, 0, hysize*sizeof(Real));
		hipMemset(pd[i].DzTxy, 0, hysize*sizeof(Real));
#else		
		hipMalloc( (Real**)&pd[i].DxTyy, fullsize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DxTzz, fullsize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DxTyz, fullsize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DyTxx, fullsize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DyTzz, fullsize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DyTxz, fullsize*sizeof(Real) );//HG
		hipMalloc( (Real**)&pd[i].DzTxx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzTyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzTxy, fullsize*sizeof(Real) );
		hipMemset(pd[i].DxTyy, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DxTzz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DxTyz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyTxx, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyTzz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyTxz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzTxx, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzTyy, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzTxy, 0, fullsize*sizeof(Real));
#endif
		
		hipMalloc( (Real**)&pd[i].DxTxx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DxTxy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DxTxz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DxVx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DxVy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&pd[i].DxVz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc pd.dx\n",err, hipGetErrorString(err) );
		
		hipMemset(pd[i].DxTxx, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DxTxy, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DxTxz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DxVx,  0, fullsize*sizeof(Real));
		hipMemset(pd[i].DxVy,  0, fullsize*sizeof(Real));
		err = hipMemset(pd[i].DxVz,  0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset Pd.Dx\n",err, hipGetErrorString(err) );

		
		hipMalloc( (Real**)&pd[i].DyTyy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DyTxy, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DyTyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DyVx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DyVy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&pd[i].DyVz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc pd.dy\n",err, hipGetErrorString(err) );
		
		hipMemset(pd[i].DyTyy, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyTxy, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyTyz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyVx,  0, fullsize*sizeof(Real));
		hipMemset(pd[i].DyVy,  0, fullsize*sizeof(Real));
		err = hipMemset(pd[i].DyVz,  0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset Pd.Dy\n",err, hipGetErrorString(err) );

		
		hipMalloc( (Real**)&pd[i].DzTzz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzTxz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzTyz, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzVx, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&pd[i].DzVy, fullsize*sizeof(Real) );
		err = hipMalloc( (Real**)&pd[i].DzVz, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc pd.dz\n",err, hipGetErrorString(err) );
		
		hipMemset(pd[i].DzTzz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzTxz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzTyz, 0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzVx,  0, fullsize*sizeof(Real));
		hipMemset(pd[i].DzVy,  0, fullsize*sizeof(Real));
		err = hipMemset(pd[i].DzVz,  0, fullsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset Pd.Dz\n",err, hipGetErrorString(err) );
		//-------------------------------------wavefield partial derivative--------------------------------------

		//-------------------------------------preprocessing pars------------------------------------------------
		//coordinate derivative
		hipMalloc( (Real**)&drv[i].xix, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].xiy, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].xiz, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].etax, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].etay, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].etaz, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].zetax, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].zetay, fullsize*sizeof(Real) ); 
		hipMalloc( (Real**)&drv[i].zetaz, fullsize*sizeof(Real) ); 
		err = hipMalloc( (Real**)&drv[i].jac, fullsize*sizeof(Real) ); 
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc drv\n",err, hipGetErrorString(err) );

		hipMemset( drv[i].xix, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].xiy, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].xiz, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].etax, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].etay, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].etaz, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].zetax, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].zetay, 0, fullsize*sizeof(Real) );
		hipMemset( drv[i].zetaz, 0, fullsize*sizeof(Real) );
		err = hipMemset( drv[i].jac, 0, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset Drv\n",err, hipGetErrorString(err) );

		//media pars
		hipMalloc( (Real**)&mpa[i].alpha, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mpa[i].beta, fullsize*sizeof(Real) );
		hipMalloc( (Real**)&mpa[i].rho, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc mpa\n",err, hipGetErrorString(err) );

		hipMemset( mpa[i].alpha, 0, fullsize*sizeof(Real) );
		hipMemset( mpa[i].beta, 0, fullsize*sizeof(Real) );
		err = hipMemset( mpa[i].rho, 0, fullsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Memset mpa\n",err, hipGetErrorString(err) );
		
		//force
		if(nfrc)
		{
			hipMalloc( (int**)&frc[i].locx, nfrc*sizeof(int) );
			hipMalloc( (int**)&frc[i].locy, nfrc*sizeof(int) );
			hipMalloc( (int**)&frc[i].locz, nfrc*sizeof(int) );
			hipMalloc( (Real**)&frc[i].fx, nfrc*sizeof(Real) );
			hipMalloc( (Real**)&frc[i].fy, nfrc*sizeof(Real) );
			hipMalloc( (Real**)&frc[i].fz, nfrc*sizeof(Real) );
			err = hipMalloc( (Real**)&frc[i].stf, nfrc*nstf*sizeof(Real) );
			if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc frc\n",err, hipGetErrorString(err) );
#ifdef SrcSmooth	
			hipMalloc( (Real**)&frc[i].dnorm, nfrc*LenNorm*LenNorm*LenNorm*sizeof(Real) );
#endif
		}

		//moment
		if(nmnt)
		{
			hipMalloc( (int**)&mnt[i].locx, nmnt*sizeof(int) );
			hipMalloc( (int**)&mnt[i].locy, nmnt*sizeof(int) );
			hipMalloc( (int**)&mnt[i].locz, nmnt*sizeof(int) );
			hipMalloc( (Real**)&mnt[i].mxx, nmnt*sizeof(Real) );
			hipMalloc( (Real**)&mnt[i].myy, nmnt*sizeof(Real) );
			hipMalloc( (Real**)&mnt[i].mzz, nmnt*sizeof(Real) );
			hipMalloc( (Real**)&mnt[i].mxy, nmnt*sizeof(Real) );
			hipMalloc( (Real**)&mnt[i].mxz, nmnt*sizeof(Real) );
			hipMalloc( (Real**)&mnt[i].myz, nmnt*sizeof(Real) );
			err = hipMalloc( (Real**)&mnt[i].stf, nmnt*nstf*sizeof(Real) );
			if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc mnt\n",err, hipGetErrorString(err) );
#ifdef SrcSmooth	
			hipMalloc( (Real**)&mnt[i].dnorm, nmnt*LenNorm*LenNorm*LenNorm*sizeof(Real) );
#endif
		}

		//PeakVel 
		if(PVF)
		{
			hipMalloc( (Real**)&Dpv[i].Vx, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
			hipMalloc( (Real**)&Dpv[i].Vy, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
			err=hipMalloc( (Real**)&Dpv[i].Vz, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
			if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc Dpv.vz\n",err, hipGetErrorString(err) );
			
			hipMemset( Dpv[i].Vx, 0, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
			hipMemset( Dpv[i].Vy, 0, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
			err=hipMemset( Dpv[i].Vz, 0, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
			if(err != 0) printf("err = %d, errS=%s, error may occur at memset Dpv.vz\n",err, hipGetErrorString(err) );
		}

		
		//Velocity partial derivative conversion coefficient //should copperate with wave tensor accessing index.
		hipMalloc( (Real**)&matVx2Vz[i], (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*SeisGeo*SeisGeo*sizeof(Real) );
		err = hipMalloc( (Real**)&matVy2Vz[i], (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*SeisGeo*SeisGeo*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc Mat2Vy2Vz\n",err, hipGetErrorString(err) );

		hipMemset(matVx2Vz[i], 0, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*SeisGeo*SeisGeo*sizeof(Real));
		err = hipMemset(matVy2Vz[i], 0, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*SeisGeo*SeisGeo*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset Mat2Vy2Vz\n",err, hipGetErrorString(err) );
		//-------------------------------------preprocessing pars------------------------------------------------

		
		//------------------------------------absorb damping pars---------------------------------------------------
		err = hipMalloc( (int**)&apr[i].nabs, SeisGeo*2*sizeof(int) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc apr\n",err, hipGetErrorString(err) );
#ifdef CFSPML	
		//-----------------------------------------------------------
		//ADE wave field in X-dir
		hipMalloc( (Real**)&Ax[i].Txx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Tyy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Tzz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Txy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Txz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Tyz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Vx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&Ax[i].Vy, axsize*sizeof(Real) );
		err = hipMalloc( (Real**)&Ax[i].Vz, axsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc Ax\n",err, hipGetErrorString(err) );

		hipMemset(Ax[i].Txx, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Tyy, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Tzz, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Txy, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Txz, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Tyz, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Vx, 0, axsize*sizeof(Real));
		hipMemset(Ax[i].Vy, 0, axsize*sizeof(Real));
		err = hipMemset(Ax[i].Vz, 0, axsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset Ax\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&mAx[i].Txx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Tyy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Tzz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Txy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Txz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Tyz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Vx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&mAx[i].Vy, axsize*sizeof(Real) );
		err = hipMalloc( (Real**)&mAx[i].Vz, axsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc mAx\n",err, hipGetErrorString(err) );

		hipMemset(mAx[i].Txx, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Tyy, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Tzz, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Txy, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Txz, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Tyz, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Vx, 0, axsize*sizeof(Real));
		hipMemset(mAx[i].Vy, 0, axsize*sizeof(Real));
		err = hipMemset(mAx[i].Vz, 0, axsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset mAx\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&hAx[i].Txx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Tyy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Tzz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Txy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Txz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Tyz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Vx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&hAx[i].Vy, axsize*sizeof(Real) );
		err = hipMalloc( (Real**)&hAx[i].Vz, axsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc hAx\n",err, hipGetErrorString(err) );

		hipMemset(hAx[i].Txx, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Tyy, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Tzz, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Txy, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Txz, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Tyz, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Vx, 0, axsize*sizeof(Real));
		hipMemset(hAx[i].Vy, 0, axsize*sizeof(Real));
		err = hipMemset(hAx[i].Vz, 0, axsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset hAx\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&tAx[i].Txx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Tyy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Tzz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Txy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Txz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Tyz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Vx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&tAx[i].Vy, axsize*sizeof(Real) );
		err = hipMalloc( (Real**)&tAx[i].Vz, axsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc tAx\n",err, hipGetErrorString(err) );

		hipMemset(tAx[i].Txx, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Tyy, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Tzz, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Txy, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Txz, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Tyz, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Vx, 0, axsize*sizeof(Real));
		hipMemset(tAx[i].Vy, 0, axsize*sizeof(Real));
		err = hipMemset(tAx[i].Vz, 0, axsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset tAx\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&FAx[i].Txx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Tyy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Tzz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Txy, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Txz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Tyz, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Vx, axsize*sizeof(Real) );
		hipMalloc( (Real**)&FAx[i].Vy, axsize*sizeof(Real) );
		err = hipMalloc( (Real**)&FAx[i].Vz, axsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc FAx\n",err, hipGetErrorString(err) );

		hipMemset(FAx[i].Txx, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Tyy, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Tzz, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Txy, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Txz, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Tyz, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Vx, 0, axsize*sizeof(Real));
		hipMemset(FAx[i].Vy, 0, axsize*sizeof(Real));
		err = hipMemset(FAx[i].Vz, 0, axsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset FAx\n",err, hipGetErrorString(err) );

		//-----------------------------------------------------------
		//ADE wave field in Y-dir
		hipMalloc( (Real**)&Ay[i].Txx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Tyy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Tzz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Txy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Txz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Tyz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Vx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&Ay[i].Vy, aysize*sizeof(Real) );
		err = hipMalloc( (Real**)&Ay[i].Vz, aysize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc Ay.Vz\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&mAy[i].Txx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Tyy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Tzz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Txy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Txz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Tyz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Vx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&mAy[i].Vy, aysize*sizeof(Real) );
		err = hipMalloc( (Real**)&mAy[i].Vz, aysize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc mAy\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&hAy[i].Txx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Tyy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Tzz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Txy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Txz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Tyz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Vx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&hAy[i].Vy, aysize*sizeof(Real) );
		err = hipMalloc( (Real**)&hAy[i].Vz, aysize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc hAy\n",err, hipGetErrorString(err) );
		
		hipMalloc( (Real**)&tAy[i].Txx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Tyy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Tzz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Txy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Txz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Tyz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Vx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&tAy[i].Vy, aysize*sizeof(Real) );
		err = hipMalloc( (Real**)&tAy[i].Vz, aysize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc tAy\n",err, hipGetErrorString(err) );
		
		hipMalloc( (Real**)&FAy[i].Txx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Tyy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Tzz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Txy, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Txz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Tyz, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Vx, aysize*sizeof(Real) );
		hipMalloc( (Real**)&FAy[i].Vy, aysize*sizeof(Real) );
		err = hipMalloc( (Real**)&FAy[i].Vz, aysize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc FAy\n",err, hipGetErrorString(err) );
		
		hipMemset(Ay[i].Txx, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Tyy, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Tzz, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Txy, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Txz, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Tyz, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Vx, 0, aysize*sizeof(Real));
		hipMemset(Ay[i].Vy, 0, aysize*sizeof(Real));
		err = hipMemset(Ay[i].Vz, 0, aysize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset Ay\n",err, hipGetErrorString(err) );

		hipMemset(mAy[i].Txx, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Tyy, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Tzz, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Txy, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Txz, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Tyz, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Vx, 0, aysize*sizeof(Real));
		hipMemset(mAy[i].Vy, 0, aysize*sizeof(Real));
		err = hipMemset(mAy[i].Vz, 0, aysize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset mAy\n",err, hipGetErrorString(err) );

		hipMemset(hAy[i].Txx, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Tyy, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Tzz, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Txy, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Txz, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Tyz, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Vx, 0, aysize*sizeof(Real));
		hipMemset(hAy[i].Vy, 0, aysize*sizeof(Real));
		err = hipMemset(hAy[i].Vz, 0, aysize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset mAy\n",err, hipGetErrorString(err) );

		hipMemset(tAy[i].Txx, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Tyy, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Tzz, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Txy, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Txz, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Tyz, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Vx, 0, aysize*sizeof(Real));
		hipMemset(tAy[i].Vy, 0, aysize*sizeof(Real));
		err = hipMemset(tAy[i].Vz, 0, aysize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset tAy\n",err, hipGetErrorString(err) );

		hipMemset(FAy[i].Txx, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Tyy, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Tzz, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Txy, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Txz, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Tyz, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Vx, 0, aysize*sizeof(Real));
		hipMemset(FAy[i].Vy, 0, aysize*sizeof(Real));
		err = hipMemset(FAy[i].Vz, 0, aysize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset FAy\n",err, hipGetErrorString(err) );

		//-----------------------------------------------------------
		//ADE wave field in Z-dir
		hipMalloc( (Real**)&Az[i].Txx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Tyy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Tzz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Txy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Txz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Tyz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Vx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&Az[i].Vy, azsize*sizeof(Real) );
		err = hipMalloc( (Real**)&Az[i].Vz, azsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc Az\n",err, hipGetErrorString(err) );
		
		hipMalloc( (Real**)&mAz[i].Txx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Tyy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Tzz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Txy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Txz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Tyz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Vx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&mAz[i].Vy, azsize*sizeof(Real) );
		err = hipMalloc( (Real**)&mAz[i].Vz, azsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc Az\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&hAz[i].Txx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Tyy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Tzz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Txy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Txz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Tyz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Vx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&hAz[i].Vy, azsize*sizeof(Real) );
		err = hipMalloc( (Real**)&hAz[i].Vz, azsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc hAz\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&tAz[i].Txx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Tyy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Tzz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Txy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Txz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Tyz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Vx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&tAz[i].Vy, azsize*sizeof(Real) );
		err = hipMalloc( (Real**)&tAz[i].Vz, azsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc tAz\n",err, hipGetErrorString(err) );

		hipMalloc( (Real**)&FAz[i].Txx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Tyy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Tzz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Txy, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Txz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Tyz, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Vx, azsize*sizeof(Real) );
		hipMalloc( (Real**)&FAz[i].Vy, azsize*sizeof(Real) );
		err = hipMalloc( (Real**)&FAz[i].Vz, azsize*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc FAz.Vz\n",err, hipGetErrorString(err) );

		hipMemset(Az[i].Txx, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Tyy, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Tzz, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Txy, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Txz, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Tyz, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Vx, 0, azsize*sizeof(Real));
		hipMemset(Az[i].Vy, 0, azsize*sizeof(Real));
		err = hipMemset(Az[i].Vz, 0, azsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset Az\n",err, hipGetErrorString(err) );

		hipMemset(mAz[i].Txx, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Tyy, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Tzz, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Txy, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Txz, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Tyz, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Vx, 0, azsize*sizeof(Real));
		hipMemset(mAz[i].Vy, 0, azsize*sizeof(Real));
		err = hipMemset(mAz[i].Vz, 0, azsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset mAz\n",err, hipGetErrorString(err) );

		hipMemset(hAz[i].Txx, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Tyy, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Tzz, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Txy, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Txz, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Tyz, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Vx, 0, azsize*sizeof(Real));
		hipMemset(hAz[i].Vy, 0, azsize*sizeof(Real));
		err = hipMemset(hAz[i].Vz, 0, azsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset hAz\n",err, hipGetErrorString(err) );

		hipMemset(tAz[i].Txx, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Tyy, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Tzz, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Txy, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Txz, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Tyz, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Vx, 0, azsize*sizeof(Real));
		hipMemset(tAz[i].Vy, 0, azsize*sizeof(Real));
		err = hipMemset(tAz[i].Vz, 0, azsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset tAz\n",err, hipGetErrorString(err) );

		hipMemset(FAz[i].Txx, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Tyy, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Tzz, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Txy, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Txz, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Tyz, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Vx, 0, azsize*sizeof(Real));
		hipMemset(FAz[i].Vy, 0, azsize*sizeof(Real));
		err = hipMemset(FAz[i].Vz, 0, azsize*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset FAz\n",err, hipGetErrorString(err) );

		//---------------------------------------------------------------
		//CFS PML's par
		hipMalloc( (Real**)&apr[i].APDx, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*sizeof(Real));
		hipMalloc( (Real**)&apr[i].APDy, (Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real));
		hipMalloc( (Real**)&apr[i].APDz, cdx.nz*sizeof(Real));
		hipMalloc( (Real**)&apr[i].Bx, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*sizeof(Real));
		hipMalloc( (Real**)&apr[i].By, (Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real));
		hipMalloc( (Real**)&apr[i].Bz, cdx.nz*sizeof(Real));
		hipMalloc( (Real**)&apr[i].DBx, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*sizeof(Real));
		hipMalloc( (Real**)&apr[i].DBy, (Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real));
		err = hipMalloc( (Real**)&apr[i].DBz, cdx.nz*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at malloc apr.DBz\n",err, hipGetErrorString(err) );
		hipMalloc( (int**)&apr[i].CLoc, 26*6*sizeof(int) );
#else	
		//Sponge Layer's par
		hipMalloc( (Real**)&apr[i].Ex, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*sizeof(Real) );
		hipMalloc( (Real**)&apr[i].Ey, (Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );
		hipMalloc( (Real**)&apr[i].Ez, cdx.nz*sizeof(Real) );
		hipMalloc( (int**)&apr[i].ELoc, 6*6*sizeof(int) );
#endif

		//------------------------------------absorb damping pars---------------------------------------------------

	}
	
	Kcheck( "hipMalloc check in CP initialization of calculate program!" );
	
#ifdef MPI_DEBUG
int gdb_break=1;
while(gdb_break){};
#endif
	
	printf("---accomplished GPU device preparation work at Process[%d]\n",myid);

}

ChildProcs::~ChildProcs()
{
	fprintf(stdout,"into data free at Procs[%d],in calculate.cu\n",HostMpiRank);
	if(!Mflag)
	{
		for(int i=0;i<Cid.DNum;i++)
		{
			hipSetDevice( Cid.Rank[i] );
			
#ifdef CFSPML		
			hipFree(apr[i].CLoc);
			hipFree(apr[i].DBz);	hipFree(apr[i].DBy);	hipFree(apr[i].DBx);
			hipFree(apr[i].Bz);	hipFree(apr[i].By);	hipFree(apr[i].Bx);
			hipFree(apr[i].APDz);	hipFree(apr[i].APDy);	hipFree(apr[i].APDx);

			//free ADE
			hipFree(FAz[i].Vz);	hipFree(FAz[i].Vy);	hipFree(FAz[i].Vx);
			hipFree(FAz[i].Tyz);	hipFree(FAz[i].Txz);	hipFree(FAz[i].Txy);
			hipFree(FAz[i].Tzz);	hipFree(FAz[i].Tyy);	hipFree(FAz[i].Txx);

			hipFree(tAz[i].Vz);	hipFree(tAz[i].Vy);	hipFree(tAz[i].Vx);
			hipFree(tAz[i].Tyz);	hipFree(tAz[i].Txz);	hipFree(tAz[i].Txy);
			hipFree(tAz[i].Tzz);	hipFree(tAz[i].Tyy);	hipFree(tAz[i].Txx);

			hipFree(hAz[i].Vz);	hipFree(hAz[i].Vy);	hipFree(hAz[i].Vx);
			hipFree(hAz[i].Tyz);	hipFree(hAz[i].Txz);	hipFree(hAz[i].Txy);
			hipFree(hAz[i].Tzz);	hipFree(hAz[i].Tyy);	hipFree(hAz[i].Txx);

			hipFree(mAz[i].Vz);	hipFree(mAz[i].Vy);	hipFree(mAz[i].Vx);
			hipFree(mAz[i].Tyz);	hipFree(mAz[i].Txz);	hipFree(mAz[i].Txy);
			hipFree(mAz[i].Tzz);	hipFree(mAz[i].Tyy);	hipFree(mAz[i].Txx);

			hipFree(Az[i].Vz);	hipFree(Az[i].Vy);	hipFree(Az[i].Vx);
			hipFree(Az[i].Tyz);	hipFree(Az[i].Txz);	hipFree(Az[i].Txy);
			hipFree(Az[i].Tzz);	hipFree(Az[i].Tyy);	hipFree(Az[i].Txx);

			hipFree(FAy[i].Vz);	hipFree(FAy[i].Vy);	hipFree(FAy[i].Vx);
			hipFree(FAy[i].Tyz);	hipFree(FAy[i].Txz);	hipFree(FAy[i].Txy);
			hipFree(FAy[i].Tzz);	hipFree(FAy[i].Tyy);	hipFree(FAy[i].Txx);

			hipFree(tAy[i].Vz);	hipFree(tAy[i].Vy);	hipFree(tAy[i].Vx);
			hipFree(tAy[i].Tyz);	hipFree(tAy[i].Txz);	hipFree(tAy[i].Txy);
			hipFree(tAy[i].Tzz);	hipFree(tAy[i].Tyy);	hipFree(tAy[i].Txx);

			hipFree(hAy[i].Vz);	hipFree(hAy[i].Vy);	hipFree(hAy[i].Vx);
			hipFree(hAy[i].Tyz);	hipFree(hAy[i].Txz);	hipFree(hAy[i].Txy);
			hipFree(hAy[i].Tzz);	hipFree(hAy[i].Tyy);	hipFree(hAy[i].Txx);

			hipFree(mAy[i].Vz);	hipFree(mAy[i].Vy);	hipFree(mAy[i].Vx);
			hipFree(mAy[i].Tyz);	hipFree(mAy[i].Txz);	hipFree(mAy[i].Txy);
			hipFree(mAy[i].Tzz);	hipFree(mAy[i].Tyy);	hipFree(mAy[i].Txx);

			hipFree(Ay[i].Vz);	hipFree(Ay[i].Vy);	hipFree(Ay[i].Vx);
			hipFree(Ay[i].Tyz);	hipFree(Ay[i].Txz);	hipFree(Ay[i].Txy);
			hipFree(Ay[i].Tzz);	hipFree(Ay[i].Tyy);	hipFree(Ay[i].Txx);

			hipFree(FAx[i].Vz);	hipFree(FAx[i].Vy);	hipFree(FAx[i].Vx);
			hipFree(FAx[i].Tyz);	hipFree(FAx[i].Txz);	hipFree(FAx[i].Txy);
			hipFree(FAx[i].Tzz);	hipFree(FAx[i].Tyy);	hipFree(FAx[i].Txx);

			hipFree(tAx[i].Vz);	hipFree(tAx[i].Vy);	hipFree(tAx[i].Vx);
			hipFree(tAx[i].Tyz);	hipFree(tAx[i].Txz);	hipFree(tAx[i].Txy);
			hipFree(tAx[i].Tzz);	hipFree(tAx[i].Tyy);	hipFree(tAx[i].Txx);

			hipFree(hAx[i].Vz);	hipFree(hAx[i].Vy);	hipFree(hAx[i].Vx);
			hipFree(hAx[i].Tyz);	hipFree(hAx[i].Txz);	hipFree(hAx[i].Txy);
			hipFree(hAx[i].Tzz);	hipFree(hAx[i].Tyy);	hipFree(hAx[i].Txx);

			hipFree(mAx[i].Vz);	hipFree(mAx[i].Vy);	hipFree(mAx[i].Vx);
			hipFree(mAx[i].Tyz);	hipFree(mAx[i].Txz);	hipFree(mAx[i].Txy);
			hipFree(mAx[i].Tzz);	hipFree(mAx[i].Tyy);	hipFree(mAx[i].Txx);

			hipFree(Ax[i].Vz);	hipFree(Ax[i].Vy);	hipFree(Ax[i].Vx);
			hipFree(Ax[i].Tyz);	hipFree(Ax[i].Txz);	hipFree(Ax[i].Txy);
			hipFree(Ax[i].Tzz);	hipFree(Ax[i].Tyy);	hipFree(Ax[i].Txx);

#else		
			hipFree(apr[i].ELoc);
			hipFree(apr[i].Ez);	hipFree(apr[i].Ey);	hipFree(apr[i].Ex);
#endif		
			hipFree(apr[i].nabs);

			hipFree(matVy2Vz[i]);	hipFree(matVx2Vz[i]);
			
			if(PVF)
			{
				hipFree(Dpv[i].Vx); hipFree(Dpv[i].Vy); hipFree(Dpv[i].Vz);
			}

			if(fpn)
			{

				delete [] IM[i].mxx; delete [] IM[i].myy; delete [] IM[i].mzz;
				delete [] IM[i].mxy; delete [] IM[i].mxz; delete [] IM[i].myz;
#ifdef SrcSmooth	
				hipFree(Rmnt[i].dnorm);
#endif
				hipFree(Rmnt[i].myz);	hipFree(Rmnt[i].mxz);	hipFree(Rmnt[i].mxy);
				hipFree(Rmnt[i].mzz);	hipFree(Rmnt[i].myy);	hipFree(Rmnt[i].mxx);
				hipFree(Rmnt[i].locz);	hipFree(Rmnt[i].locy);	hipFree(Rmnt[i].locx);
			}

			if(nmnt)
			{
#ifdef SrcSmooth	
				hipFree(mnt[i].dnorm);
#endif
				hipFree(mnt[i].stf);
				hipFree(mnt[i].myz);	hipFree(mnt[i].mxz);	hipFree(mnt[i].mxy);
				hipFree(mnt[i].mzz);	hipFree(mnt[i].myy);	hipFree(mnt[i].mxx);
				hipFree(mnt[i].locz);	hipFree(mnt[i].locy);	hipFree(mnt[i].locx);
			}

			if(nfrc)
			{
#ifdef SrcSmooth	
				hipFree(frc[i].dnorm);
#endif
				hipFree(frc[i].stf);
				hipFree(frc[i].fz);	hipFree(frc[i].fy);	hipFree(frc[i].fx);
				hipFree(frc[i].locz);	hipFree(frc[i].locy);	hipFree(frc[i].locx);
			}

			hipFree(mpa[i].rho);	hipFree(mpa[i].beta);	hipFree(mpa[i].alpha);

			hipFree(drv[i].jac);
			hipFree(drv[i].zetaz);	hipFree(drv[i].zetay);	hipFree(drv[i].zetax);
			hipFree(drv[i].etaz);	hipFree(drv[i].etay);	hipFree(drv[i].etax);
			hipFree(drv[i].xiz);	hipFree(drv[i].xiy);	hipFree(drv[i].xix);
			
			hipFree(pd[i].DzVz);	hipFree(pd[i].DzVy);	hipFree(pd[i].DzVx);
			hipFree(pd[i].DzTyz);	hipFree(pd[i].DzTxz);	hipFree(pd[i].DzTxy);
			hipFree(pd[i].DzTzz);	hipFree(pd[i].DzTyy);	hipFree(pd[i].DzTxx);

			hipFree(pd[i].DyVz);	hipFree(pd[i].DyVy);	hipFree(pd[i].DyVx);
			hipFree(pd[i].DyTyz);	hipFree(pd[i].DyTxz);	hipFree(pd[i].DyTxy);
			hipFree(pd[i].DyTzz);	hipFree(pd[i].DyTyy);	hipFree(pd[i].DyTxx);

			hipFree(pd[i].DxVz);	hipFree(pd[i].DxVy);	hipFree(pd[i].DxVx);
			hipFree(pd[i].DxTyz);	hipFree(pd[i].DxTxz);	hipFree(pd[i].DxTxy);
			hipFree(pd[i].DxTzz);	hipFree(pd[i].DxTyy);	hipFree(pd[i].DxTxx);

			hipFree(tW[i].Vz);	hipFree(tW[i].Vy);	hipFree(tW[i].Vx);
			hipFree(tW[i].Tyz);	hipFree(tW[i].Txz);	hipFree(tW[i].Txy);
			hipFree(tW[i].Tzz);	hipFree(tW[i].Tyy);	hipFree(tW[i].Txx);

			hipFree(hW[i].Vz);	hipFree(hW[i].Vy);	hipFree(hW[i].Vx);
			hipFree(hW[i].Tyz);	hipFree(hW[i].Txz);	hipFree(hW[i].Txy);
			hipFree(hW[i].Tzz);	hipFree(hW[i].Tyy);	hipFree(hW[i].Txx);

			hipFree(mW[i].Vz);	hipFree(mW[i].Vy);	hipFree(mW[i].Vx);
			hipFree(mW[i].Tyz);	hipFree(mW[i].Txz);	hipFree(mW[i].Txy);
			hipFree(mW[i].Tzz);	hipFree(mW[i].Tyy);	hipFree(mW[i].Txx);

			hipFree(W[i].Vz);	hipFree(W[i].Vy);	hipFree(W[i].Vx);
			hipFree(W[i].Tyz);	hipFree(W[i].Txz);	hipFree(W[i].Txy);
			hipFree(W[i].Tzz);	hipFree(W[i].Tyy);	hipFree(W[i].Txx);
			
			hipFree(FW[i].Vz);	hipFree(FW[i].Vy);	hipFree(FW[i].Vx);
			hipFree(FW[i].Tyz);	hipFree(FW[i].Txz);	hipFree(FW[i].Txy);
			hipFree(FW[i].Tzz);	hipFree(FW[i].Tyy);	hipFree(FW[i].Txx);

			//delete [] h_FW[i].Vz;	delete [] h_FW[i].Vy;	delete [] h_FW[i].Vx;
			//delete [] h_FW[i].Tyz;	delete [] h_FW[i].Txz;	delete [] h_FW[i].Txy;
			//delete [] h_FW[i].Tzz;	delete [] h_FW[i].Tyy;	delete [] h_FW[i].Txx;
			
			if(fpn)
			{
				delete [] DFpt[i].locz;	delete [] DFpt[i].locy;	delete [] DFpt[i].locx;
				delete [] DFpt[i].Gsn;	delete [] DFpt[i].Rsn;
			}

			delete [] Dpt[i].locz;	delete [] Dpt[i].locy;	delete [] Dpt[i].locx;
			delete [] Dpt[i].Gsn;	delete [] Dpt[i].Rsn;

			hipFree(D_Dpt[i].locz);	hipFree(D_Dpt[i].locy);	hipFree(D_Dpt[i].locx);
			hipFree(D_Dpt[i].Gsn);	hipFree(D_Dpt[i].Rsn);

#ifdef DevicePick			
			hipFree(DPW[i].Tyz);	hipFree(DPW[i].Txz);	hipFree(DPW[i].Txy);
			hipFree(DPW[i].Tzz);	hipFree(DPW[i].Tyy);	hipFree(DPW[i].Txx);
			hipFree(DPW[i].Vz);	hipFree(DPW[i].Vy);	hipFree(DPW[i].Vx);
#endif

		}

		//free peak vel
		if(PVF)
		{
			delete [] Hpv.Vx; delete [] Hpv.Vy; delete [] Hpv.Vz;
		}

#ifndef PointOnly
		for(int j=0;j<nsnap;j++)
		{
			for(int i=0;i<Cid.DNum;i++)
			{
#ifdef DevicePick				
				if(HSpt[j].cmp==2 || HSpt[j].cmp==3)
				{
					hipFree(DSW[j][i].Tyz);	hipFree(DSW[j][i].Txz);	hipFree(DSW[j][i].Txy);
					hipFree(DSW[j][i].Tzz);	hipFree(DSW[j][i].Tyy);	hipFree(DSW[j][i].Txx);
				}
				if(HSpt[j].cmp==1 || HSpt[j].cmp==3)
				{
					hipFree(DSW[j][i].Vz);	hipFree(DSW[j][i].Vy);	hipFree(DSW[j][i].Vx);
				}
#endif				

				hipFree(D_DSpt[j][i].Gsn);	hipFree(D_DSpt[j][i].Rsn);
				hipFree(D_DSpt[j][i].locz);	hipFree(D_DSpt[j][i].locy);	hipFree(D_DSpt[j][i].locx);

				delete [] DSpt[j][i].Gsn;	delete [] DSpt[j][i].Rsn;
				delete [] DSpt[j][i].locz;	delete [] DSpt[j][i].locy;	delete [] DSpt[j][i].locx;
			}
			
			if(HSpt[j].cmp==2 || HSpt[j].cmp==3)
			{
				delete [] HSW[j].Tyz;	delete [] HSW[j].Txz;	delete [] HSW[j].Txy;
				delete [] HSW[j].Tzz;	delete [] HSW[j].Tyy;	delete [] HSW[j].Txx;
			}
			if(HSpt[j].cmp==1 || HSpt[j].cmp==3)
			{
				delete [] HSW[j].Vz;	delete [] HSW[j].Vy;	delete [] HSW[j].Vx;
			}

		}
#endif

		for(int i=0;i<Cid.DNum;i++)
			hipDeviceReset();

		delete [] GD.Vz;	delete [] GD.Vy;	delete [] GD.Vx;
		delete [] GD.Tyz;	delete [] GD.Txz;	delete [] GD.Txy;	
		delete [] GD.Tzz;	delete [] GD.Tyy;	delete [] GD.Txx;

		delete [] HPW.Tyz;	delete [] HPW.Txz;	delete [] HPW.Txy;
		delete [] HPW.Tzz;	delete [] HPW.Tyy;	delete [] HPW.Txx;
		delete [] HPW.Vz;	delete [] HPW.Vy;	delete [] HPW.Vx;
		
		//host side focus buffer
		if(fpn)
		{
#ifdef SrcSmooth		
			delete [] H_Rmnt.dnorm;
#endif
			delete [] H_Rmnt.myz;	delete [] H_Rmnt.mxz;	delete [] H_Rmnt.mxy;
			delete [] H_Rmnt.mzz;	delete [] H_Rmnt.myy;	delete [] H_Rmnt.mxx;
			delete [] H_Rmnt.locz;	delete [] H_Rmnt.locy;	delete [] H_Rmnt.locx;
		}

		//point buffer
		delete [] Hpt.locz;	delete [] Hpt.locy;	delete [] Hpt.locx;
		delete [] Hpt.Gsn;	delete [] Hpt.Rsn;

		//snap buffer
		for(int i=0;i<nsnap;i++)
		{
			delete [] HSpt[i].Rsn;	delete [] HSpt[i].Gsn;
			delete [] HSpt[i].locx;	delete [] HSpt[i].locy;	delete [] HSpt[i].locz;
		}
		
		if(fpn)
		{
			//focus buffer
			delete [] HFpt.locz;	delete [] HFpt.locy;	delete [] HFpt.locx;
			delete [] HFpt.Gsn;	delete [] HFpt.Rsn;
		}

		//free host side struct array
#ifdef CFSPML
		delete [] FAz;	delete [] tAz;	delete [] hAz;	delete [] mAz;	delete [] Az;
		delete [] FAy;	delete [] tAy;	delete [] hAy;	delete [] mAy;	delete [] Ay;
		delete [] FAx;	delete [] tAx;	delete [] hAx;	delete [] mAx;	delete [] Ax;
#endif
		
		delete [] H_apr.nabs;

		delete [] IM;	delete [] Rmnt;	
		delete [] mnt;	delete [] frc;	delete [] apr;
		delete [] matVy2Vz;	delete [] matVx2Vz;
		delete [] mpa;	delete [] drv;
		delete [] pd;
		delete [] tW;	delete [] hW;	delete [] mW;	delete [] W;
		//delete [] h_FW;	
		delete [] FW;
		delete [] DPW;
		delete [] D_Dpt;	delete [] Dpt;
		if(fpn) delete [] DFpt;
		if(PVF) delete [] Dpv;

#ifndef PointOnly
		for(int i=0;i<nsnap;i++)
		{
			delete [] DSW[i];
			delete [] DSpt[i];	delete [] D_DSpt[i];
		}
		delete [] DSpt;	delete [] D_DSpt;
		delete [] DSW;	delete [] HSW;
#endif		
		delete [] HSpt;

		//free Cid pars
		delete [] BPG;
		delete [] Cid.yu;	delete [] Cid.yd;	delete [] Cid.xr;	delete [] Cid.xl;
		delete [] Cid.Size;	delete [] Cid.Rank;	delete [] Cid.np;	delete [] Cid.fp;
		for(int i=0;i<Cid.DNum;i++)
			delete [] Cid.Snp[i];
		delete [] Cid.Snp;
	
	}
	

	//free boundarybuffer
	delete [] IraB.Vz;	delete [] IraB.Vy;	delete [] IraB.Vx;
	delete [] IraB.Tyz;	delete [] IraB.Txz;	delete [] IraB.Txy;
	delete [] IraB.Tzz;	delete [] IraB.Tyy;	delete [] IraB.Txx;

	delete [] CSpn;

	fprintf(stdout,"data free at Procs[%d],in calculate.cu\n",HostMpiRank);
}

void ChildProcs::VelCoeff()
{
	//This subroutine used to compute trasfrom coeffcients
	int i;
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		VelPDcoeff<<<BPG[i].y,BPG[i].x>>>(drv[i], mpa[i], apr[i], matVx2Vz[i], matVy2Vz[i]);
		
		CC( hipDeviceSynchronize() );
	}
	fprintf(stdout,"accomplished VelPDcoeff computation at PCS[%d]\n",HostMpiRank);
}

void ChildProcs::RKite(int RKindex, int currT, int Xvec, int Yvec, int Zvec)
{
	//Note-1
	//forward and backward differential direction informations
	//the odd-order is opposite to even order for 4-Step Runge Kuuta 
	//(Thesis Equation 2.26 and Equation 2.37)
	// forward was represented by actual parameter value 1, 
	// and backward was represented by actual parameter value -1;
	
	//Note-2
	// in the RK begin part, use time increase 0
	// in the RK inner part, use time increase 1
	// in the RK final part, use time increase 2
	// time increase 0, 1 and 2 corresponding to the RK alpha time 0, 0.5, 0.5 and 1
	// at here it was used to extract force and moment source time function value

	//Note-3
	//before RK begin, in synchronization work, transfer FW to W,
	//at RK begin,
	//	first, calculate the space-domain partial derivative of W to get "pd"---P(T6V3)/P(xi,eta,zeta)
	//	then, mutiply the "pd" with covariant variables to get the time domain partial derivative hW ---P(T6V3)/Pt
	//	then, update tempral differential wave field---W by sum hW and FW
	//	and update final wave field---tW by sum hW and FW
	//at RK inner,
	//	this part execute two times
	//	first, differentiate W to get pd
	//	then, mutiply pd with covariant variables to get hW
	//	then, update W by sum hW and FW
	//	and update tW by sum tW and hW
	//at RK final,
	//	first, differentiate W to get pd
	//	then, mutiply pd with covariant variables to get hW
	//	then, update tW(will be represented by FW) by sum tW and hW
	//after RK final, transfer FW to outside and do storing works.
	//So,
	//	FW, keep unchanged, represent last big time step's wave field;
	//	W,  changes every iteration and use for next small step's differention work;
	//	hW, changed every iteration, represent this small step's wave filed;
	//	tW, changed every iteration, represent this big time step's final wave field;
	//	input FW, output FW


	int i;
	int tinc;
	int Tindex;
	Real time;
	Real alpha,beta;
	hipError_t err;
	
	if(RKindex == 0)
	{
		tinc = 0;
		alpha = RK4A[0];
		beta = RK4B[0];
	}
	else if( RKindex == 1 || RKindex == 2 )
	{
		tinc = 1;
		alpha = RK4A[RKindex];
		beta = RK4B[RKindex];
	}
	else
	{
		tinc = 2;//RKindex=3
		beta = RK4B[3];
	}
	Tindex = 2*currT + tinc;//use for source and moment
	time = (currT + tinc*0.5)*stept;//use for focus
	
	//calculate current focal data{ Cid.fp[i] length }
	if(fpn && time!=InterpTime)
	{
		InterpFocus(time);
		InterpTime = time;
	}
	
	//for(i=0;i<0;i++)
	for(i=0;i<Cid.DNum;i++)
	{
		
		err = hipSetDevice( Cid.Rank[i] );
		if(err!=0) printf("errS=%s, error may occur before RKite SetDev\n",hipGetErrorString(err) );
		
#ifdef DisBug
if(i==0 && HostMpiRank==1&& currT>=0)
{		
printf("\n at Procs[%d].Dev[%d]----->CurrtTime= %d RKindex=%d, tinc=%d, time=%f, Tindex= %d, RKite starts(%d,%d,%d),"
	"load mnt=%d frc=%d, focus=%d input flags=%d, %d, %d refer to ",
	HostMpiRank,Cid.Rank[i], currT, RKindex, tinc, time, Tindex,	
	zbx,zby,zbz,this->nmnt,this->nfrc, Cid.fp[i], Xvec,Yvec,Zvec);//AAA
if(Xvec==1) printf("F"); else printf("B");
if(Yvec==1) printf("F"); else printf("B");
if(Zvec==1) printf("F\n"); else printf("B\n");
}
#endif

		//-----------------------------------verification---------mpa/drv/wave field  transport---------------
		//generatewave<<<BPG[i],ThreadPerBlock>>>(FW[i], currT, Cstart);
		//generatewave<<<BlockPerGrid,ThreadPerBlock>>>(FW[i], currT, Cstart);
		//err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error at PCS [%d] Lanuch Perform/Generate\n",err, hipGetErrorString(err), HostMpiRank );
		
		if(this->HyGrid)
		{
		CalDiff<<<BPG[i],ThreadPerBlock>>>(Xvec, Yvec, Zvec, ConIndex, steph, matVx2Vz[i], matVy2Vz[i], mW[i], pd[i]);//mW used as W
		err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch HG-DiffCL\n",err, hipGetErrorString(err) );
		hipDeviceSynchronize();

		CalWave<<<BPG[i],ThreadPerBlock>>>(ConIndex, drv[i], mpa[i], pd[i], apr[i], matVx2Vz[i], matVy2Vz[i], hW[i],
							    mAx[i], hAx[i], mAy[i], hAy[i], mAz[i], hAz[i]);
		err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch HG-WaveCL\n",err, hipGetErrorString(err) );
		hipDeviceSynchronize();
		}
		else
		{
		CalDiffCL<<<BPG[i],ThreadPerBlock>>>(Xvec, Yvec, Zvec, ConIndex, steph, matVx2Vz[i], matVy2Vz[i], mW[i], pd[i]);//mW used as W
		err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch DiffCL\n",err, hipGetErrorString(err) );
		hipDeviceSynchronize();

		CalWaveCL<<<BPG[i],ThreadPerBlock>>>(ConIndex, drv[i], mpa[i], pd[i], apr[i], matVx2Vz[i], matVy2Vz[i], hW[i],
							    mAx[i], hAx[i], mAy[i], hAy[i], mAz[i], hAz[i]);
		err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch WaveCL\n",err, hipGetErrorString(err) );
		hipDeviceSynchronize();
		}
			
		
#ifdef CondFreeTIMG
		CalTIMG<<<BPG[i].y,BPG[i].x>>>(Xvec,Yvec,Zvec, steph, mpa[i].rho, drv[i], mW[i], hW[i], 
							   mAx[i], hAx[i], mAy[i], hAy[i], apr[i]);//mW used as W
		err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch TIMG\n",err, hipGetErrorString(err) );
		hipDeviceSynchronize();
#endif

#ifdef CondFreeVUCD
		//wrong
		CalVUCD<<<BPG[i].y,BPG[i].x>>>(Xvec,Yvec,Zvec, steph, matVx2Vz[i], matVy2Vz[i], mpa[i], drv[i], mW[i], hW[i],
							mAx[i], hAx[i], mAy[i], hAy[i], apr[i]);//mW used as W
		err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch VUCD\n",err, hipGetErrorString(err) );
		hipDeviceSynchronize();
#endif

		err = hipGetLastError(); 
		if(err!=0)printf("err=%d, errS=%s, error check after source at PCS[%d]Dev[%d](%d,%d,%d)\n",
				err, hipGetErrorString(err),HostMpiRank,Cid.Rank[i],this->nfrc,this->nmnt,Cid.fp[i]);
		
		if(this->nfrc)
		{
			LoadForce<<<BPG[i],ThreadPerBlock>>>(Tindex, cdx, steph, nfrc, nstf, frc[i], drv[i].jac, mpa[i].rho, hW[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch FORCE\n",err, hipGetErrorString(err) );
			hipDeviceSynchronize();
		}

		if(this->nmnt)
		{
			LoadMoment<<<BPG[i],ThreadPerBlock>>>(Tindex, cdx, steph, nmnt, nstf, mnt[i], drv[i].jac, hW[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch MOM\n",err, hipGetErrorString(err) );
			hipDeviceSynchronize();
		}
		
		if(Cid.fp[i])
		{
			LoadRmom<<<BPG[i],ThreadPerBlock>>>(cdx, steph, Cid.fp[i], Rmnt[i], drv[i].jac, hW[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch MOM\n",err, hipGetErrorString(err) );
			hipDeviceSynchronize();
		}

		err = hipGetLastError(); 
		if(err!=0)printf("err=%d, errS=%s, error check after source at PCS[%d]Dev[%d](%d,%d,%d)\n",
				err, hipGetErrorString(err),HostMpiRank,Cid.Rank[i],this->nfrc,this->nmnt,Cid.fp[i]);
		
		if(RKindex ==0)
		{
			IterationBegin<<<BPG[i],ThreadPerBlock>>>(stept, alpha, beta, FW[i], hW[i], tW[i], W[i],
				apr[i].nabs,FAx[i], hAx[i], tAx[i], Ax[i], 	FAy[i], hAy[i], tAy[i], Ay[i], 	FAz[i], hAz[i], tAz[i], Az[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch IteBegin\n",err, hipGetErrorString(err) );
		}
		else if(RKindex==1 || RKindex==2)
		{
			IterationInner<<<BPG[i],ThreadPerBlock>>>(stept, alpha, beta, FW[i], hW[i], tW[i], W[i],
				apr[i].nabs,FAx[i], hAx[i], tAx[i], Ax[i], 	FAy[i], hAy[i], tAy[i],Ay[i], 	FAz[i], hAz[i], tAz[i], Az[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch IteInn\n",err, hipGetErrorString(err) );
		}
		else
		{
			if(PVF)
			{//apply extract peak velocity from W
			IterationFinalPV<<<BPG[i],ThreadPerBlock>>>(stept, beta, Dpv[i], hW[i], tW[i], W[i],
				apr[i].nabs,hAx[i], tAx[i], Ax[i], 	hAy[i], tAy[i],Ay[i], 	hAz[i],tAz[i], Az[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch IteFin\n",err, hipGetErrorString(err) );
			}
			else
			{
			IterationFinal<<<BPG[i],ThreadPerBlock>>>(stept, beta, hW[i], tW[i], W[i],
				apr[i].nabs,hAx[i], tAx[i], Ax[i], 	hAy[i], tAy[i],Ay[i], 	hAz[i],tAz[i], Az[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after Lanuch IteFin\n",err, hipGetErrorString(err) );
			}
#ifdef withABS
#ifndef CFSPML			
			AbsExp<<<BPG[i],ThreadPerBlock>>>(apr[i].Ex, apr[i].Ey, apr[i].Ez, apr[i].nabs, W[i]);
			err = hipGetLastError(); if(err!=0)printf("err=%d, errS=%s, error may occur after AbsExp\n",err, hipGetErrorString(err) );
#endif		
#endif		
		}



		CC( hipDeviceSynchronize() );//OK no problem
			
	}
	
	//(*currT)++;//simulate one step forward

	

}

void ChildProcs::GatherData(wfield HOST, wfield *DEVICE, int kind)
{
	//from seperate device to full node
	//fullsize = Csize*cdx.ny*cdx.nz;//seperate node-size, Csize = Cxn + 2*LenFD
	//fullsize = (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*cdx.nz;//valid device-size with boundary

	//kind = 1, hipMemcpyHostToDevice;	H2D	to scatter computing parameters such as drv,mpa etc.
	//kind = 2, hipMemcpyDeviceToHost;	D2H	to gather wavefield.
	//kind = 3, hipMemcpyDeviceToDevice;	D2D	to be continue.

	int i;
	int idx,idy;
	int Rindex,Gindex;
	
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );

		for(idx=Cid.xl[i];idx<=Cid.xr[i];idx++)
			for(idy=Cid.yd[i];idy<=Cid.yu[i];idy++)
			{
				Gindex = idx*cdx.ny*cdx.nz + idy*cdx.nz;
				Rindex = (idx-Cid.xl[i]+LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*cdx.nz + (idy-Cid.yd[i]+LenFD)*cdx.nz;

				//hipMemcpy(GD.Vx+Gindex, FW[i].Vx+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
				
				if(kind==1)
				{
					hipMemcpy(DEVICE[i].Vx+Rindex, HOST.Vx+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Vy+Rindex, HOST.Vy+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Vz+Rindex, HOST.Vz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Txx+Rindex, HOST.Txx+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Tyy+Rindex, HOST.Tyy+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Tzz+Rindex, HOST.Tzz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Txy+Rindex, HOST.Txy+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Txz+Rindex, HOST.Txz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
					hipMemcpy(DEVICE[i].Tyz+Rindex, HOST.Tyz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
				}
				else if(kind==2)
				{
					hipMemcpy(HOST.Vx+Gindex, DEVICE[i].Vx+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Vy+Gindex, DEVICE[i].Vy+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Vz+Gindex, DEVICE[i].Vz+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Txx+Gindex, DEVICE[i].Txx+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Tyy+Gindex, DEVICE[i].Tyy+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Tzz+Gindex, DEVICE[i].Tzz+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Txy+Gindex, DEVICE[i].Txy+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Txz+Gindex, DEVICE[i].Txz+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
					hipMemcpy(HOST.Tyz+Gindex, DEVICE[i].Tyz+Rindex, cdx.nz*sizeof(Real), hipMemcpyDeviceToHost);
				}
				else
					fprintf(stdout,"WARNING: doing D2D node-size level Gather/Scatter works(kind=%d)\n",kind);

			}

	
	}
	//printf("Procs[%d], pass GatherData()\n",HostMpiRank);
}

void ChildProcs::SynTopo()
{
	//hipMalloc( (Real**)&matVx2Vz[i], (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*SeisGeo*SeisGeo*sizeof(Real) );
	//hipMallocManaged( (Real**)&matVx2Vz, Csize*cdx.ny*SeisGeo*SeisGeo*sizeof(Real) );//should copperate with wave tensor accessing index.

	int i,j,k;
	int idx,idy;
	int Rindex,Gindex;

	Real *MXZ,*MYZ;
	MXZ = new Real [Csize*cdx.ny*SeisGeo*SeisGeo]();
	MYZ = new Real [Csize*cdx.ny*SeisGeo*SeisGeo]();
	
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );

		printf("myid=%d,Cstart=%d, Csize=%d,cdx.ny=%d,range=(%d, %d, %d, %d)\n",HostMpiRank,Cstart,Csize,cdx.ny,Cid.xl[i],Cid.xr[i],Cid.yd[i],Cid.yu[i]);
		
		for(idx=Cid.xl[i];idx<=Cid.xr[i];idx++)
			for(idy=Cid.yd[i];idy<=Cid.yu[i];idy++)
			{
				Gindex = idx*cdx.ny*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;
				Rindex = (idx-Cid.xl[i]+LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*SeisGeo*SeisGeo + (idy-Cid.yd[i]+LenFD)*SeisGeo*SeisGeo;
				
				hipMemcpy(MXZ+Gindex, matVx2Vz[i]+Rindex, SeisGeo*SeisGeo*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy(MYZ+Gindex, matVy2Vz[i]+Rindex, SeisGeo*SeisGeo*sizeof(Real), hipMemcpyDeviceToHost);
			}
	}
	
	for(i=0;i<Csize;i++)
		for(j=0;j<cdx.ny;j++)
		{
			printf("mXz[%d][%d]=",i+Cstart,j);
			for(k=0;k<SeisGeo*SeisGeo;k++)
			{
				Gindex = i*cdx.ny*SeisGeo*SeisGeo+j*SeisGeo*SeisGeo+k;
				printf("%g\t",MXZ[Gindex]);
			}
			cout<<endl;
		}

	
	delete [] MYZ;
	delete [] MXZ;

}

void ChildProcs::SynPV()
{
	if(PVF==0)//does not apply peak vel extraction
		return;

	//Hpv.Vx = new Real[Csize*cdx.ny](); Hpv.Vy = new Real[Csize*cdx.ny](); Hpv.Vz = new Real[Csize*cdx.ny]();
	//hipMalloc( (Real**)&Dpv[i].Vx, (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*sizeof(Real) );

	int i;
	int idx;
	int Src,Dst,size;

	
	for(i=0;i<Cid.DNum;i++)
	{
		size = Cid.yu[i]-Cid.yd[i]+1;

		hipSetDevice( Cid.Rank[i] );

		//printf("myid=%d,Cstart=%d, Csize=%d,cdx.ny=%d,range=(%d, %d, %d, %d)\n",HostMpiRank,Cstart,Csize,cdx.ny,Cid.xl[i],Cid.xr[i],Cid.yd[i],Cid.yu[i]);
		
		for(idx=Cid.xl[i];idx<=Cid.xr[i];idx++)
		{
			Src = (idx-Cid.xl[i]+LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD) + LenFD;
			Dst = idx*cdx.ny + Cid.yd[i];
			//Dst = idx*cdx.ny + LenFD;

			hipMemcpy(Hpv.Vx+Dst, Dpv[i].Vx+Src, size*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(Hpv.Vy+Dst, Dpv[i].Vy+Src, size*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(Hpv.Vz+Dst, Dpv[i].Vz+Src, size*sizeof(Real), hipMemcpyDeviceToHost);
		}
		
	}
	
}

void ChildProcs::SynData()
{//only for display seperate device use
	//from seperate device to host, device size
	
	int i;
	int size;
	
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		
		size = (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*cdx.nz;//valid device-size with boundary
	
		hipMemcpy(h_FW[i].Vx, FW[i].Vx, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Vy, FW[i].Vy, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Vz, FW[i].Vz, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Txx, FW[i].Txx, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Tyy, FW[i].Tyy, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Tzz, FW[i].Tzz, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Txy, FW[i].Txy, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Txz, FW[i].Txz, size*sizeof(Real), hipMemcpyDeviceToHost);
		hipMemcpy(h_FW[i].Tyz, FW[i].Tyz, size*sizeof(Real), hipMemcpyDeviceToHost);
	
	}

	//printf("Procs[%d], pass SynData()\n",HostMpiRank);
}

void ChildProcs::ShareData()
{
	int i,j,k;
	int srcDev,dstDev;
	int srcLoc,dstLoc;
	int size;
	hipError_t err;
	
	//printf("totally device number is %d, Ycolumn=%d, Xcolumn=%d\n",Cid.DNum,Cid.ydim,Cid.xdim);
	//fullsize = (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz;//with boundary device-size
	//Processing in X-direction, delivering Y
	//forward-dir

	for(i=0;i<Cid.xdim-1;i++)
	{
		for(j=0;j<Cid.ydim;j++)
		{
			dstDev = i*Cid.ydim+j;
			srcDev = (i+1)*Cid.ydim+j;
			//printf("PCS[%d]set device[%d][%d](%d)<-----------get from device[%d][%d](%d):  size=%d, from %d, to %d"
			//	"\tinDST[%d]: xl=%d xr=%d yd=%d yu=%d"
			//	"\tinSRC[%d]: xl=%d xr=%d yd=%d yu=%d\n",
			//	HostMpiRank,i,j,dstDev,i+1,j,srcDev,
			//	Cid.yu[dstDev]-Cid.yd[dstDev]+1+2*LenFD, LenFD, LenFD + Cid.xr[dstDev]-Cid.xl[dstDev]+1,
			//	dstDev,Cid.xl[dstDev],Cid.xr[dstDev],Cid.yd[dstDev],Cid.yu[dstDev],
			//	srcDev,Cid.xl[srcDev],Cid.xr[srcDev],Cid.yd[srcDev],Cid.yu[srcDev]);
			
			size = (Cid.yu[dstDev]-Cid.yd[dstDev]+1+2*LenFD)*cdx.nz;//should be same in X dir, single silce of NY*NZ
			srcLoc = LenFD*size;//inner of left
			dstLoc = (LenFD + Cid.xr[dstDev]-Cid.xl[dstDev]+1)*size;//outter of right

			hipSetDevice( Cid.Rank[dstDev] );
			hipMemcpyPeer( W[dstDev].Vx+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vx+srcLoc,  Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Vy+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vy+srcLoc,  Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Vz+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vz+srcLoc,  Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Txx+dstLoc, Cid.Rank[dstDev], W[srcDev].Txx+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Tyy+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyy+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Tzz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tzz+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Txy+dstLoc, Cid.Rank[dstDev], W[srcDev].Txy+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Txz+dstLoc, Cid.Rank[dstDev], W[srcDev].Txz+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
		  err = hipMemcpyPeer( W[dstDev].Tyz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyz+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			if(err != 0) printf("err = %d, errS=%s, error may occur in all of X-dir forward delivering part\n",err, hipGetErrorString(err) );
			
			hipDeviceSynchronize();
			
		}
	}
	
	//backward-dir
	for(i=Cid.xdim-1;i>0;i--)
	{
		for(j=0;j<Cid.ydim;j++)
		{
			dstDev = i*Cid.ydim+j;
			srcDev = (i-1)*Cid.ydim+j;
			
			//printf("PCS[%d]set device[%d][%d](%d)<-----------get from device[%d][%d](%d):  size=%d, from %d, to %d"
			//	"\tinDST[%d]: xl=%d xr=%d yd=%d yu=%d"
			//	"\tinSRC[%d]: xl=%d xr=%d yd=%d yu=%d\n",
			//	HostMpiRank,i,j,dstDev,i-1,j,srcDev,
			//	Cid.yu[dstDev]-Cid.yd[dstDev]+1+2*LenFD, Cid.xr[dstDev]-Cid.xl[dstDev]+1, 0,
			//	dstDev,Cid.xl[dstDev],Cid.xr[dstDev],Cid.yd[dstDev],Cid.yu[dstDev],
			//	srcDev,Cid.xl[srcDev],Cid.xr[srcDev],Cid.yd[srcDev],Cid.yu[srcDev]);
			
			size = (Cid.yu[dstDev]-Cid.yd[dstDev]+1+2*LenFD)*cdx.nz;//should be same in X dir, single silce of NY*NZ
			srcLoc = (Cid.xr[srcDev]-Cid.xl[srcDev]+1)*size;//inner of right
			dstLoc = 0;//outter of left

			err = hipSetDevice( Cid.Rank[dstDev] );
			if(err != 0) printf("err = %d, errS=%s error may occur in all of X-dir backward DevSet\n",err, hipGetErrorString(err) );
			hipMemcpyPeer( W[dstDev].Vx+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vx+srcLoc,  Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Vy+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vy+srcLoc,  Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Vz+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vz+srcLoc,  Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Txx+dstLoc, Cid.Rank[dstDev], W[srcDev].Txx+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Tyy+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyy+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Tzz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tzz+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Txy+dstLoc, Cid.Rank[dstDev], W[srcDev].Txy+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			hipMemcpyPeer( W[dstDev].Txz+dstLoc, Cid.Rank[dstDev], W[srcDev].Txz+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
		  err = hipMemcpyPeer( W[dstDev].Tyz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyz+srcLoc, Cid.Rank[srcDev], LenFD*size*sizeof(Real));
			if(err != 0) printf("err = %d, errS=%s error may occur in all of X-dir backward delivering part\n",err, hipGetErrorString(err) );
			hipDeviceSynchronize();
			
		}
	}
	
	//Processing in Y-direction, delivering X
	//forward-dir
	for(i=0;i<Cid.xdim;i++)
	{
		for(j=0;j<Cid.ydim-1;j++)
		{
			dstDev = i*Cid.ydim+j;
			srcDev = i*Cid.ydim+j+1;
			
			//seperate delivering in X
			for(k=0;k<Cid.xr[dstDev]-Cid.xl[dstDev]+1+2*LenFD;k++)
			{
				size = LenFD*cdx.nz;//every X term have LenFDs Y terms.
				srcLoc = k*(Cid.yu[srcDev]-Cid.yd[srcDev]+1+2*LenFD)*cdx.nz + size;//inner of bottom
				dstLoc = k*(Cid.yu[dstDev]-Cid.yd[dstDev]+1+2*LenFD)*cdx.nz + (LenFD + Cid.yu[dstDev]-Cid.yd[dstDev]+1)*cdx.nz;//outter of top

				err = hipSetDevice( Cid.Rank[dstDev] );
				if(err != 0) printf("err = %d, errS=%s error may occur in all of Y-dir forward SetDev\n",err, hipGetErrorString(err) );
				hipMemcpyPeer( W[dstDev].Vx+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vx+srcLoc,  Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Vy+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vy+srcLoc,  Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Vz+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vz+srcLoc,  Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Txx+dstLoc, Cid.Rank[dstDev], W[srcDev].Txx+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Tyy+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyy+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Tzz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tzz+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Txy+dstLoc, Cid.Rank[dstDev], W[srcDev].Txy+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Txz+dstLoc, Cid.Rank[dstDev], W[srcDev].Txz+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
			  err = hipMemcpyPeer( W[dstDev].Tyz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyz+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				if(err != 0) printf("err = %d, errS=%s error may occur in all of Y-dir forward delivering part\n",err, hipGetErrorString(err) );
				
				hipDeviceSynchronize();

			}
		}
	}
	
	//backward-dir
	for(i=0;i<Cid.xdim;i++)
	{
		for(j=Cid.ydim-1;j>0;j--)
		{
			dstDev = i*Cid.ydim+j;
			srcDev = i*Cid.ydim+j-1;
			//printf("set device[%d][%d](%d)<-----------get from device[%d][%d](%d)\n",i,j,dstDev,i,j-1,srcDev);
			
			//seperate delivering in X
			for(k=0;k<Cid.xr[dstDev]-Cid.xl[dstDev]+1+2*LenFD;k++)
			{
				size = LenFD*cdx.nz;//every X term have LenFDs Y terms.
				srcLoc = k*(Cid.yu[srcDev]-Cid.yd[srcDev]+1+2*LenFD)*cdx.nz + (Cid.yu[srcDev]-Cid.yd[srcDev]+1)*cdx.nz;//inner of top
				dstLoc = k*(Cid.yu[dstDev]-Cid.yd[dstDev]+1+2*LenFD)*cdx.nz + 0;//outter of bottom

				err = hipSetDevice( Cid.Rank[dstDev] );
				if(err != 0) printf("err = %d, errS=%s error may occur in all of Y-dir backward SetDev\n",err, hipGetErrorString(err) );
				hipMemcpyPeer( W[dstDev].Vx+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vx+srcLoc,  Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Vy+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vy+srcLoc,  Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Vz+dstLoc,  Cid.Rank[dstDev], W[srcDev].Vz+srcLoc,  Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Txx+dstLoc, Cid.Rank[dstDev], W[srcDev].Txx+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Tyy+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyy+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Tzz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tzz+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Txy+dstLoc, Cid.Rank[dstDev], W[srcDev].Txy+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				hipMemcpyPeer( W[dstDev].Txz+dstLoc, Cid.Rank[dstDev], W[srcDev].Txz+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
			  err = hipMemcpyPeer( W[dstDev].Tyz+dstLoc, Cid.Rank[dstDev], W[srcDev].Tyz+srcLoc, Cid.Rank[srcDev], size*sizeof(Real));
				if(err != 0) printf("err = %d, errS=%s error may occur in all of Y-dir backward delivering part\n",err, hipGetErrorString(err) );
				
				hipDeviceSynchronize();

			}
		}
	}
	
	
}

void ChildProcs::IntraBoundGS(int GSflag)
{
	int i,j,k;
	int Dev;
	int size;
	int srcLoc,dstLoc;
	
	//before ShareData copy valid Y
	//after ShareData copy full Y, interweave
	
	if(GSflag)
	{
		//gather
		//from inner of Dev-left to IraB-left
		i = 0;
		for(j=0;j<Cid.ydim;j++)
		{
			Dev = i*Cid.ydim+j;
			hipSetDevice( Cid.Rank[Dev] );
			size = (Cid.yu[Dev]-Cid.yd[Dev]+1)*cdx.nz;
			for(k=0;k<LenFD;k++)
			{
				srcLoc = (LenFD+k)*(Cid.yu[Dev]-Cid.yd[Dev]+1+2*LenFD)*cdx.nz + LenFD*cdx.nz;
				dstLoc = k*cdx.ny*cdx.nz + Cid.yd[Dev]*cdx.nz;

				hipMemcpy( IraB.Vx+dstLoc,  W[Dev].Vx+srcLoc,  size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Vy+dstLoc,  W[Dev].Vy+srcLoc,  size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Vz+dstLoc,  W[Dev].Vz+srcLoc,  size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Txx+dstLoc, W[Dev].Txx+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Tyy+dstLoc, W[Dev].Tyy+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Tzz+dstLoc, W[Dev].Tzz+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Txy+dstLoc, W[Dev].Txy+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Txz+dstLoc, W[Dev].Txz+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Tyz+dstLoc, W[Dev].Tyz+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
			}
		}

		//from inner of Dev-right to IraB-right
		i = Cid.xdim-1;
		for(j=0;j<Cid.ydim;j++)
		{
			Dev = i*Cid.ydim+j;
			hipSetDevice( Cid.Rank[Dev] );
			size = (Cid.yu[Dev]-Cid.yd[Dev]+1)*cdx.nz;
			for(k=0;k<LenFD;k++)
			{
				srcLoc = (Cid.xr[Dev]-Cid.xl[Dev]+1+k)*(Cid.yu[Dev]-Cid.yd[Dev]+1+2*LenFD)*cdx.nz + LenFD*cdx.nz;
				dstLoc = (LenFD+k)*cdx.ny*cdx.nz + Cid.yd[Dev]*cdx.nz;

				hipMemcpy( IraB.Vx+dstLoc,  W[Dev].Vx+srcLoc,  size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Vy+dstLoc,  W[Dev].Vy+srcLoc,  size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Vz+dstLoc,  W[Dev].Vz+srcLoc,  size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Txx+dstLoc, W[Dev].Txx+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Tyy+dstLoc, W[Dev].Tyy+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Tzz+dstLoc, W[Dev].Tzz+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Txy+dstLoc, W[Dev].Txy+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Txz+dstLoc, W[Dev].Txz+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipMemcpy( IraB.Tyz+dstLoc, W[Dev].Tyz+srcLoc, size*sizeof(Real), hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
			}
		}
	}
	else
	{
		//scatter
		//from IraB-left to outter of Dev_left
		i = 0;
		for(j=0;j<Cid.ydim;j++)
		{
			Dev = i*Cid.ydim+j;
			hipSetDevice( Cid.Rank[Dev] );
			size = (Cid.yu[Dev]-Cid.yd[Dev]+1+2*LenFD)*cdx.nz;
			for(k=0;k<LenFD;k++)
			{
				srcLoc = k*cdx.ny*cdx.nz + (Cid.yd[Dev]-LenFD)*cdx.nz;
				dstLoc = k*(Cid.yu[Dev]-Cid.yd[Dev]+1+2*LenFD)*cdx.nz;

				hipMemcpy( W[Dev].Vx+dstLoc,  IraB.Vx+srcLoc,  size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Vy+dstLoc,  IraB.Vy+srcLoc,  size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Vz+dstLoc,  IraB.Vz+srcLoc,  size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Txx+dstLoc, IraB.Txx+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Tyy+dstLoc, IraB.Tyy+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Tzz+dstLoc, IraB.Tzz+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Txy+dstLoc, IraB.Txy+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Txz+dstLoc, IraB.Txz+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Tyz+dstLoc, IraB.Tyz+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipDeviceSynchronize();
			}
		}

		//from IraB-right to outter of Dev-right
		i = Cid.xdim-1;
		for(j=0;j<Cid.ydim;j++)
		{
			Dev = i*Cid.ydim+j;
			hipSetDevice( Cid.Rank[Dev] );
			size = (Cid.yu[Dev]-Cid.yd[Dev]+1+2*LenFD)*cdx.nz;
			for(k=0;k<LenFD;k++)
			{
				srcLoc = (LenFD+k)*cdx.ny*cdx.nz + (Cid.yd[Dev]-LenFD)*cdx.nz;
				dstLoc = (Cid.xr[Dev]-Cid.xl[Dev]+1+LenFD+k)*(Cid.yu[Dev]-Cid.yd[Dev]+1+2*LenFD)*cdx.nz;

				hipMemcpy( W[Dev].Vx+dstLoc,  IraB.Vx+srcLoc,  size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Vy+dstLoc,  IraB.Vy+srcLoc,  size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Vz+dstLoc,  IraB.Vz+srcLoc,  size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Txx+dstLoc, IraB.Txx+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Tyy+dstLoc, IraB.Tyy+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Tzz+dstLoc, IraB.Tzz+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Txy+dstLoc, IraB.Txy+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Txz+dstLoc, IraB.Txz+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipMemcpy( W[Dev].Tyz+dstLoc, IraB.Tyz+srcLoc, size*sizeof(Real), hipMemcpyHostToDevice);
				hipDeviceSynchronize();
			}
		}
	}

}

void ChildProcs::ParH2D()
{
	//from node to device, distribute parameters, free host buffer
	//full size drv/mpa and special size apr,frc,mnt

	//fullsize = Csize*cdx.ny*cdx.nz;//seperate node-size, Csize = Cxn + 2*LenFD
	//fullsize = (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*cdx.nz;//valid device-size with boundary

	//kind = 1, hipMemcpyHostToDevice;	H2D	to scatter computing parameters such as drv,mpa etc.
	//kind = 2, hipMemcpyDeviceToHost;	D2H	to gather wavefield.
	//kind = 3, hipMemcpyDeviceToDevice;	D2D	to be continue.
	
	hipError_t err;
	int i;
	int idx,idy;
	int Rindex,Gindex;

	fprintf(stdout,"Procs[%d] into parH2D\n",HostMpiRank);
	
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );

		for(idx=Cid.xl[i]-LenFD;idx<=Cid.xr[i]+LenFD;idx++)
			for(idy=Cid.yd[i]-LenFD;idy<=Cid.yu[i]+LenFD;idy++)
			{//copy full device size
				Gindex = idx*cdx.ny*cdx.nz + idy*cdx.nz;
				Rindex = (idx-Cid.xl[i]+LenFD)*(Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*cdx.nz + (idy-Cid.yd[i]+LenFD)*cdx.nz;

				//drv;
				err = hipMemcpy(drv[i].xix+Rindex,   H_drv.xix+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_xix\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].xiy+Rindex,   H_drv.xiy+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_xiy\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].xiz+Rindex,   H_drv.xiz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_xiz\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].etax+Rindex,  H_drv.etax+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_etax\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].etay+Rindex,  H_drv.etay+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_etay\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].etaz+Rindex,  H_drv.etaz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_etaz\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].zetax+Rindex, H_drv.zetax+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_zetax\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].zetay+Rindex, H_drv.zetay+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_zetay\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].zetaz+Rindex, H_drv.zetaz+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d mmecpy drv_zetaz\n",err, hipGetErrorString(err) );
				err = hipMemcpy(drv[i].jac+Rindex,   H_drv.jac+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy jac\n",err, hipGetErrorString(err) );

				//mpa
				err = hipMemcpy(mpa[i].alpha+Rindex, H_mpa.alpha+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy alpha\n",err, hipGetErrorString(err) );
				err = hipMemcpy(mpa[i].beta+Rindex,  H_mpa.beta+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy beta\n",err, hipGetErrorString(err) );
				err = hipMemcpy(mpa[i].rho+Rindex,   H_mpa.rho+Gindex, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
			if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy rho\n",err, hipGetErrorString(err) );

				hipDeviceSynchronize();
			}
		
		//frc
		if(nfrc)
		{
			hipMemcpy(frc[i].locx, H_frc.locx, nfrc*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(frc[i].locy, H_frc.locy, nfrc*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(frc[i].locz, H_frc.locz, nfrc*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(frc[i].fx,   H_frc.fx,   nfrc*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(frc[i].fy,   H_frc.fy,   nfrc*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(frc[i].fz,   H_frc.fz,   nfrc*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(frc[i].stf,  H_frc.stf,  nfrc*nstf*sizeof(Real), hipMemcpyHostToDevice);
#ifdef SrcSmooth
			hipMemcpy(frc[i].dnorm,  H_frc.dnorm,  nfrc*LenNorm*LenNorm*LenNorm*sizeof(Real), hipMemcpyHostToDevice);
#endif
			hipDeviceSynchronize();
		}

		//mnt
		if(nmnt)
		{
			hipMemcpy(mnt[i].locx, H_mnt.locx, nmnt*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].locy, H_mnt.locy, nmnt*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].locz, H_mnt.locz, nmnt*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].mxx,  H_mnt.mxx,  nmnt*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].myy,  H_mnt.myy,  nmnt*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].mzz,  H_mnt.mzz,  nmnt*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].mxy,  H_mnt.mxy,  nmnt*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].mxz,  H_mnt.mxz,  nmnt*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].myz,  H_mnt.myz,  nmnt*sizeof(Real), hipMemcpyHostToDevice);
			hipMemcpy(mnt[i].stf,  H_mnt.stf,  nmnt*nstf*sizeof(Real), hipMemcpyHostToDevice);
#ifdef SrcSmooth
			hipMemcpy(mnt[i].dnorm,  H_mnt.dnorm,  nmnt*LenNorm*LenNorm*LenNorm*sizeof(Real), hipMemcpyHostToDevice);
#endif

			hipDeviceSynchronize();
		}

		//apr
		err = hipMemcpy(apr[i].nabs, H_apr.nabs, SeisGeo*2*sizeof(int), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr nabs\n",err, hipGetErrorString(err) );
#ifdef CFSPML
		err = hipMemcpy(apr[i].APDx, H_apr.APDx+Cid.xl[i]-LenFD, (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl apdx par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].APDy, H_apr.APDy+Cid.yd[i]-LenFD, (Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl apdy par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].APDz, H_apr.APDz, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl apdz par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].Bx,   H_apr.Bx+Cid.xl[i]-LenFD, (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl bx par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].By,   H_apr.By+Cid.yd[i]-LenFD, (Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl by par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].Bz,   H_apr.Bz, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl Bz par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].DBx,  H_apr.DBx+Cid.xl[i]-LenFD, (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl dx par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].DBy,  H_apr.DBy+Cid.yd[i]-LenFD, (Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl dy par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].DBz,  H_apr.DBz, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl DB par\n",err, hipGetErrorString(err) );
		err = hipMemcpy(apr[i].CLoc, H_apr.CLoc, 26*6*sizeof(int), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at parh2d memcpy apr cfl cloc par\n",err, hipGetErrorString(err) );
#else
		hipMemcpy(apr[i].Ex,   H_apr.Ex+Cid.xl[i]-LenFD, (Cid.xr[i]-Cid.xl[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy(apr[i].Ey,   H_apr.Ey+Cid.yd[i]-LenFD, (Cid.yu[i]-Cid.yd[i]+1+2*LenFD)*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy(apr[i].Ez,   H_apr.Ez, cdx.nz*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy(apr[i].ELoc, H_apr.ELoc, 6*6*sizeof(int), hipMemcpyHostToDevice);
#endif
		hipDeviceSynchronize();
	
	}

	//free host par buffer
#ifdef CFSPML
	delete [] H_apr.CLoc;
	delete [] H_apr.DBz;	delete [] H_apr.DBy;	delete [] H_apr.DBx;
	delete [] H_apr.Bz;	delete [] H_apr.By;	delete [] H_apr.Bx;
	delete [] H_apr.APDz;	delete [] H_apr.APDy;	delete [] H_apr.APDx;
#else
	delete [] H_apr.ELoc;
	delete [] H_apr.Ez;	delete [] H_apr.Ey;	delete [] H_apr.Ex;
#endif
	//delete [] H_apr.nabs;
	
	//can not free focus here, becaues it will used in every step

	if(nmnt)
	{
#ifdef SrcSmooth		
		delete [] H_mnt.dnorm;
#endif
		delete [] H_mnt.stf;
		delete [] H_mnt.myz;	delete [] H_mnt.mxz;	delete [] H_mnt.mxy;
		delete [] H_mnt.mzz;	delete [] H_mnt.myy;	delete [] H_mnt.mxx;
		delete [] H_mnt.locz;	delete [] H_mnt.locy;	delete [] H_mnt.locx;
	}
	
	if(nfrc)
	{
#ifdef SrcSmooth		
		delete [] H_frc.dnorm;
#endif
		delete [] H_frc.stf;
		delete [] H_frc.fz;	delete [] H_frc.fy;	delete [] H_frc.fx;
		delete [] H_frc.locz;	delete [] H_frc.locy;	delete [] H_frc.locx;
	}

	delete [] H_mpa.rho;	delete [] H_mpa.beta;	delete [] H_mpa.alpha;

	delete [] H_drv.jac;
	delete [] H_drv.zetaz;	delete [] H_drv.zetay;	delete [] H_drv.zetax;
	delete [] H_drv.etaz;	delete [] H_drv.etay;	delete [] H_drv.etax;
	delete [] H_drv.xiz;	delete [] H_drv.xiy;	delete [] H_drv.xix;

	fprintf(stdout,"Procs[%d], pass ParH2D()\n",HostMpiRank);
}

void ChildProcs::C2DSnapPick()
{
#ifndef PointOnly
	
	int i,j,k;
	int numD;
	int nTime;
	hipError_t err;
/*	
	for(i=0;i<nsnap;i++)
		for(j=0;j<CSpn[i];j++)
			printf("inCal-Snapshot[%d],PCS[%d]->Rsn[%4d],Gsn[%4d]->(%3d,%3d,%3d),tinv=%d,cmp=%d\n",i+1,HostMpiRank,
				HSpt[i].Rsn[j],HSpt[i].Gsn[j],HSpt[i].locx[j],HSpt[i].locy[j],HSpt[i].locz[j],HSpt[i].tinv,HSpt[i].cmp);
*/	

	for(i=0;i<nsnap;i++)
	{
		for(k=0;k<Cid.DNum;k++)
		{
			numD=0;
			for(j=0;j<CSpn[i];j++)
			{
				if(HSpt[i].locx[j]>=Cid.xl[k]+Cstart && HSpt[i].locx[j]<=Cid.xr[k]+Cstart &&
				   HSpt[i].locy[j]>=Cid.yd[k] && HSpt[i].locy[j]<=Cid.yu[k])
				{
					numD++;
				}
			}
			Cid.Snp[k][i] = numD;
			//printf("inCal-Snap[%d],PCS[%d],Dev[%d],have point %d(CID->%d), in range(%d,%d)and(%d,%d)\n",
			//	i+1,HostMpiRank,k,numD,Cid.Snp[k][i],Cid.xl[k],Cid.xr[k],Cid.yd[k],Cid.yu[k]);
			
			DSpt[i][k].Rsn = new int[numD]();	DSpt[i][k].Gsn = new int[numD]();
			DSpt[i][k].locx = new int[numD]();	DSpt[i][k].locy = new int[numD]();	DSpt[i][k].locz = new int[numD]();

			err = hipSetDevice(Cid.Rank[k]);
			if(err != 0) printf("err = %d, errS=%s, error may occur at setdev\n",err, hipGetErrorString(err) );
			hipMalloc( (int**)&D_DSpt[i][k].Rsn, numD*sizeof(int) );
			hipMalloc( (int**)&D_DSpt[i][k].Gsn, numD*sizeof(int) );
			hipMalloc( (int**)&D_DSpt[i][k].locx, numD*sizeof(int) );
			hipMalloc( (int**)&D_DSpt[i][k].locy, numD*sizeof(int) );
			err = hipMalloc( (int**)&D_DSpt[i][k].locz, numD*sizeof(int) );
			if(err != 0) printf("err = %d, errS=%s, error may occur at malloc\n",err, hipGetErrorString(err) );
		}

	}

	for(i=0;i<nsnap;i++)
	{
		for(k=0;k<Cid.DNum;k++)
		{
			numD=0;
			for(j=0;j<CSpn[i];j++)
			{
				if(HSpt[i].locx[j]>=Cid.xl[k]+Cstart && HSpt[i].locx[j]<=Cid.xr[k]+Cstart &&
				   HSpt[i].locy[j]>=Cid.yd[k] && HSpt[i].locy[j]<=Cid.yu[k])
				{
					DSpt[i][k].Rsn[numD] = numD;
					DSpt[i][k].Gsn[numD] = HSpt[i].Rsn[j];
					DSpt[i][k].locx[numD] = HSpt[i].locx[j];
					DSpt[i][k].locy[numD] = HSpt[i].locy[j];
					DSpt[i][k].locz[numD] = HSpt[i].locz[j];
					numD++;
				}
			}
			DSpt[i][k].tinv = HSpt[i].tinv;
			DSpt[i][k].cmp = HSpt[i].cmp;
			
			if(Cid.Snp[k][i])
			{
				//printf("MemcpyStep:snap%d,pcs%d,dev%d,numD=%d\n",i+1,HostMpiRank,k,numD);
				err = hipSetDevice(Cid.Rank[k]);
				if(err != 0) printf("err = %d, errS=%s, error may occur at setdev\n",err, hipGetErrorString(err) );
				hipMemcpy(D_DSpt[i][k].Rsn, DSpt[i][k].Rsn, numD*sizeof(int), hipMemcpyHostToDevice);	
				hipMemcpy(D_DSpt[i][k].Gsn, DSpt[i][k].Gsn, numD*sizeof(int), hipMemcpyHostToDevice);	
				hipMemcpy(D_DSpt[i][k].locx, DSpt[i][k].locx, numD*sizeof(int), hipMemcpyHostToDevice);	
				hipMemcpy(D_DSpt[i][k].locy, DSpt[i][k].locy, numD*sizeof(int), hipMemcpyHostToDevice);	
				err = hipMemcpy(D_DSpt[i][k].locz, DSpt[i][k].locz, numD*sizeof(int), hipMemcpyHostToDevice);	
				if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy\n",err, hipGetErrorString(err) );
				D_DSpt[i][k].tinv = DSpt[i][k].tinv;
				D_DSpt[i][k].cmp = DSpt[i][k].cmp;//transfer to device side par struct, do Value-Trans when use
			}
		}

	}
	
	
	//HSW and DSW allocation
	for(i=0;i<nsnap;i++)
	{
		nTime = ceil(1.0*this->nt/HSpt[i].tinv);
		//printf("for snap[%d],PCS[%d]->HSW holds time points as %d and spatial point as %d \n",i+1,HostMpiRank,nTime,CSpn[i]);
		
		if(HSpt[i].cmp==1 || HSpt[i].cmp==3)
		{
			HSW[i].Vx = new Real[ nTime*CSpn[i] ]();	HSW[i].Vy = new Real[ nTime*CSpn[i] ]();	HSW[i].Vz = new Real[ nTime*CSpn[i] ]();
		}
		if(HSpt[i].cmp==2 || HSpt[i].cmp==3)
		{
			HSW[i].Txx = new Real[ nTime*CSpn[i] ](); 	HSW[i].Tyy = new Real[ nTime*CSpn[i] ]();	HSW[i].Tzz = new Real[ nTime*CSpn[i] ]();
			HSW[i].Txy = new Real[ nTime*CSpn[i] ]();	HSW[i].Txz = new Real[ nTime*CSpn[i] ]();	HSW[i].Tyz = new Real[ nTime*CSpn[i] ]();
		}

#ifdef DevicePick
		for(k=0;k<Cid.DNum;k++)
		{
			//printf("inDev-Snap[%d],PCS[%d],Dev[%d]->HSW holds time points as %d and spatial point as %d \n",i+1,HostMpiRank,k,nTime,Cid.Snp[k][i]);
			if(Cid.Snp[k][i])
			{
				err = hipSetDevice(Cid.Rank[k]);
				if(err != 0) printf("err = %d, errS=%s, error may occur at setdev\n",err, hipGetErrorString(err) );
				if(HSpt[i].cmp==1 || HSpt[i].cmp==3)
				{
					hipMalloc( (Real**)&DSW[i][k].Vx, sizeof(Real)*nTime*Cid.Snp[k][i]);	
					hipMalloc( (Real**)&DSW[i][k].Vy, sizeof(Real)*nTime*Cid.Snp[k][i]);	
					err = hipMalloc( (Real**)&DSW[i][k].Vz, sizeof(Real)*nTime*Cid.Snp[k][i]);
					if(err != 0) printf("err = %d, errS=%s, error may occur at malloc 1\n",err, hipGetErrorString(err) );
				}
				if(HSpt[i].cmp==2 || HSpt[i].cmp==3)
				{
					hipMalloc( (Real**)&DSW[i][k].Txx, sizeof(Real)*nTime*Cid.Snp[k][i]); 	
					hipMalloc( (Real**)&DSW[i][k].Tyy, sizeof(Real)*nTime*Cid.Snp[k][i]);	
					hipMalloc( (Real**)&DSW[i][k].Tzz, sizeof(Real)*nTime*Cid.Snp[k][i]);
					hipMalloc( (Real**)&DSW[i][k].Txy, sizeof(Real)*nTime*Cid.Snp[k][i]);	
					hipMalloc( (Real**)&DSW[i][k].Txz, sizeof(Real)*nTime*Cid.Snp[k][i]);	
					err = hipMalloc( (Real**)&DSW[i][k].Tyz, sizeof(Real)*nTime*Cid.Snp[k][i]);
					if(err != 0) printf("err = %d, errS=%s, error may occur at malloc for snap[%d]pcs[%d]dev[%d]\n",
							     err, hipGetErrorString(err), i+1,HostMpiRank,k );
				}
			}
		}
#endif
	
	}
#endif
	fprintf(stdout,"Procs[%d], pass snappick()\n",HostMpiRank);

}
void ChildProcs::SWpick(wfield *speW, int currT)
{
#ifndef PointOnly	
	int i,j,k;
	int time,nTime;
	int src,dst;
	int idx,idy,idz;

	
	for(j=0;j<this->nsnap;j++)
	{
		if( currT%HSpt[j].tinv != 0 )
			continue;

		time = currT/HSpt[j].tinv;
		nTime = ceil(1.0*this->nt/HSpt[j].tinv);
		//printf("for snap[%d],INOtime=%d,----->pickTime=%d\n",j+1,currT,time);
		
		for(i=0;i<Cid.DNum;i++)
		{
			if(Cid.Snp[i][j])
			{
				hipSetDevice( Cid.Rank[i] );
				//wave pick by kernel, Abandoned
				//SnapWavefieldPick<<<BlockPerGrid,ThreadPerBlock>>>(speW[i], this->DSW[j][i], D_DSpt[j][i], time, Cid.Snp[i][j], nTime);
				//CC(hipDeviceSynchronize());
				for(k=0;k<Cid.Snp[i][j];k++)
				{
					//(Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz;
					idx = DSpt[j][i].locx[k]-Cstart-(Cid.xl[i]-LenFD);
					idy = DSpt[j][i].locy[k]-(Cid.yd[i]-LenFD);
					idz = DSpt[j][i].locz[k];
					
#ifdef DevicePick
					//pick to Device
					dst = DSpt[j][i].Rsn[k]*nTime + time;
					src = idx*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz + idy*cdx.nz + idz;
					if(DSpt[j][i].cmp==1 || DSpt[j][i].cmp==3)
					{
						hipMemcpy(DSW[j][i].Vx+dst, speW[i].Vx+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Vy+dst, speW[i].Vy+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Vz+dst, speW[i].Vz+src, sizeof(Real), hipMemcpyDeviceToDevice);
					}
					if(DSpt[j][i].cmp==2 || DSpt[j][i].cmp==3)
					{
						hipMemcpy(DSW[j][i].Txx+dst, speW[i].Txx+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Tyy+dst, speW[i].Tyy+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Tzz+dst, speW[i].Tzz+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Txy+dst, speW[i].Txy+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Txz+dst, speW[i].Txz+src, sizeof(Real), hipMemcpyDeviceToDevice);
						hipMemcpy(DSW[j][i].Tyz+dst, speW[i].Tyz+src, sizeof(Real), hipMemcpyDeviceToDevice);
					}
					
#else
					
					//pick to Host
					dst = DSpt[j][i].Gsn[k]*nTime + time;
					src = idx*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz + idy*cdx.nz + idz;
					if(DSpt[j][i].cmp==1 || DSpt[j][i].cmp==3)
					{
						hipMemcpy(HSW[j].Vx+dst, speW[i].Vx+src, sizeof(Real),   hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Vy+dst, speW[i].Vy+src, sizeof(Real),   hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Vz+dst, speW[i].Vz+src, sizeof(Real),   hipMemcpyDeviceToHost);
					}
					if(DSpt[j][i].cmp==2 || DSpt[j][i].cmp==3)
					{
						hipMemcpy(HSW[j].Txx+dst, speW[i].Txx+src, sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Tyy+dst, speW[i].Tyy+src, sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Tzz+dst, speW[i].Tzz+src, sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Txy+dst, speW[i].Txy+src, sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Txz+dst, speW[i].Txz+src, sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[j].Tyz+dst, speW[i].Tyz+src, sizeof(Real), hipMemcpyDeviceToHost);
					}
					
#endif
				}

			}

		}

	}
#endif	

}
void ChildProcs::SWgather(int currT)
{
	//from Device to Host, DSW[i][j] to HSW[i], D2H
	//point is low dimension, time is fast dimension
#ifndef PointOnly	
#ifdef DevicePick	
	int i,j,k;
	int src,dst;
	int Tlen,nTime;//here is total time length, in iteration is time step
	for(k=0;k<this->nsnap;k++)
	{
		Tlen = ceil(1.0*currT/HSpt[k].tinv);
		nTime = ceil(1.0*this->nt/HSpt[k].tinv);
		//printf("for snap[%d],OTtime=%d,----->tinv=%d,Tlen=%d,nTime=%d\n",k+1,currT,HSpt[k].tinv,Tlen,nTime);
		
		for(i=0;i<Cid.DNum;i++)
		{
			if(Cid.Snp[i][k])
			{
				hipSetDevice( Cid.Rank[i] );
				for(j=0;j<Cid.Snp[i][k];j++)
				{
					src = DSpt[k][i].Rsn[j]*nTime;
					dst = DSpt[k][i].Gsn[j]*nTime;
					if(HSpt[k].cmp==1 || HSpt[k].cmp==3)
					{
						hipMemcpy(HSW[k].Vx+dst,  DSW[k][i].Vx+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Vy+dst,  DSW[k][i].Vy+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Vz+dst,  DSW[k][i].Vz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
					}
					if(HSpt[k].cmp==2 || HSpt[k].cmp==3)
					{
						hipMemcpy(HSW[k].Txx+dst, DSW[k][i].Txx+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Tyy+dst, DSW[k][i].Tyy+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Tzz+dst, DSW[k][i].Tzz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Txy+dst, DSW[k][i].Txy+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Txz+dst, DSW[k][i].Txz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
						hipMemcpy(HSW[k].Tyz+dst, DSW[k][i].Tyz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
					}
				}
			}
		}
		
	}
#endif	
#endif

}

void ChildProcs::C2DFocalPick()
{
	int i,j,numD;
	hipError_t err;
	
	//if(HostMpiRank==3)
	for(i=0;i<Cid.DNum;i++)
	{
		numD = 0;
		for(j=0;j<this->fpn;j++)
			if(HFpt.locx[j]>=Cid.xl[i]+Cstart && HFpt.locx[j]<=Cid.xr[i]+Cstart && HFpt.locy[j]>=Cid.yd[i] && HFpt.locy[j]<=Cid.yu[i])
			{
				numD++;
			}

		//malloc host side DFpt; 
		Cid.fp[i] = numD;
		DFpt[i].Rsn = new int[numD]();	DFpt[i].Gsn = new int[numD](); 
		DFpt[i].locx = new int[numD]();	DFpt[i].locy = new int[numD]();	DFpt[i].locz = new int[numD](); 
		
		//malloc device side focal data
		hipSetDevice(Cid.Rank[i]);
		hipMalloc( (int**)&Rmnt[i].locx, numD*sizeof(int) );
		hipMalloc( (int**)&Rmnt[i].locy, numD*sizeof(int) );
		hipMalloc( (int**)&Rmnt[i].locz, numD*sizeof(int) );
		hipMalloc( (Real**)&Rmnt[i].mxx, numD*sizeof(Real) );
		hipMalloc( (Real**)&Rmnt[i].myy, numD*sizeof(Real) );
		hipMalloc( (Real**)&Rmnt[i].mzz, numD*sizeof(Real) );
		hipMalloc( (Real**)&Rmnt[i].mxy, numD*sizeof(Real) );
		hipMalloc( (Real**)&Rmnt[i].mxz, numD*sizeof(Real) );
		err = hipMalloc( (Real**)&Rmnt[i].myz, numD*sizeof(Real) );
		if(err != 0) printf("err = %d, errS=%s, error may occur at Malloc Rmnt\n",err, hipGetErrorString(err) );
#ifdef SrcSmooth	
		hipMalloc( (Real**)&Rmnt[i].dnorm, numD*LenNorm*LenNorm*LenNorm*sizeof(Real) );
#endif

		hipMemset( Rmnt[i].locx, 0, numD*sizeof(int));
		hipMemset( Rmnt[i].locy, 0, numD*sizeof(int));
		hipMemset( Rmnt[i].locz, 0, numD*sizeof(int));
		hipMemset( Rmnt[i].mxx, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].myy, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].mzz, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].mxy, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].mxz, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].myz, 0, numD*sizeof(Real));
#ifdef SrcSmooth	
		err = hipMemset( Rmnt[i].dnorm, 0, numD*LenNorm*LenNorm*LenNorm*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset Rmnt\n",
				err, hipGetErrorString(err) );
#endif

		//malloc host side interp focal data
		IM[i].mxx = new Real[numD](); IM[i].myy = new Real[numD](); IM[i].mzz = new Real[numD]();
		IM[i].mxy = new Real[numD](); IM[i].mxz = new Real[numD](); IM[i].myz = new Real[numD]();

	}
	
#ifdef SrcSmooth
	int Src;
	Src=0;
#endif
	
	for(i=0;i<Cid.DNum;i++)
	{
		//assign DFpt in host and device side
		numD = 0;
		for(j=0;j<this->fpn;j++)
			if(HFpt.locx[j]>=Cid.xl[i]+Cstart && HFpt.locx[j]<=Cid.xr[i]+Cstart && HFpt.locy[j]>=Cid.yd[i] && HFpt.locy[j]<=Cid.yu[i])
			{
				DFpt[i].Rsn[numD] = numD;
				DFpt[i].Gsn[numD] = HFpt.Rsn[j];
				DFpt[i].locx[numD] = HFpt.locx[j];
				DFpt[i].locy[numD] = HFpt.locy[j];
				DFpt[i].locz[numD] = HFpt.locz[j];
				numD++;
			}
		
		hipSetDevice(Cid.Rank[i]);
		hipMemcpy(Rmnt[i].locx, DFpt[i].locx, Cid.fp[i]*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(Rmnt[i].locy, DFpt[i].locy, Cid.fp[i]*sizeof(int), hipMemcpyHostToDevice);
		err = hipMemcpy(Rmnt[i].locz, DFpt[i].locz, Cid.fp[i]*sizeof(int), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy Rmnt loc\n", err, hipGetErrorString(err) );
#ifdef SrcSmooth
		//start from 0, copy Cid.fp[i], then shift Cid.fp[i];
		//printf("at PCS[%d]Dev[%d], memcpy dorm, from %d\n",HostMpiRank, i, Src);
		err = hipMemcpy(Rmnt[i].dnorm,  H_Rmnt.dnorm+Src,  Cid.fp[i]*LenNorm*LenNorm*LenNorm*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy Rmnt loc\n", err, hipGetErrorString(err) );
		Src = (Src + Cid.fp[i])*LenNorm*LenNorm*LenNorm;
#endif

	}
	
	fprintf(stdout,"PCS[%d], pass focalpick\n",HostMpiRank);
}

Real ChildProcs::ExtractValue(Real *stf, Real Tstart, Real Tend, Real DT, Real time)
{
	int PL, PR;
	Real value;

	value=0;
	
	if( time > Tend || time< Tstart )
		value = 0;
	else
	{
		if( fabs( time-DT*floor(time/DT) ) < 1e-5 )
		{
			PL = (int) (time/DT);
			PR = PL;
		}
		else
		{
			PL = (int)(time/DT);
			PR = PL+1;
		}

		value = stf[PL] + (stf[PR]-stf[PL])*( (time-DT*PL)/DT );
	}

	return value;

}

void ChildProcs::InterpFocus(Real time)
{
	int i,j;
	int Src;
	int numD;
	hipError_t err;
	
	Real Tstart,Tend;//orginal focal data time sereis length
	//current time---->time = (currt + tinc*0.5)*stept;

	Tstart = 0;	Tend = FDT*(FNT-1);

	//H_Rmnt.mxx[ fpn*FNT ]
	//IM[ Cid.DNum ].mxx[ Cid.fp[i] ]
	//Rmnt[ Cid.DNum ].mxx[ Cid.fp[i] ]
		
	//interp
	for(i=0;i<Cid.DNum;i++)
	{
		for(j=0;j<Cid.fp[i];j++)
		{
			Src = DFpt[i].Gsn[j] * FNT;
			//Dst = DFpt[i].Rsn[j];//j
			// from H_Rmnt.mxx[Src] 
			// to IM[i].mxx[Dst]

			IM[i].mxx[j] =  ExtractValue( H_Rmnt.mxx+Src, Tstart, Tend, FDT, time);
			IM[i].myy[j] =  ExtractValue( H_Rmnt.myy+Src, Tstart, Tend, FDT, time);
			IM[i].mzz[j] =  ExtractValue( H_Rmnt.mzz+Src, Tstart, Tend, FDT, time);
			IM[i].mxy[j] =  ExtractValue( H_Rmnt.mxy+Src, Tstart, Tend, FDT, time);
			IM[i].mxz[j] =  ExtractValue( H_Rmnt.mxz+Src, Tstart, Tend, FDT, time);
			IM[i].myz[j] =  ExtractValue( H_Rmnt.myz+Src, Tstart, Tend, FDT, time);

		}
	}

	//memcpy
	for(i=0;i<Cid.DNum;i++)
	{
		numD = Cid.fp[i];
		
		hipSetDevice(Cid.Rank[i]);
		//flush first
		err = hipMemset( Rmnt[i].mxx, 0, numD*sizeof(Real));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memset Rmnt mxx at interpstep\n", err, hipGetErrorString(err) );
		hipMemset( Rmnt[i].myy, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].mzz, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].mxy, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].mxz, 0, numD*sizeof(Real));
		hipMemset( Rmnt[i].myz, 0, numD*sizeof(Real));
		
		err = hipMemcpy( Rmnt[i].mxx,  IM[i].mxx,  numD*sizeof(Real), hipMemcpyHostToDevice);
		if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy Rmnt mxx at intepstep\n", err, hipGetErrorString(err) );
		hipMemcpy( Rmnt[i].myy,  IM[i].myy,  numD*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy( Rmnt[i].mzz,  IM[i].mzz,  numD*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy( Rmnt[i].mxy,  IM[i].mxy,  numD*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy( Rmnt[i].mxz,  IM[i].mxz,  numD*sizeof(Real), hipMemcpyHostToDevice);
		hipMemcpy( Rmnt[i].myz,  IM[i].myz,  numD*sizeof(Real), hipMemcpyHostToDevice);

	}

}

void ChildProcs::C2DPointPick()
{
	int i,j,numD;
	hipError_t err;
	
	//if(HostMpiRank==3)
	for(i=0;i<Cid.DNum;i++)
	{
		numD = 0;
		for(j=0;j<this->ppn;j++)
			if(Hpt.locx[j]>=Cid.xl[i]+Cstart && Hpt.locx[j]<=Cid.xr[i]+Cstart && Hpt.locy[j]>=Cid.yd[i] && Hpt.locy[j]<=Cid.yu[i])
			{
				numD++;
			}

		//malloc host side Dpt and device side Dpt;
		Cid.np[i] = numD;
		Dpt[i].Rsn = new int[numD]();	Dpt[i].Gsn = new int[numD](); 
		Dpt[i].locx = new int[numD]();	Dpt[i].locy = new int[numD]();	Dpt[i].locz = new int[numD](); 
		
		hipSetDevice(Cid.Rank[i]);
		hipMalloc( (int**)&D_Dpt[i].Rsn, numD*sizeof(int));
		hipMalloc( (int**)&D_Dpt[i].Gsn, numD*sizeof(int));
		hipMalloc( (int**)&D_Dpt[i].locx, numD*sizeof(int));
		hipMalloc( (int**)&D_Dpt[i].locy, numD*sizeof(int));
		err=hipMalloc( (int**)&D_Dpt[i].locz, numD*sizeof(int));
		if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy point loc\n", err, hipGetErrorString(err) );
		
#ifdef DevicePick		
		//malloc device wave point buffer
		hipMalloc( (Real**)&DPW[i].Vx, nt*Cid.np[i]*sizeof(Real) );
		hipMalloc( (Real**)&DPW[i].Vy, nt*Cid.np[i]*sizeof(Real) ); 
		hipMalloc( (Real**)&DPW[i].Vz, nt*Cid.np[i]*sizeof(Real) ); 
		hipMalloc( (Real**)&DPW[i].Txx, nt*Cid.np[i]*sizeof(Real) ); 
		hipMalloc( (Real**)&DPW[i].Tyy, nt*Cid.np[i]*sizeof(Real) ); 
		hipMalloc( (Real**)&DPW[i].Tzz, nt*Cid.np[i]*sizeof(Real) ); 
		hipMalloc( (Real**)&DPW[i].Txy, nt*Cid.np[i]*sizeof(Real) ); 
		hipMalloc( (Real**)&DPW[i].Txz, nt*Cid.np[i]*sizeof(Real) ); 
		err=hipMalloc( (Real**)&DPW[i].Tyz, nt*Cid.np[i]*sizeof(Real) ); 
		if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy DPW \n", err, hipGetErrorString(err) );
#endif

	}
	
	for(i=0;i<Cid.DNum;i++)
	{
		//assign Dpt in host and device side
		numD = 0;
		for(j=0;j<this->ppn;j++)
			if(Hpt.locx[j]>=Cid.xl[i]+Cstart && Hpt.locx[j]<=Cid.xr[i]+Cstart && Hpt.locy[j]>=Cid.yd[i] && Hpt.locy[j]<=Cid.yu[i])
			{
				Dpt[i].Rsn[numD] = numD;
				Dpt[i].Gsn[numD] = Hpt.Rsn[j];
				Dpt[i].locx[numD] = Hpt.locx[j];
				Dpt[i].locy[numD] = Hpt.locy[j];
				Dpt[i].locz[numD] = Hpt.locz[j];
				numD++;
			}
	
		hipSetDevice(Cid.Rank[i]);
		hipMemcpy(D_Dpt[i].Rsn, Dpt[i].Rsn, Cid.np[i]*sizeof(int), hipMemcpyHostToDevice);	
		hipMemcpy(D_Dpt[i].Gsn, Dpt[i].Gsn, Cid.np[i]*sizeof(int), hipMemcpyHostToDevice);	
		hipMemcpy(D_Dpt[i].locx, Dpt[i].locx, Cid.np[i]*sizeof(int), hipMemcpyHostToDevice);	
		hipMemcpy(D_Dpt[i].locy, Dpt[i].locy, Cid.np[i]*sizeof(int), hipMemcpyHostToDevice);	
		err=hipMemcpy(D_Dpt[i].locz, Dpt[i].locz, Cid.np[i]*sizeof(int), hipMemcpyHostToDevice);	
		if(err != 0) printf("err = %d, errS=%s, error may occur at memcpy point loc\n", err, hipGetErrorString(err) );
	}
	fprintf(stdout,"Procs[%d], pass pointpick()\n",HostMpiRank);

}
void ChildProcs::PWgather(int currT)
{

#ifdef DevicePick	
	//from Device to Host, DPW[i] to HPW, D2H
	//point is low dimension, time is fast dimension
	int i,j;
	int src,dst;
	int Tlen;
	Tlen = currT;//gen ju shi jian wei zhi tiao zheng
	//Tlen = currT+1;//gen ju shi jian wei zhi tiao zheng
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		for(j=0;j<Cid.np[i];j++)
		{
			src = Dpt[i].Rsn[j]*this->nt;
			dst = Dpt[i].Gsn[j]*this->nt;
			hipMemcpy(HPW.Vx+dst, DPW[i].Vx+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Vy+dst, DPW[i].Vy+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Vz+dst, DPW[i].Vz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Txx+dst, DPW[i].Txx+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Tyy+dst, DPW[i].Tyy+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Tzz+dst, DPW[i].Tzz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Txy+dst, DPW[i].Txy+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Txz+dst, DPW[i].Txz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Tyz+dst, DPW[i].Tyz+src, Tlen*sizeof(Real), hipMemcpyDeviceToHost);
		}
	}
#endif	
	
}
void ChildProcs::PWpick(wfield *speW, int currT)
{
	int i,j;
	int src,dst;
	int idx,idy,idz;

	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		//WavefieldPick<<<BlockPerGrid,ThreadPerBlock>>>(speW[i], this->DPW[i], D_Dpt[i], currT, Cid.np[i], nt);
		//CC(hipDeviceSynchronize());
		for(j=0;j<Cid.np[i];j++)
		{
			idx = Dpt[i].locx[j]-Cstart-(Cid.xl[i]-LenFD);
			idy = Dpt[i].locy[j]-(Cid.yd[i]-LenFD);
			idz = Dpt[i].locz[j];
			
#ifdef DevicePick			
			//pick to Device
			dst = Dpt[i].Rsn[j]*nt+currT;
			src = idx*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz + idy*cdx.nz + idz;
			hipMemcpy(DPW[i].Vx+dst, speW[i].Vx+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Vy+dst, speW[i].Vy+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Vz+dst, speW[i].Vz+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Txx+dst, speW[i].Txx+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Tyy+dst, speW[i].Tyy+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Tzz+dst, speW[i].Tzz+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Txy+dst, speW[i].Txy+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Txz+dst, speW[i].Txz+src, sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(DPW[i].Tyz+dst, speW[i].Tyz+src, sizeof(Real), hipMemcpyDeviceToDevice);
#else			
			//pick to Host side
			dst = Dpt[i].Gsn[j]*nt+currT;
			src = idx*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz + idy*cdx.nz + idz;
			hipMemcpy(HPW.Vx+dst, speW[i].Vx+src, sizeof(Real),   hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Vy+dst, speW[i].Vy+src, sizeof(Real),   hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Vz+dst, speW[i].Vz+src, sizeof(Real),   hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Txx+dst, speW[i].Txx+src, sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Tyy+dst, speW[i].Tyy+src, sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Tzz+dst, speW[i].Tzz+src, sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Txy+dst, speW[i].Txy+src, sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Txz+dst, speW[i].Txz+src, sizeof(Real), hipMemcpyDeviceToHost);
			hipMemcpy(HPW.Tyz+dst, speW[i].Tyz+src, sizeof(Real), hipMemcpyDeviceToHost);
			
#endif	
		}

	}
}


//------------------------------private-------------------------
void ChildProcs::GpuAbility(const char *filename)
{
	char parpath[SeisStrLen];
	char name[SeisStrLen2];
	char errstr[SeisStrLen];
	char devfile[SeisStrLen];
	int i,j;
	int B1,B2,B3,T1,T2,T3;
	int deviceNum;
	int pid=0;
	char hostname[256];
	char Str[256];

	FILE *fp;
	fp = fopen(filename,"r");
	if(!fp)
	{
		sprintf(errstr,"Fail to open main par file %s in GpuAbility", filename);
		errprt(Fail2Open,errstr);
	}
	com.get_conf(fp, "seispath", 3, parpath);
	com.get_conf(fp, "device_filename", 3, name);
	com.get_conf(fp, "BlockPerGrid", 3, &B1);
	com.get_conf(fp, "BlockPerGrid", 4, &B2);
	com.get_conf(fp, "BlockPerGrid", 5, &B3);
	com.get_conf(fp, "ThreadPerBlock", 3, &T1);
	com.get_conf(fp, "ThreadPerBlock", 4, &T2);
	com.get_conf(fp, "ThreadPerBlock", 5, &T3);
	fclose(fp);

	sprintf(devfile,"%s/%s",parpath,name);

	//comfirm lanuch parameters should with boundary
	cdx.nx <= B2 ? BlockPerGrid.y = cdx.nx : BlockPerGrid.y = B2;
	cdx.ny <= B1 ? BlockPerGrid.x = cdx.ny : BlockPerGrid.x = B1;
	cdx.nz <= T1 ? ThreadPerBlock.x = cdx.nz : ThreadPerBlock.x = T1;
	BlockPerGrid.z = 1;
	ThreadPerBlock.y = 1; 
	ThreadPerBlock.z = 1;
	//BPG(B1,B2,1) TPB(T1,1,1)
	//<<<BPG,TPB>>>

	//check device number
	gethostname(hostname,256);
	pid=getpid();
	hipGetDeviceCount(&deviceNum);

	printf("\nOn current node %s, Rank is %d and PID is %d, has %d GPU device\n",hostname,HostMpiRank,pid,deviceNum);

	//DEC confirmation
	fp = fopen(devfile,"r");

	memset(Str,'\0',256*sizeof(char));
	sprintf(Str,"used_device_number_%s",hostname);

	com.get_conf(fp, Str, 3, &Cid.DNum);
	if(Cid.DNum > deviceNum)
		Cid.DNum = deviceNum;

	Cid.Snp = new int*[Cid.DNum];
	for(int iii=0;iii<Cid.DNum;iii++)
		Cid.Snp[iii] = new int [this->nsnap]();
	Cid.fp = new int [Cid.DNum]();//focal number
	Cid.np = new int [Cid.DNum]();//point number
	Cid.Rank = new int [Cid.DNum]();
	Cid.Size = new int [Cid.DNum]();
	Cid.xl = new int [Cid.DNum]();
	Cid.xr = new int [Cid.DNum]();
	Cid.yd = new int [Cid.DNum]();
	Cid.yu = new int [Cid.DNum]();
	BPG = new dim3 [Cid.DNum]();

	memset(Str,'\0',256*sizeof(char));
	sprintf(Str,"device_ydims_%s",hostname);
	com.get_conf(fp, Str, 3, &Cid.ydim);

	memset(Str,'\0',256*sizeof(char));
	sprintf(Str,"used_device_id_%s",hostname);
	
	for(i=0;i<Cid.DNum;i++)
		com.get_conf(fp, Str, 3+i, &Cid.Rank[i]);
	printf("On this node %s, totally used %d devices, the Device ID is :",hostname,Cid.DNum);
	for(i=0;i<Cid.DNum;i++)
		printf("%d ",Cid.Rank[i]);
	cout<<endl;
	
	fclose(fp);
	
	
	//enable P2P
	//enable unified memory
	int tempflag=0;
	//hipError_t errmessage;
	hipDeviceProp_t deviceProp;
	cout<<"Check needed GPU device features (P2P, ManagedMemory)\n";
	for(i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		hipGetDeviceProperties(&deviceProp,Cid.Rank[i]);
		
		//P2P
		for(j=0;j<Cid.DNum;j++)
		{
			if(Cid.Rank[i]==Cid.Rank[j])
				continue;
			checkCudaErrors(hipDeviceCanAccessPeer(&tempflag,Cid.Rank[i],Cid.Rank[j]));//on I to active J
			if(tempflag)
			{
				if(hipSuccess != hipDeviceEnablePeerAccess(Cid.Rank[j],0))
					printf("On device %d to active device %d, errmessage is %s\n",
					Cid.Rank[i],Cid.Rank[j],hipGetErrorString(hipGetLastError()));
			}
		}
		
		//Unified Memory
		if(!deviceProp.managedMemory)
			printf("On device %d doesn't support managed memory\n",Cid.Rank[i]);
		//concurrent managed access
		if(!deviceProp.concurrentManagedAccess)
			printf("On device %d doesn't support concurrently managed memory access by CPU\n",Cid.Rank[i]);
	}
	
}

void ChildProcs::wavesyn(wfield *Output, wfield *Input)
{
	int inputflag=1;
	int size;
	
	for(int i=0;i<Cid.DNum;i++)
	{
		hipSetDevice( Cid.Rank[i] );
		size = (Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz;
		if(inputflag)
		{//should use this, in GPU side;
			hipMemcpy(Output[i].Txx, Input[i].Txx, size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Tyy, Input[i].Tyy, size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Tzz, Input[i].Tzz, size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Txy, Input[i].Txy, size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Txz, Input[i].Txz, size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Tyz, Input[i].Tyz, size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Vx,  Input[i].Vx,  size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Vy,  Input[i].Vy,  size*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(Output[i].Vz,  Input[i].Vz,  size*sizeof(Real), hipMemcpyDeviceToDevice);
		}
		else
		{
			memcpy(Output[i].Txx, Input[i].Txx, size*sizeof(Real) );
			memcpy(Output[i].Tyy, Input[i].Tyy, size*sizeof(Real) );
			memcpy(Output[i].Tzz, Input[i].Tzz, size*sizeof(Real) );
			memcpy(Output[i].Txy, Input[i].Txy, size*sizeof(Real) );
			memcpy(Output[i].Txz, Input[i].Txz, size*sizeof(Real) );
			memcpy(Output[i].Tyz, Input[i].Tyz, size*sizeof(Real) );
			memcpy(Output[i].Vx,  Input[i].Vx,  size*sizeof(Real) );
			memcpy(Output[i].Vy,  Input[i].Vy,  size*sizeof(Real) );
			memcpy(Output[i].Vz,  Input[i].Vz,  size*sizeof(Real) );
		}
	}

}

void ChildProcs::abssyn(int TransDir)
{
	int Xsize,Ysize,Zsize;
	//before a new RKite, input FW as fixed field,
	//everytimes reflush mW and update W,
	//after RKite, store FW from W.
	//So, TD=1 means FW to mW,	TD=2 means W to mW,	TD=3 means W to FW;

#ifdef DisBug
	//printf("input ABS-syn direction is %d\n",TransDir);
#endif
	
	if(TransDir!=1 && TransDir!=2 && TransDir!=3)
		printf("Absorption wavefield synchronizing direction error\n");
	for(int i=0; i<Cid.DNum; i++)
	{
		hipSetDevice( Cid.Rank[i] );
		Xsize = (H_apr.nabs[0]+H_apr.nabs[1])*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD)*cdx.nz;
		Ysize = (H_apr.nabs[2]+H_apr.nabs[3])*(Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*cdx.nz;
		Zsize = (H_apr.nabs[4]+H_apr.nabs[5])*(Cid.xr[i]-Cid.xl[i]+1 + 2*LenFD)*(Cid.yu[i]-Cid.yd[i]+1 + 2*LenFD);
		
		if(TransDir==1)
		{
			hipMemcpy(mAx[i].Txx, FAx[i].Txx, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Tyy, FAx[i].Tyy, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Tzz, FAx[i].Tzz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Txy, FAx[i].Txy, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Txz, FAx[i].Txz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Tyz, FAx[i].Tyz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Vx,  FAx[i].Vx,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Vy,  FAx[i].Vy,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Vz,  FAx[i].Vz,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);

			hipMemcpy(mAy[i].Txx, FAy[i].Txx, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Tyy, FAy[i].Tyy, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Tzz, FAy[i].Tzz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Txy, FAy[i].Txy, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Txz, FAy[i].Txz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Tyz, FAy[i].Tyz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Vx,  FAy[i].Vx,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Vy,  FAy[i].Vy,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Vz,  FAy[i].Vz,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);

			hipMemcpy(mAz[i].Txx, FAz[i].Txx, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Tyy, FAz[i].Tyy, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Tzz, FAz[i].Tzz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Txy, FAz[i].Txy, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Txz, FAz[i].Txz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Tyz, FAz[i].Tyz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Vx,  FAz[i].Vx,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Vy,  FAz[i].Vy,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Vz,  FAz[i].Vz,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
		}

		if(TransDir==2)
		{
			hipMemcpy(mAx[i].Txx, Ax[i].Txx, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Tyy, Ax[i].Tyy, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Tzz, Ax[i].Tzz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Txy, Ax[i].Txy, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Txz, Ax[i].Txz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Tyz, Ax[i].Tyz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Vx,  Ax[i].Vx,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Vy,  Ax[i].Vy,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAx[i].Vz,  Ax[i].Vz,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);

			hipMemcpy(mAy[i].Txx, Ay[i].Txx, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Tyy, Ay[i].Tyy, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Tzz, Ay[i].Tzz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Txy, Ay[i].Txy, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Txz, Ay[i].Txz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Tyz, Ay[i].Tyz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Vx,  Ay[i].Vx,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Vy,  Ay[i].Vy,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAy[i].Vz,  Ay[i].Vz,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);

			hipMemcpy(mAz[i].Txx, Az[i].Txx, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Tyy, Az[i].Tyy, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Tzz, Az[i].Tzz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Txy, Az[i].Txy, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Txz, Az[i].Txz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Tyz, Az[i].Tyz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Vx,  Az[i].Vx,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Vy,  Az[i].Vy,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(mAz[i].Vz,  Az[i].Vz,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
		}

		if(TransDir==3)
		{
			hipMemcpy(FAx[i].Txx, Ax[i].Txx, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Tyy, Ax[i].Tyy, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Tzz, Ax[i].Tzz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Txy, Ax[i].Txy, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Txz, Ax[i].Txz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Tyz, Ax[i].Tyz, Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Vx,  Ax[i].Vx,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Vy,  Ax[i].Vy,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAx[i].Vz,  Ax[i].Vz,  Xsize*sizeof(Real), hipMemcpyDeviceToDevice);

			hipMemcpy(FAy[i].Txx, Ay[i].Txx, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Tyy, Ay[i].Tyy, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Tzz, Ay[i].Tzz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Txy, Ay[i].Txy, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Txz, Ay[i].Txz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Tyz, Ay[i].Tyz, Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Vx,  Ay[i].Vx,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Vy,  Ay[i].Vy,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAy[i].Vz,  Ay[i].Vz,  Ysize*sizeof(Real), hipMemcpyDeviceToDevice);

			hipMemcpy(FAz[i].Txx, Az[i].Txx, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Tyy, Az[i].Tyy, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Tzz, Az[i].Tzz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Txy, Az[i].Txy, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Txz, Az[i].Txz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Tyz, Az[i].Tyz, Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Vx,  Az[i].Vx,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Vy,  Az[i].Vy,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
			hipMemcpy(FAz[i].Vz,  Az[i].Vz,  Zsize*sizeof(Real), hipMemcpyDeviceToDevice);
		}
		

	}

}



//-----------------------------------------kernel-----------------------------------------

__global__ void VelPDcoeff(derivF drv, mdparF mpa, apara apr, Real *matVx2Vz, Real *matVy2Vz)
{
	//this part only used under free surface condition and that means must apply CondFree macro
	//when apply free surface, the top layer doesn't need absorbtion, so only need to calculate
	//damped velocity partial derivative on four side and one bottom (without the inner of the top layer)

	//int i,j,k;
	//  <<<BPG.y, BPG.x>>>
	//gridDim.x<=cdx.nx  blockDim.x<=cdx.ny
	int countX,countY;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int xiaoI;
	int i;

	Real e11,e12,e13,e21,e22,e23,e31,e32,e33;
	Real lambda,miu,lam2mu;
	Real A[9],B[9],C[9],temp[9];
	Real Bzx,Bzy;//use for PML
	
	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.x)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.x + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=blockDim.x)
			{
				idy = countY + threadIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{  
					idz = ipam[8]+LenFD-1;

					Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
#ifdef CFSPML
					Bzx = apr.Bz[idz]/apr.Bx[idx];	Bzy = apr.Bz[idz]/apr.By[idy];
#else	
					Bzx = 1.0;	Bzy = 1.0;
#endif
					miu = mpa.rho[Gindex]*mpa.beta[Gindex]*mpa.beta[Gindex];
					lam2mu = mpa.rho[Gindex]*mpa.alpha[Gindex]*mpa.alpha[Gindex];
					lambda = lam2mu - 2.0*miu;

					e11 = drv.xix[Gindex];
					e12 = drv.xiy[Gindex];
					e13 = drv.xiz[Gindex];
					e21 = drv.etax[Gindex];
					e22 = drv.etay[Gindex];
					e23 = drv.etaz[Gindex];
					e31 = drv.zetax[Gindex];
					e32 = drv.zetay[Gindex];
					e33 = drv.zetaz[Gindex];
					
					A[0] = lam2mu*e31*e31 + miu*(e32*e32+e33*e33);
					A[1] = lambda*e31*e32 + miu*e32*e31;
					A[2] = lambda*e31*e33 + miu*e33*e31;
					A[3] = lambda*e32*e31 + miu*e31*e32;
					A[4] = lam2mu*e32*e32 + miu*(e31*e31+e33*e33);
					A[5] = lambda*e32*e33 + miu*e33*e32;
					A[6] = lambda*e33*e31 + miu*e31*e33;
					A[7] = lambda*e33*e32 + miu*e32*e33;
					A[8] = lam2mu*e33*e33 + miu*(e31*e31+e32*e32);

					matinv(A);

					B[0] = lam2mu*e31*e11 + miu*(e32*e12+e33*e13);
					B[1] = lambda*e31*e12 + miu*e32*e11;
					B[2] = lambda*e31*e13 + miu*e33*e11;
					B[3] = lambda*e32*e11 + miu*e31*e12;
					B[4] = lam2mu*e32*e12 + miu*(e31*e11+e33*e13);
					B[5] = lambda*e32*e13 + miu*e33*e12;
					B[6] = lambda*e33*e11 + miu*e31*e13;
					B[7] = lambda*e33*e12 + miu*e32*e13;
					B[8] = lam2mu*e33*e13 + miu*(e32*e12+e31*e11);

					C[0] = lam2mu*e31*e21 + miu*(e32*e22+e33*e23);
					C[1] = lambda*e31*e22 + miu*e32*e21;
					C[2] = lambda*e31*e23 + miu*e33*e21;
					C[3] = lambda*e32*e21 + miu*e31*e22;
					C[4] = lam2mu*e32*e22 + miu*(e31*e21+e33*e23);
					C[5] = lambda*e32*e23 + miu*e33*e22;
					C[6] = lambda*e33*e21 + miu*e31*e23;
					C[7] = lambda*e33*e22 + miu*e32*e23;
					C[8] = lam2mu*e33*e23 + miu*(e31*e21+e32*e22);

					xiaoI=idx*(ipam[5]-ipam[4]+1+2*LenFD)*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;//valid Y

					matmul(A,B,temp);
					for(i=0;i<SeisGeo*SeisGeo;i++)
						matVx2Vz[xiaoI+i] = -1*temp[i]*Bzx;

					matmul(A,C,temp);
					for(i=0;i<SeisGeo*SeisGeo;i++)
						matVy2Vz[xiaoI+i] = -1*temp[i]*Bzy;

				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__device__ void matmul(Real *A, Real *B, Real *C)
{
	int n;
	int i,j,k;
	n=3;

	for(i=0;i<n;i++)
		for(j=0;j<n;j++)
		{
			C[i*n+j] = 0.0;
			for(k=0;k<n;k++)
				C[i*n+j]=C[i*n+j]+A[i*n+k]*B[k*n+j];
		}
}
__device__ void matinv(Real *A)
{
	int i,j,k,n;
	
	n=3;
	Real con;
	for(i=0;i<n;i++)
	{
		con = A[i*n+i];
		A[i*n+i] = 1;
		for(j=0;j<n;j++)
			A[i*n+j]=A[i*n+j]/con;

		for(j=0;j<n;j++)
			if(j!=i)
			{
				con = A[j*n+i];
				A[j*n+i] = 0;
				for(k=0;k<n;k++)
					A[j*n+k]=A[j*n+k]-A[i*n+k]*con;
			}
	}
}


__global__ void perform()
{
	printf("display data: ");
	for(int i=0;i<11;i++)
		printf("%d ",ipam[i]);
	printf("\n");
}

__global__ void generatewave(wfield wfake, int time, int Cstart)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	Real value;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

							value = ( time*1E11 + (idx+Cstart+1+ipam[2]-LenFD)*1E6 + (idy+1+ipam[4]-LenFD)*1E3 + idz+1 )/1E6; 

							wfake.Vx[Gindex] = 1*1000 + value;
							wfake.Vy[Gindex] = 2*1000 + value;
							wfake.Vz[Gindex] = 3*1000 + value;
							wfake.Txx[Gindex] = 4*1000 + value;
							wfake.Tyy[Gindex] = 5*1000 + value;
							wfake.Tzz[Gindex] = 6*1000 + value;
							wfake.Txy[Gindex] = 7*1000 + value;
							wfake.Txz[Gindex] = 8*1000 + value;
							wfake.Tyz[Gindex] = 9*1000 + value;
							
							//check pass
							//if(idx==53 && idy==53 && idz==53)
							//	printf("index=%d, value=%lf,Txx=%lf\n",Gindex,value,wfake.Txx[Gindex]);

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void CalDiff(int Xvec, int Yvec, int Zvec, int ConIndex, Real steph, Real *CoVx, Real* CoVy, wfield W, PartialD pd)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int xiaoI;
#ifdef HYindex	
	int Hyindex;
#endif

	Real xstep, ystep, zstep;
	int xinc, yinc, zinc;
	
	xstep = steph*Xvec;
	ystep = steph*Yvec;
	zstep = steph*Zvec;
	xinc = Xvec*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD);//skip cdx.ny*cdx.nz
	yinc = Yvec*(ipam[8]+2*LenFD);//skip cdx.nz
	zinc = Zvec*1;//skip 1


	//generally use DRP/opt MacCormack scheme to get derivative, as Equation 2.23 and coefficients is Equation 2.24 in Thesis.
	//for the top layer transfrom the derivative of xi and eta to get zeta direction derivative, as Equation 3.4 in Thesis.
	
	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current device compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						//if(idz<ipam[8]+LenFD && idz>=ConIndex)//vaild point with one virtual bounds
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{

							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							xiaoI = idx*(ipam[5]-ipam[4]+1+2*LenFD)*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;//valid Y
							
							if(idz>=ConIndex)
							{
#ifdef HYindex
								Hyindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+LenFD-ConIndex) 
									+ idy*(ipam[8]+LenFD-ConIndex) + idz-ConIndex;
								pd.DxTyy[Hyindex] = DRPFD( W.Tyy, Gindex, xstep, xinc);
								pd.DxTzz[Hyindex] = DRPFD( W.Tzz, Gindex, xstep, xinc);
								pd.DxTyz[Hyindex] = DRPFD( W.Tyz, Gindex, xstep, xinc);
								pd.DyTxx[Hyindex] = DRPFD( W.Txx, Gindex, ystep, yinc);
								pd.DyTzz[Hyindex] = DRPFD( W.Tzz, Gindex, ystep, yinc);
								pd.DyTxz[Hyindex] = DRPFD( W.Txz, Gindex, ystep, yinc);
								pd.DzTxx[Hyindex] = DRPFD( W.Txx, Gindex, zstep, zinc);
								pd.DzTyy[Hyindex] = DRPFD( W.Tyy, Gindex, zstep, zinc);
								pd.DzTxy[Hyindex] = DRPFD( W.Txy, Gindex, zstep, zinc);
#else						
								pd.DxTyy[Gindex] = DRPFD( W.Tyy, Gindex, xstep, xinc);
								pd.DxTzz[Gindex] = DRPFD( W.Tzz, Gindex, xstep, xinc);
								pd.DxTyz[Gindex] = DRPFD( W.Tyz, Gindex, xstep, xinc);
								pd.DyTxx[Gindex] = DRPFD( W.Txx, Gindex, ystep, yinc);
								pd.DyTzz[Gindex] = DRPFD( W.Tzz, Gindex, ystep, yinc);
								pd.DyTxz[Gindex] = DRPFD( W.Txz, Gindex, ystep, yinc);
								pd.DzTxx[Gindex] = DRPFD( W.Txx, Gindex, zstep, zinc);
								pd.DzTyy[Gindex] = DRPFD( W.Tyy, Gindex, zstep, zinc);
								pd.DzTxy[Gindex] = DRPFD( W.Txy, Gindex, zstep, zinc);
#endif
							}

							pd.DxTxx[Gindex] = DRPFD( W.Txx, Gindex, xstep, xinc);
							pd.DxTxy[Gindex] = DRPFD( W.Txy, Gindex, xstep, xinc);
							pd.DxTxz[Gindex] = DRPFD( W.Txz, Gindex, xstep, xinc);
							pd.DxVx[Gindex] = DRPFD( W.Vx, Gindex, xstep, xinc);
							pd.DxVy[Gindex] = DRPFD( W.Vy, Gindex, xstep, xinc);
							pd.DxVz[Gindex] = DRPFD( W.Vz, Gindex, xstep, xinc);

							pd.DyTyy[Gindex] = DRPFD( W.Tyy, Gindex, ystep, yinc);
							pd.DyTxy[Gindex] = DRPFD( W.Txy, Gindex, ystep, yinc);
							pd.DyTyz[Gindex] = DRPFD( W.Tyz, Gindex, ystep, yinc);
							pd.DyVx[Gindex] = DRPFD( W.Vx, Gindex, ystep, yinc);
							pd.DyVy[Gindex] = DRPFD( W.Vy, Gindex, ystep, yinc);
							pd.DyVz[Gindex] = DRPFD( W.Vz, Gindex, ystep, yinc);

							pd.DzTzz[Gindex] = DRPFD( W.Tzz, Gindex, zstep, zinc);
							pd.DzTxz[Gindex] = DRPFD( W.Txz, Gindex, zstep, zinc);
							pd.DzTyz[Gindex] = DRPFD( W.Tyz, Gindex, zstep, zinc);
#ifndef CondFree //no free surface == full space == should apply ABS							
							pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);
							pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);
							pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);
#endif

							//  P(V3)/P(zeta), for VLOW, should deal with 3 top layer and other layer, totally 4 cases.
							//		   for VUCD, should deal with 3 top layer and other layer, totally 4 cases.
							//		   for Default, only deal with 1 top layer and other layer, totally 2 cases.
							//		   VLOW and VUCD, pick one!


#ifndef CondFreeVUCD//Velocity free surface condition---Unilateral compact difference
#ifdef CondFreeVLOW
							if(idz == ipam[8]+LenFD-1)//surface layer
							{
								//202
						pd.DzVx[Gindex] = CoVx[xiaoI+0]*pd.DxVx[Gindex] + CoVx[xiaoI+1]*pd.DxVy[Gindex] + CoVx[xiaoI+2]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+0]*pd.DyVx[Gindex] + CoVy[xiaoI+1]*pd.DyVy[Gindex] + CoVy[xiaoI+2]*pd.DyVz[Gindex];
						pd.DzVy[Gindex] = CoVx[xiaoI+3]*pd.DxVx[Gindex] + CoVx[xiaoI+4]*pd.DxVy[Gindex] + CoVx[xiaoI+5]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+3]*pd.DyVx[Gindex] + CoVy[xiaoI+4]*pd.DyVy[Gindex] + CoVy[xiaoI+5]*pd.DyVz[Gindex];
						pd.DzVz[Gindex] = CoVx[xiaoI+6]*pd.DxVx[Gindex] + CoVx[xiaoI+7]*pd.DxVy[Gindex] + CoVx[xiaoI+8]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+6]*pd.DyVx[Gindex] + CoVy[xiaoI+7]*pd.DyVy[Gindex] + CoVy[xiaoI+8]*pd.DyVz[Gindex];
							}
							else if(idz == ipam[8]+LenFD-2)//one layer inner surface
							{
								//201
								pd.DzVx[Gindex] = M22FD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = M22FD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = M22FD( W.Vz, Gindex, zstep, zinc);
							}
							else if(idz == ipam[8]+LenFD-3)//two layer inner surface
							{
								//200
								pd.DzVx[Gindex] = M24FD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = M24FD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = M24FD( W.Vz, Gindex, zstep, zinc);
							}
							else
							{
								pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);
							}
#endif// end of with VLOW
#endif// end of without VUCD

#if !defined(CondFreeVLOW) && !defined(CondFreeVUCD)
#ifdef CondFree
							//if there is a free surface condition
							//the Dz in top surface must be accquired by Dx and Dy
							if(idz == ipam[8]+LenFD-1)//surface layer
							{
						pd.DzVx[Gindex] = CoVx[xiaoI+0]*pd.DxVx[Gindex] + CoVx[xiaoI+1]*pd.DxVy[Gindex] + CoVx[xiaoI+2]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+0]*pd.DyVx[Gindex] + CoVy[xiaoI+1]*pd.DyVy[Gindex] + CoVy[xiaoI+2]*pd.DyVz[Gindex];
						pd.DzVy[Gindex] = CoVx[xiaoI+3]*pd.DxVx[Gindex] + CoVx[xiaoI+4]*pd.DxVy[Gindex] + CoVx[xiaoI+5]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+3]*pd.DyVx[Gindex] + CoVy[xiaoI+4]*pd.DyVy[Gindex] + CoVy[xiaoI+5]*pd.DyVz[Gindex];
						pd.DzVz[Gindex] = CoVx[xiaoI+6]*pd.DxVx[Gindex] + CoVx[xiaoI+7]*pd.DxVy[Gindex] + CoVx[xiaoI+8]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+6]*pd.DyVx[Gindex] + CoVy[xiaoI+7]*pd.DyVy[Gindex] + CoVy[xiaoI+8]*pd.DyVz[Gindex];
							}
							else
							{
								pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);
							}
#endif//end of define CondFree(except vlow and vucd)
#endif//end of doesnot define VLOW and VUCD


						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void CalWave(int ConIndex, derivF drv, mdparF mpa, PartialD pd, apara apr, Real *CoVx, Real *CoVy, wfield hW,
			  wfield Ax, wfield hAx, wfield Ay, wfield hAy, wfield Az, wfield hAz)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
#ifdef HYindex	
	int Hyindex;
#endif	

	Real lambda,miu,rho,lam2mu;
	Real xix,xiy,xiz, etx,ety,etz, ztx,zty,ztz;//covariants
	
	Real DxiVx,DetVx,DztVx, DxiVy,DetVy,DztVy, DxiVz,DetVz,DztVz;
	Real DxiTxx,DetTxx,DztTxx, DxiTyy,DetTyy,DztTyy, DxiTzz,DetTzz,DztTzz;
	Real DxiTxy,DetTxy,DztTxy, DxiTxz,DetTxz,DztTxz, DxiTyz,DetTyz,DztTyz;
	Real Bx,By,Bz;//absorb boundary pars

#ifdef CFSPML
	Real APDx,APDy,APDz, DBx,DBy,DBz;
	int Pidx,tempIdx;
#ifdef CondFree	
	Real DzVx1,DzVx2, DzVy1,DzVy2, DzVz1,DzVz2;
	int xiaoI;
#endif	
#endif



	//the time-domain derivative is get by two equations, the momentum equation and the genaralized hooke's equation, 
	//which is Equation 2.20 and 2.21 respectively. And those two equation will also apply to the TIMG and VUCD free 
	//surface conditions.

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current device compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						//if(idz<ipam[8]+LenFD && idz>=ConIndex)//contain convers interface
						if(idz<ipam[8]+LenFD)//contain convers interface
						{

							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							
							rho = mpa.rho[Gindex];
							miu = rho*mpa.beta[Gindex]*mpa.beta[Gindex];
							lam2mu = rho*mpa.alpha[Gindex]*mpa.alpha[Gindex];
							lambda = lam2mu - 2.0*miu;
							rho = 1.0/rho;

							xix = drv.xix[Gindex];
							ety = drv.etay[Gindex];
							ztz = drv.zetaz[Gindex];
							
							if(idz>=ConIndex)
							{
								xiy = drv.xiy[Gindex];
								xiz = drv.xiz[Gindex];
								etx = drv.etax[Gindex];
								etz = drv.etaz[Gindex];
								ztx = drv.zetax[Gindex];
								zty = drv.zetay[Gindex];
#ifdef HYindex
								Hyindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+LenFD-ConIndex) 
									+ idy*(ipam[8]+LenFD-ConIndex) + idz-ConIndex;
								//hVx
								DxiVx = (xix*pd.DxTxx[Gindex] + xiy*pd.DxTxy[Gindex] + xiz*pd.DxTxz[Gindex])*rho;
								DetVx = (etx*pd.DyTxx[Hyindex] + ety*pd.DyTxy[Gindex] + etz*pd.DyTxz[Hyindex])*rho;//HYGRID
								DztVx = (ztx*pd.DzTxx[Hyindex] + zty*pd.DzTxy[Hyindex] + ztz*pd.DzTxz[Gindex])*rho;
								
								//hVy
								DxiVy = (xix*pd.DxTxy[Gindex] + xiy*pd.DxTyy[Hyindex] + xiz*pd.DxTyz[Hyindex])*rho;//HYGRID
								DetVy = (etx*pd.DyTxy[Gindex] + ety*pd.DyTyy[Gindex] + etz*pd.DyTyz[Gindex])*rho;
								DztVy = (ztx*pd.DzTxy[Hyindex] + zty*pd.DzTyy[Hyindex] + ztz*pd.DzTyz[Gindex])*rho;
								
								//hVz
								DxiVz = (xix*pd.DxTxz[Gindex] + xiy*pd.DxTyz[Hyindex] + xiz*pd.DxTzz[Hyindex])*rho;//HYGRID
								DetVz = (etx*pd.DyTxz[Hyindex] + ety*pd.DyTyz[Gindex] + etz*pd.DyTzz[Hyindex])*rho;//HYGRID
								DztVz = (ztx*pd.DzTxz[Gindex] + zty*pd.DzTyz[Gindex] + ztz*pd.DzTzz[Gindex])*rho;
#else								
								//hVx
								DxiVx = (xix*pd.DxTxx[Gindex] + xiy*pd.DxTxy[Gindex] + xiz*pd.DxTxz[Gindex])*rho;
								DetVx = (etx*pd.DyTxx[Gindex] + ety*pd.DyTxy[Gindex] + etz*pd.DyTxz[Gindex])*rho;
								DztVx = (ztx*pd.DzTxx[Gindex] + zty*pd.DzTxy[Gindex] + ztz*pd.DzTxz[Gindex])*rho;
								
								//hVy
								DxiVy = (xix*pd.DxTxy[Gindex] + xiy*pd.DxTyy[Gindex] + xiz*pd.DxTyz[Gindex])*rho;
								DetVy = (etx*pd.DyTxy[Gindex] + ety*pd.DyTyy[Gindex] + etz*pd.DyTyz[Gindex])*rho;
								DztVy = (ztx*pd.DzTxy[Gindex] + zty*pd.DzTyy[Gindex] + ztz*pd.DzTyz[Gindex])*rho;
								
								//hVz
								DxiVz = (xix*pd.DxTxz[Gindex] + xiy*pd.DxTyz[Gindex] + xiz*pd.DxTzz[Gindex])*rho;
								DetVz = (etx*pd.DyTxz[Gindex] + ety*pd.DyTyz[Gindex] + etz*pd.DyTzz[Gindex])*rho;
								DztVz = (ztx*pd.DzTxz[Gindex] + zty*pd.DzTyz[Gindex] + ztz*pd.DzTzz[Gindex])*rho;
#endif


								//hTxx
								DxiTxx = lam2mu*xix*pd.DxVx[Gindex] + lambda*xiy*pd.DxVy[Gindex] + lambda*xiz*pd.DxVz[Gindex];
								DetTxx = lam2mu*etx*pd.DyVx[Gindex] + lambda*ety*pd.DyVy[Gindex] + lambda*etz*pd.DyVz[Gindex];
								DztTxx = lam2mu*ztx*pd.DzVx[Gindex] + lambda*zty*pd.DzVy[Gindex] + lambda*ztz*pd.DzVz[Gindex];

								//hTyy
								DxiTyy = lambda*xix*pd.DxVx[Gindex] + lam2mu*xiy*pd.DxVy[Gindex] + lambda*xiz*pd.DxVz[Gindex];
								DetTyy = lambda*etx*pd.DyVx[Gindex] + lam2mu*ety*pd.DyVy[Gindex] + lambda*etz*pd.DyVz[Gindex];
								DztTyy = lambda*ztx*pd.DzVx[Gindex] + lam2mu*zty*pd.DzVy[Gindex] + lambda*ztz*pd.DzVz[Gindex];

								//hTzz
								DxiTzz = lambda*xix*pd.DxVx[Gindex] + lambda*xiy*pd.DxVy[Gindex] + lam2mu*xiz*pd.DxVz[Gindex];
								DetTzz = lambda*etx*pd.DyVx[Gindex] + lambda*ety*pd.DyVy[Gindex] + lam2mu*etz*pd.DyVz[Gindex];
								DztTzz = lambda*ztx*pd.DzVx[Gindex] + lambda*zty*pd.DzVy[Gindex] + lam2mu*ztz*pd.DzVz[Gindex];

								//hTxy
								DxiTxy = (xiy*pd.DxVx[Gindex] + xix*pd.DxVy[Gindex])*miu;
								DetTxy = (ety*pd.DyVx[Gindex] + etx*pd.DyVy[Gindex])*miu;
								DztTxy = (zty*pd.DzVx[Gindex] + ztx*pd.DzVy[Gindex])*miu;

								//hTxz
								DxiTxz = (xiz*pd.DxVx[Gindex] + xix*pd.DxVz[Gindex])*miu;
								DetTxz = (etz*pd.DyVx[Gindex] + etx*pd.DyVz[Gindex])*miu;
								DztTxz = (ztz*pd.DzVx[Gindex] + ztx*pd.DzVz[Gindex])*miu;

								//hTyz
								DxiTyz = (xiz*pd.DxVy[Gindex] + xiy*pd.DxVz[Gindex])*miu;
								DetTyz = (etz*pd.DyVy[Gindex] + ety*pd.DyVz[Gindex])*miu;
								DztTyz = (ztz*pd.DzVy[Gindex] + zty*pd.DzVz[Gindex])*miu;
							}
							else
							{
								//hVx
								DxiVx = rho*xix*pd.DxTxx[Gindex];
								DetVx = rho*ety*pd.DyTxy[Gindex];
								DztVx = ztz*pd.DzTxz[Gindex]*rho;

								//hVy
								DxiVy = rho*xix*pd.DxTxy[Gindex];
								DetVy = rho*ety*pd.DyTyy[Gindex];
								DztVy = ztz*pd.DzTyz[Gindex]*rho;

								//hVz
								DxiVz = rho*xix*pd.DxTxz[Gindex];
								DetVz = rho*ety*pd.DyTyz[Gindex];
								DztVz = ztz*pd.DzTzz[Gindex]*rho;

								//hTxx
								DxiTxx = lam2mu*xix*pd.DxVx[Gindex];
								DetTxx = lambda*ety*pd.DyVy[Gindex];
								DztTxx = lambda*ztz*pd.DzVz[Gindex];

								//hTyy
								DxiTyy = lambda*xix*pd.DxVx[Gindex];
								DetTyy = lam2mu*ety*pd.DyVy[Gindex];
								DztTyy = lambda*ztz*pd.DzVz[Gindex];

								//hTzz
								DxiTzz = lambda*xix*pd.DxVx[Gindex];
								DetTzz = lambda*ety*pd.DyVy[Gindex];
								DztTzz = lam2mu*ztz*pd.DzVz[Gindex];

								//hTxy
								DxiTxy = miu*xix*pd.DxVy[Gindex];
								DetTxy = miu*ety*pd.DyVx[Gindex];
								DztTxy = 0.0;

								//hTxz
								DxiTxz = miu*xix*pd.DxVz[Gindex];
								DetTxz = 0.0;
								DztTxz = ztz*pd.DzVx[Gindex]*miu;

								//hTyz
								DxiTyz = 0.0;
								DetTyz = miu*ety*pd.DyVz[Gindex];
								DztTyz = ztz*pd.DzVy[Gindex]*miu;
							}


#ifdef CFSPML
	APDx = apr.APDx[idx];	APDy = apr.APDy[idy];	APDz = apr.APDz[idz];
	Bx = apr.Bx[idx];	By = apr.By[idy];	Bz = apr.Bz[idz];
	DBx = apr.DBx[idx];	DBy = apr.DBy[idy];	DBz = apr.DBz[idz];
#else	
	Bx = 1.0;	By = 1.0;	Bz = 1.0;
#endif

							//time domain partial derivative--->wave field
							hW.Txx[Gindex] = DxiTxx/Bx + DetTxx/By + DztTxx/Bz;
							hW.Tyy[Gindex] = DxiTyy/Bx + DetTyy/By + DztTyy/Bz;
							hW.Tzz[Gindex] = DxiTzz/Bx + DetTzz/By + DztTzz/Bz;
							hW.Txy[Gindex] = DxiTxy/Bx + DetTxy/By + DztTxy/Bz;
							hW.Txz[Gindex] = DxiTxz/Bx + DetTxz/By + DztTxz/Bz;
							hW.Tyz[Gindex] = DxiTyz/Bx + DetTyz/By + DztTyz/Bz;
							hW.Vx[Gindex] = DxiVx/Bx + DetVx/By + DztVx/Bz;
							hW.Vy[Gindex] = DxiVy/Bx + DetVy/By + DztVy/Bz;
							hW.Vz[Gindex] = DxiVz/Bx + DetVz/By + DztVz/Bz;

#ifdef DisBug
//if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
//	printf("at PCS[%d]DEV[%d](%d,%d,%d),CalWave->hW.Txx=%e, DxiTxx=%e, DetTxx=%e, DztTxx=%e, Bx=%e,By=%e,Bz=%e\n",
//		ipam[2],ipam[1], zbx,zby,zbz, hW.Txx[Gindex], DxiTxx,DetTxx,DztTxx, Bx,By,Bz);
#endif

#ifdef CFSPML
#ifdef CondFree
							//top surface partial derivative conversion
							xiaoI=idx*(ipam[5]-ipam[4]+1+2*LenFD)*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;//valid Y
							
							if(idz == ipam[8]+LenFD-1)//surface layer
							{
						DzVx1 = CoVx[xiaoI+0]*pd.DxVx[Gindex] + CoVx[xiaoI+1]*pd.DxVy[Gindex] + CoVx[xiaoI+2]*pd.DxVz[Gindex];
						DzVx2 = CoVy[xiaoI+0]*pd.DyVx[Gindex] + CoVy[xiaoI+1]*pd.DyVy[Gindex] + CoVy[xiaoI+2]*pd.DyVz[Gindex];
						DzVy1 = CoVx[xiaoI+3]*pd.DxVx[Gindex] + CoVx[xiaoI+4]*pd.DxVy[Gindex] + CoVx[xiaoI+5]*pd.DxVz[Gindex];
						DzVy2 = CoVy[xiaoI+3]*pd.DyVx[Gindex] + CoVy[xiaoI+4]*pd.DyVy[Gindex] + CoVy[xiaoI+5]*pd.DyVz[Gindex];
						DzVz1 = CoVx[xiaoI+6]*pd.DxVx[Gindex] + CoVx[xiaoI+7]*pd.DxVy[Gindex] + CoVx[xiaoI+8]*pd.DxVz[Gindex];
						DzVz2 = CoVy[xiaoI+6]*pd.DyVx[Gindex] + CoVy[xiaoI+7]*pd.DyVy[Gindex] + CoVy[xiaoI+8]*pd.DyVz[Gindex];
							}
#endif							
							tempIdx = idx+(ipam[2]-LenFD)+ipam[9];//idx+ipam[9]
							if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])//X-dir
							{
						tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Ax.Vx[Pidx]/Bx;  
						hW.Vy[Gindex]  -= Ax.Vy[Pidx]/Bx;  
						hW.Vz[Gindex]  -= Ax.Vz[Pidx]/Bx;
						hW.Txx[Gindex] -= Ax.Txx[Pidx]/Bx;
						hW.Tyy[Gindex] -= Ax.Tyy[Pidx]/Bx;
						hW.Tzz[Gindex] -= Ax.Tzz[Pidx]/Bx;
						hW.Txy[Gindex] -= Ax.Txy[Pidx]/Bx;
						hW.Txz[Gindex] -= Ax.Txz[Pidx]/Bx;
						hW.Tyz[Gindex] -= Ax.Tyz[Pidx]/Bx;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAx.Vx[Pidx]  = DxiVx*DBx - APDx*Ax.Vx[Pidx];
						hAx.Vy[Pidx]  = DxiVy*DBx - APDx*Ax.Vy[Pidx];
						hAx.Vz[Pidx]  = DxiVz*DBx - APDx*Ax.Vz[Pidx];
						hAx.Txx[Pidx] = DxiTxx*DBx - APDx*Ax.Txx[Pidx];
						hAx.Tyy[Pidx] = DxiTyy*DBx - APDx*Ax.Tyy[Pidx];
						hAx.Tzz[Pidx] = DxiTzz*DBx - APDx*Ax.Tzz[Pidx];
						hAx.Txy[Pidx] = DxiTxy*DBx - APDx*Ax.Txy[Pidx];
						hAx.Txz[Pidx] = DxiTxz*DBx - APDx*Ax.Txz[Pidx];
						hAx.Tyz[Pidx] = DxiTyz*DBx - APDx*Ax.Tyz[Pidx];

#ifdef CondFree
						//top surface 
								if(idz == ipam[8]+LenFD-1)
								{
							hAx.Txx[Pidx] += DBx*Bx*( lam2mu*ztx*DzVx1 + lambda*zty*DzVy1 + lambda*ztz*DzVz1);
							hAx.Tyy[Pidx] += DBx*Bx*( lambda*ztx*DzVx1 + lam2mu*zty*DzVy1 + lambda*ztz*DzVz1);
							hAx.Tzz[Pidx] += DBx*Bx*( lambda*ztx*DzVx1 + lambda*zty*DzVy1 + lam2mu*ztz*DzVz1);
							hAx.Txy[Pidx] += DBx*Bx*( zty*DzVx1 + ztx*DzVy1 )*miu; 
							hAx.Txz[Pidx] += DBx*Bx*( ztz*DzVx1 + ztx*DzVz1 )*miu; 
							hAx.Tyz[Pidx] += DBx*Bx*( ztz*DzVy1 + zty*DzVz1 )*miu; 
								}
#endif						

							}
							
							tempIdx = idy + (ipam[4]-LenFD);//idy
							if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])//Y-dir
							{
						tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];		
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;		
						
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Ay.Vx[Pidx]/By;  
						hW.Vy[Gindex]  -= Ay.Vy[Pidx]/By;  
						hW.Vz[Gindex]  -= Ay.Vz[Pidx]/By;
						hW.Txx[Gindex] -= Ay.Txx[Pidx]/By;
						hW.Tyy[Gindex] -= Ay.Tyy[Pidx]/By;
						hW.Tzz[Gindex] -= Ay.Tzz[Pidx]/By;
						hW.Txy[Gindex] -= Ay.Txy[Pidx]/By;
						hW.Txz[Gindex] -= Ay.Txz[Pidx]/By;
						hW.Tyz[Gindex] -= Ay.Tyz[Pidx]/By;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAy.Vx[Pidx]  = DetVx*DBy - APDy*Ay.Vx[Pidx];
						hAy.Vy[Pidx]  = DetVy*DBy - APDy*Ay.Vy[Pidx];
						hAy.Vz[Pidx]  = DetVz*DBy - APDy*Ay.Vz[Pidx];
						hAy.Txx[Pidx] = DetTxx*DBy - APDy*Ay.Txx[Pidx];
						hAy.Tyy[Pidx] = DetTyy*DBy - APDy*Ay.Tyy[Pidx];
						hAy.Tzz[Pidx] = DetTzz*DBy - APDy*Ay.Tzz[Pidx];
						hAy.Txy[Pidx] = DetTxy*DBy - APDy*Ay.Txy[Pidx];
						hAy.Txz[Pidx] = DetTxz*DBy - APDy*Ay.Txz[Pidx];
						hAy.Tyz[Pidx] = DetTyz*DBy - APDy*Ay.Tyz[Pidx];

#ifdef CondFree
						//top surface 
								if(idz == ipam[8]+LenFD-1)
								{
							hAy.Txx[Pidx] += DBy*By*( lam2mu*ztx*DzVx2 + lambda*zty*DzVy2 + lambda*ztz*DzVz2);
							hAy.Tyy[Pidx] += DBy*By*( lambda*ztx*DzVx2 + lam2mu*zty*DzVy2 + lambda*ztz*DzVz2);
							hAy.Tzz[Pidx] += DBy*By*( lambda*ztx*DzVx2 + lambda*zty*DzVy2 + lam2mu*ztz*DzVz2);
							hAy.Txy[Pidx] += DBy*By*( zty*DzVx2 + ztx*DzVy2 )*miu; 
							hAy.Txz[Pidx] += DBy*By*( ztz*DzVx2 + ztx*DzVz2 )*miu; 
							hAy.Tyz[Pidx] += DBy*By*( ztz*DzVy2 + zty*DzVz2 )*miu; 
								}
#endif						
							
							}


							if(idz<=apr.nabs[4]+LenFD-1 || idz>=ipam[8]+LenFD-apr.nabs[5])//Z-dir
							{
						idz<apr.nabs[4]+LenFD ? Pidx=idz-LenFD : Pidx=idz-(ipam[8]+LenFD-apr.nabs[5])+apr.nabs[4];		
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) + idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;		
							
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Az.Vx[Pidx]/Bz;  
						hW.Vy[Gindex]  -= Az.Vy[Pidx]/Bz;  
						hW.Vz[Gindex]  -= Az.Vz[Pidx]/Bz;
						hW.Txx[Gindex] -= Az.Txx[Pidx]/Bz;
						hW.Tyy[Gindex] -= Az.Tyy[Pidx]/Bz;
						hW.Tzz[Gindex] -= Az.Tzz[Pidx]/Bz;
						hW.Txy[Gindex] -= Az.Txy[Pidx]/Bz;
						hW.Txz[Gindex] -= Az.Txz[Pidx]/Bz;
						hW.Tyz[Gindex] -= Az.Tyz[Pidx]/Bz;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAz.Vx[Pidx]  = DztVx*DBz - APDz*Az.Vx[Pidx];
						hAz.Vy[Pidx]  = DztVy*DBz - APDz*Az.Vy[Pidx];
						hAz.Vz[Pidx]  = DztVz*DBz - APDz*Az.Vz[Pidx];
						hAz.Txx[Pidx] = DztTxx*DBz - APDz*Az.Txx[Pidx];
						hAz.Tyy[Pidx] = DztTyy*DBz - APDz*Az.Tyy[Pidx];
						hAz.Tzz[Pidx] = DztTzz*DBz - APDz*Az.Tzz[Pidx];
						hAz.Txy[Pidx] = DztTxy*DBz - APDz*Az.Txy[Pidx];
						hAz.Txz[Pidx] = DztTxz*DBz - APDz*Az.Txz[Pidx];
						hAz.Tyz[Pidx] = DztTyz*DBz - APDz*Az.Tyz[Pidx];
							
							}

#endif

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void CalDiffCL(int Xvec, int Yvec, int Zvec, int ConIndex, Real steph, Real *CoVx, Real* CoVy, wfield W, PartialD pd)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int xiaoI;
#ifdef HYindex	
	int Hyindex;
#endif

	Real xstep, ystep, zstep;
	int xinc, yinc, zinc;
	
	xstep = steph*Xvec;
	ystep = steph*Yvec;
	zstep = steph*Zvec;
	xinc = Xvec*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD);//skip cdx.ny*cdx.nz
	yinc = Yvec*(ipam[8]+2*LenFD);//skip cdx.nz
	zinc = Zvec*1;//skip 1


	//generally use DRP/opt MacCormack scheme to get derivative, as Equation 2.23 and coefficients is Equation 2.24 in Thesis.
	//for the top layer transfrom the derivative of xi and eta to get zeta direction derivative, as Equation 3.4 in Thesis.
	
	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current device compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD && idz>=ConIndex)//vaild point with one virtual bounds
						{

							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							xiaoI = idx*(ipam[5]-ipam[4]+1+2*LenFD)*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;//valid Y

#ifdef DisBug
/*
if( idx+(ipam[2]-LenFD)+ipam[9] >=96  && idx+(ipam[2]-LenFD)+ipam[9] <=100&& zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf(" -->W.Vy(%d,%d,%d)=%e\n",idx+(ipam[2]-LenFD)+ipam[9],zby,zbz,W.Vy[Gindex]);
}
*/
if( idx+(ipam[2]-LenFD)+ipam[9] ==zbx && zby == idy+(ipam[4]-LenFD) && idz>=228 && idz<=233)
{
	printf(" -->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,W.Txx[Gindex],W.Tyy[Gindex],W.Tzz[Gindex],W.Txy[Gindex],W.Txz[Gindex],W.Tyz[Gindex],W.Vx[Gindex],W.Vy[Gindex],W.Vz[Gindex]);
}

#endif

#ifdef HYindex
							//with Hyindex
							Hyindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+LenFD-ConIndex) 
								+ idy*(ipam[8]+LenFD-ConIndex) + idz-ConIndex;
							
							pd.DxTyy[Hyindex] = DRPFD( W.Tyy, Gindex, xstep, xinc);
							pd.DxTzz[Hyindex] = DRPFD( W.Tzz, Gindex, xstep, xinc);
						        pd.DxTyz[Hyindex] = DRPFD( W.Tyz, Gindex, xstep, xinc);
							pd.DyTxx[Hyindex] = DRPFD( W.Txx, Gindex, ystep, yinc);
							pd.DyTzz[Hyindex] = DRPFD( W.Tzz, Gindex, ystep, yinc);
							pd.DyTxz[Hyindex] = DRPFD( W.Txz, Gindex, ystep, yinc);
							pd.DzTxx[Hyindex] = DRPFD( W.Txx, Gindex, zstep, zinc);
							pd.DzTyy[Hyindex] = DRPFD( W.Tyy, Gindex, zstep, zinc);
							pd.DzTxy[Hyindex] = DRPFD( W.Txy, Gindex, zstep, zinc);
#else
							//with Gindex
							pd.DxTyy[Gindex] = DRPFD( W.Tyy, Gindex, xstep, xinc);
							pd.DxTzz[Gindex] = DRPFD( W.Tzz, Gindex, xstep, xinc);
							pd.DxTyz[Gindex] = DRPFD( W.Tyz, Gindex, xstep, xinc);
							pd.DyTxx[Gindex] = DRPFD( W.Txx, Gindex, ystep, yinc);
							pd.DyTzz[Gindex] = DRPFD( W.Tzz, Gindex, ystep, yinc);
							pd.DyTxz[Gindex] = DRPFD( W.Txz, Gindex, ystep, yinc);
							pd.DzTxx[Gindex] = DRPFD( W.Txx, Gindex, zstep, zinc);
							pd.DzTyy[Gindex] = DRPFD( W.Tyy, Gindex, zstep, zinc);
							pd.DzTxy[Gindex] = DRPFD( W.Txy, Gindex, zstep, zinc);
#endif

							pd.DxTxx[Gindex] = DRPFD( W.Txx, Gindex, xstep, xinc);
							pd.DxTxy[Gindex] = DRPFD( W.Txy, Gindex, xstep, xinc);
							pd.DxTxz[Gindex] = DRPFD( W.Txz, Gindex, xstep, xinc);
							pd.DxVx[Gindex] = DRPFD( W.Vx, Gindex, xstep, xinc);
							pd.DxVy[Gindex] = DRPFD( W.Vy, Gindex, xstep, xinc);
							pd.DxVz[Gindex] = DRPFD( W.Vz, Gindex, xstep, xinc);

							pd.DyTyy[Gindex] = DRPFD( W.Tyy, Gindex, ystep, yinc);
							pd.DyTxy[Gindex] = DRPFD( W.Txy, Gindex, ystep, yinc);
							pd.DyTyz[Gindex] = DRPFD( W.Tyz, Gindex, ystep, yinc);
							pd.DyVx[Gindex] = DRPFD( W.Vx, Gindex, ystep, yinc);
							pd.DyVy[Gindex] = DRPFD( W.Vy, Gindex, ystep, yinc);
							pd.DyVz[Gindex] = DRPFD( W.Vz, Gindex, ystep, yinc);

							pd.DzTzz[Gindex] = DRPFD( W.Tzz, Gindex, zstep, zinc);
							pd.DzTxz[Gindex] = DRPFD( W.Txz, Gindex, zstep, zinc);
							pd.DzTyz[Gindex] = DRPFD( W.Tyz, Gindex, zstep, zinc);
#ifndef CondFree //no free surface == full space == should apply ABS							
							pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);
							pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);
							pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);
#endif

#ifdef DisBug
/*
if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf("\tat PCS[%d]DEV[%d](%d,%d,%d),CalDiff--->DzTxx=%e, DzTyy=%e, DzTzz=%e, DzTxy=%e, DzTxz=%e, DzTyz=%e\n",
		ipam[2],ipam[1], zbx,zby,zbz, pd.DzTxx[Hyindex], pd.DzTyy[Hyindex], pd.DzTzz[Gindex],pd.DzTxy[Hyindex],pd.DzTxz[Gindex],pd.DzTyz[Gindex]);
	printf("\tat PCS[%d]DEV[%d](%d,%d,%d),CalDiff--->DxVx=%e, DxVy=%e, DxVz=%e,  Vx=%e, Vy=%e, Vz=%e\n",
		ipam[2],ipam[1], zbx,zby,zbz, pd.DxVx[Gindex], pd.DxVy[Gindex], pd.DxVz[Gindex],W.Vx[Gindex],W.Vy[Gindex],W.Vz[Gindex]);
}
*/
#endif

							//  P(V3)/P(zeta), for VLOW, should deal with 3 top layer and other layer, totally 4 cases.
							//		   for VUCD, should deal with 3 top layer and other layer, totally 4 cases.
							//		   for Default, only deal with 1 top layer and other layer, totally 2 cases.
							//		   VLOW and VUCD, pick one!


#ifndef CondFreeVUCD//Velocity free surface condition---Unilateral compact difference
#ifdef CondFreeVLOW
							if(idz == ipam[8]+LenFD-1)//surface layer
							{
								//202
						pd.DzVx[Gindex] = CoVx[xiaoI+0]*pd.DxVx[Gindex] + CoVx[xiaoI+1]*pd.DxVy[Gindex] + CoVx[xiaoI+2]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+0]*pd.DyVx[Gindex] + CoVy[xiaoI+1]*pd.DyVy[Gindex] + CoVy[xiaoI+2]*pd.DyVz[Gindex];
						pd.DzVy[Gindex] = CoVx[xiaoI+3]*pd.DxVx[Gindex] + CoVx[xiaoI+4]*pd.DxVy[Gindex] + CoVx[xiaoI+5]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+3]*pd.DyVx[Gindex] + CoVy[xiaoI+4]*pd.DyVy[Gindex] + CoVy[xiaoI+5]*pd.DyVz[Gindex];
						pd.DzVz[Gindex] = CoVx[xiaoI+6]*pd.DxVx[Gindex] + CoVx[xiaoI+7]*pd.DxVy[Gindex] + CoVx[xiaoI+8]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+6]*pd.DyVx[Gindex] + CoVy[xiaoI+7]*pd.DyVy[Gindex] + CoVy[xiaoI+8]*pd.DyVz[Gindex];
							}
							else if(idz == ipam[8]+LenFD-2)//one layer inner surface
							{
								//201
								pd.DzVx[Gindex] = M22FD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = M22FD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = M22FD( W.Vz, Gindex, zstep, zinc);
							}
							else if(idz == ipam[8]+LenFD-3)//two layer inner surface
							{
								//200
								pd.DzVx[Gindex] = M24FD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = M24FD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = M24FD( W.Vz, Gindex, zstep, zinc);
							}
							else
							{
								pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);
							}
#endif// end of with VLOW
#endif// end of without VUCD

#if !defined(CondFreeVLOW) && !defined(CondFreeVUCD)
#ifdef CondFree
							//if there is a free surface condition
							//the Dz in top surface must be accquired by Dx and Dy
							if(idz == ipam[8]+LenFD-1)//surface layer
							{
						pd.DzVx[Gindex] = CoVx[xiaoI+0]*pd.DxVx[Gindex] + CoVx[xiaoI+1]*pd.DxVy[Gindex] + CoVx[xiaoI+2]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+0]*pd.DyVx[Gindex] + CoVy[xiaoI+1]*pd.DyVy[Gindex] + CoVy[xiaoI+2]*pd.DyVz[Gindex];
						pd.DzVy[Gindex] = CoVx[xiaoI+3]*pd.DxVx[Gindex] + CoVx[xiaoI+4]*pd.DxVy[Gindex] + CoVx[xiaoI+5]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+3]*pd.DyVx[Gindex] + CoVy[xiaoI+4]*pd.DyVy[Gindex] + CoVy[xiaoI+5]*pd.DyVz[Gindex];
						pd.DzVz[Gindex] = CoVx[xiaoI+6]*pd.DxVx[Gindex] + CoVx[xiaoI+7]*pd.DxVy[Gindex] + CoVx[xiaoI+8]*pd.DxVz[Gindex]
								+ CoVy[xiaoI+6]*pd.DyVx[Gindex] + CoVy[xiaoI+7]*pd.DyVy[Gindex] + CoVy[xiaoI+8]*pd.DyVz[Gindex];
							}
							else
							{
								pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);
								pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);
								pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);
							}
#endif//end of define CondFree(except vlow and vucd)
#endif//end of doesnot define VLOW and VUCD


						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}


__global__ void CalWaveCL(int ConIndex, derivF drv, mdparF mpa, PartialD pd, apara apr, Real *CoVx, Real *CoVy, wfield hW,
			  wfield Ax, wfield hAx, wfield Ay, wfield hAy, wfield Az, wfield hAz)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
#ifdef HYindex	
	int Hyindex;
#endif

	Real lambda,miu,rho,lam2mu;
	Real xix,xiy,xiz, etx,ety,etz, ztx,zty,ztz;//covariants
	
	Real DxiVx,DetVx,DztVx, DxiVy,DetVy,DztVy, DxiVz,DetVz,DztVz;
	Real DxiTxx,DetTxx,DztTxx, DxiTyy,DetTyy,DztTyy, DxiTzz,DetTzz,DztTzz;
	Real DxiTxy,DetTxy,DztTxy, DxiTxz,DetTxz,DztTxz, DxiTyz,DetTyz,DztTyz;
	Real Bx,By,Bz;//absorb boundary pars

#ifdef CFSPML
	Real APDx,APDy,APDz, DBx,DBy,DBz;
	int Pidx,tempIdx;
#ifdef CondFree	
	Real DzVx1,DzVx2, DzVy1,DzVy2, DzVz1,DzVz2;
	int xiaoI;
#endif	
#endif

	//the time-domain derivative is get by two equations, the momentum equation and the genaralized hooke's equation, 
	//which is Equation 2.20 and 2.21 respectively. And those two equation will also apply to the TIMG and VUCD free 
	//surface conditions.

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current device compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD && idz>=ConIndex)//contain convers interface
						{

							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							
							rho = mpa.rho[Gindex];
							miu = rho*mpa.beta[Gindex]*mpa.beta[Gindex];
							lam2mu = rho*mpa.alpha[Gindex]*mpa.alpha[Gindex];
							lambda = lam2mu - 2.0*miu;
							rho = 1.0/rho;

							xix = drv.xix[Gindex];
							xiy = drv.xiy[Gindex];
							xiz = drv.xiz[Gindex];
							etx = drv.etax[Gindex];
							ety = drv.etay[Gindex];
							etz = drv.etaz[Gindex];
							ztx = drv.zetax[Gindex];
							zty = drv.zetay[Gindex];
							ztz = drv.zetaz[Gindex];

							
							//useful for float, double error at 1e-13
#ifdef HYindex	
							Hyindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+LenFD-ConIndex) 
								+ idy*(ipam[8]+LenFD-ConIndex) + idz-ConIndex;
							
							//hVx
							DxiVx = (xix*pd.DxTxx[Gindex] + xiy*pd.DxTxy[Gindex] + xiz*pd.DxTxz[Gindex])*rho;
							DetVx = (etx*pd.DyTxx[Hyindex] + ety*pd.DyTxy[Gindex] + etz*pd.DyTxz[Hyindex])*rho;//HYGRID
							DztVx = (ztx*pd.DzTxx[Hyindex] + zty*pd.DzTxy[Hyindex] + ztz*pd.DzTxz[Gindex])*rho;

							//hVy
							DxiVy = (xix*pd.DxTxy[Gindex] + xiy*pd.DxTyy[Hyindex] + xiz*pd.DxTyz[Hyindex])*rho;//HYGRID
							DetVy = (etx*pd.DyTxy[Gindex] + ety*pd.DyTyy[Gindex] + etz*pd.DyTyz[Gindex])*rho;
							DztVy = (ztx*pd.DzTxy[Hyindex] + zty*pd.DzTyy[Hyindex] + ztz*pd.DzTyz[Gindex])*rho;

							//hVz
							DxiVz = (xix*pd.DxTxz[Gindex] + xiy*pd.DxTyz[Hyindex] + xiz*pd.DxTzz[Hyindex])*rho;//HYGRID
							DetVz = (etx*pd.DyTxz[Hyindex] + ety*pd.DyTyz[Gindex] + etz*pd.DyTzz[Hyindex])*rho;//HYGRID
							DztVz = (ztx*pd.DzTxz[Gindex] + zty*pd.DzTyz[Gindex] + ztz*pd.DzTzz[Gindex])*rho;
#else	
							//hVx
							DxiVx = (xix*pd.DxTxx[Gindex] + xiy*pd.DxTxy[Gindex] + xiz*pd.DxTxz[Gindex])*rho;
							DetVx = (etx*pd.DyTxx[Gindex] + ety*pd.DyTxy[Gindex] + etz*pd.DyTxz[Gindex])*rho;
							DztVx = (ztx*pd.DzTxx[Gindex] + zty*pd.DzTxy[Gindex] + ztz*pd.DzTxz[Gindex])*rho;

							//hVy
							DxiVy = (xix*pd.DxTxy[Gindex] + xiy*pd.DxTyy[Gindex] + xiz*pd.DxTyz[Gindex])*rho;
							DetVy = (etx*pd.DyTxy[Gindex] + ety*pd.DyTyy[Gindex] + etz*pd.DyTyz[Gindex])*rho;
							DztVy = (ztx*pd.DzTxy[Gindex] + zty*pd.DzTyy[Gindex] + ztz*pd.DzTyz[Gindex])*rho;

							//hVz
							DxiVz = (xix*pd.DxTxz[Gindex] + xiy*pd.DxTyz[Gindex] + xiz*pd.DxTzz[Gindex])*rho;
							DetVz = (etx*pd.DyTxz[Gindex] + ety*pd.DyTyz[Gindex] + etz*pd.DyTzz[Gindex])*rho;
							DztVz = (ztx*pd.DzTxz[Gindex] + zty*pd.DzTyz[Gindex] + ztz*pd.DzTzz[Gindex])*rho;
#endif

							//hTxx
							DxiTxx = lam2mu*xix*pd.DxVx[Gindex] + lambda*xiy*pd.DxVy[Gindex] + lambda*xiz*pd.DxVz[Gindex];
							DetTxx = lam2mu*etx*pd.DyVx[Gindex] + lambda*ety*pd.DyVy[Gindex] + lambda*etz*pd.DyVz[Gindex];
							DztTxx = lam2mu*ztx*pd.DzVx[Gindex] + lambda*zty*pd.DzVy[Gindex] + lambda*ztz*pd.DzVz[Gindex];

							//hTyy
							DxiTyy = lambda*xix*pd.DxVx[Gindex] + lam2mu*xiy*pd.DxVy[Gindex] + lambda*xiz*pd.DxVz[Gindex];
							DetTyy = lambda*etx*pd.DyVx[Gindex] + lam2mu*ety*pd.DyVy[Gindex] + lambda*etz*pd.DyVz[Gindex];
							DztTyy = lambda*ztx*pd.DzVx[Gindex] + lam2mu*zty*pd.DzVy[Gindex] + lambda*ztz*pd.DzVz[Gindex];

							//hTzz
							DxiTzz = lambda*xix*pd.DxVx[Gindex] + lambda*xiy*pd.DxVy[Gindex] + lam2mu*xiz*pd.DxVz[Gindex];
							DetTzz = lambda*etx*pd.DyVx[Gindex] + lambda*ety*pd.DyVy[Gindex] + lam2mu*etz*pd.DyVz[Gindex];
							DztTzz = lambda*ztx*pd.DzVx[Gindex] + lambda*zty*pd.DzVy[Gindex] + lam2mu*ztz*pd.DzVz[Gindex];

							//hTxy
							DxiTxy = (xiy*pd.DxVx[Gindex] + xix*pd.DxVy[Gindex])*miu;
							DetTxy = (ety*pd.DyVx[Gindex] + etx*pd.DyVy[Gindex])*miu;
							DztTxy = (zty*pd.DzVx[Gindex] + ztx*pd.DzVy[Gindex])*miu;

							//hTxz
							DxiTxz = (xiz*pd.DxVx[Gindex] + xix*pd.DxVz[Gindex])*miu;
							DetTxz = (etz*pd.DyVx[Gindex] + etx*pd.DyVz[Gindex])*miu;
							DztTxz = (ztz*pd.DzVx[Gindex] + ztx*pd.DzVz[Gindex])*miu;

							//hTyz
							DxiTyz = (xiz*pd.DxVy[Gindex] + xiy*pd.DxVz[Gindex])*miu;
							DetTyz = (etz*pd.DyVy[Gindex] + ety*pd.DyVz[Gindex])*miu;
							DztTyz = (ztz*pd.DzVy[Gindex] + zty*pd.DzVz[Gindex])*miu;

#ifdef CFSPML
	APDx = apr.APDx[idx];	APDy = apr.APDy[idy];	APDz = apr.APDz[idz];
	Bx = apr.Bx[idx];	By = apr.By[idy];	Bz = apr.Bz[idz];
	DBx = apr.DBx[idx];	DBy = apr.DBy[idy];	DBz = apr.DBz[idz];
#else	
	Bx = 1.0;	By = 1.0;	Bz = 1.0;
#endif

							//time domain partial derivative--->wave field
							hW.Txx[Gindex] = DxiTxx/Bx + DetTxx/By + DztTxx/Bz;
							hW.Tyy[Gindex] = DxiTyy/Bx + DetTyy/By + DztTyy/Bz;
							hW.Tzz[Gindex] = DxiTzz/Bx + DetTzz/By + DztTzz/Bz;
							hW.Txy[Gindex] = DxiTxy/Bx + DetTxy/By + DztTxy/Bz;
							hW.Txz[Gindex] = DxiTxz/Bx + DetTxz/By + DztTxz/Bz;
							hW.Tyz[Gindex] = DxiTyz/Bx + DetTyz/By + DztTyz/Bz;
							hW.Vx[Gindex] = DxiVx/Bx + DetVx/By + DztVx/Bz;
							hW.Vy[Gindex] = DxiVy/Bx + DetVy/By + DztVy/Bz;
							hW.Vz[Gindex] = DxiVz/Bx + DetVz/By + DztVz/Bz;

#ifdef DisBug
if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf("\tat PCS[%d]DEV[%d](%d,%d,%d):\n\t\tCalWave->hW.Txx=%e, DxiTxx=%e, DetTxx=%e, DztTxx=%e\n"
	       "\t\thW.Vy=%e, DxiVy=%e, DetVy=%e, DztVy=%e\n"
	       "\t\thW.Vx=%e, DxiVx=%e, DetVx=%e, DztVx=%e\n",
		ipam[2],ipam[1], zbx,zby,zbz, 
		hW.Txx[Gindex], DxiTxx,DetTxx,DztTxx, 
		hW.Vy[Gindex],DxiVy,DetVy,DztVy,
		hW.Vx[Gindex],DxiVx,DetVx,DztVx);
	printf("\tDxVx=%e, DxVy=%e, DxVz=%e\n"
	       "\tDzTxy=%e,DzTyy=%e,DzTyz=%e\n"
	       "\tDzTxx=%e,DzTxy=%e,DzTxz=%e\n",
		pd.DxVx[Gindex], pd.DxVy[Gindex], pd.DxVz[Gindex], 
		pd.DzTxy[Hyindex],pd.DzTyy[Hyindex],pd.DzTyz[Gindex],
		pd.DzTxx[Hyindex],pd.DzTxy[Hyindex],pd.DzTxz[Gindex]);
	printf("xix=%e, %e, %e, etx=%e, %e, %e, ztx=%e, %e, %e\n",xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz);
	printf(" calwave-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,hW.Txx[Gindex],hW.Tyy[Gindex],hW.Tzz[Gindex],hW.Txy[Gindex],hW.Txz[Gindex],hW.Tyz[Gindex],hW.Vx[Gindex],hW.Vy[Gindex],hW.Vz[Gindex]);
}
#endif

#ifdef CFSPML
#ifdef CondFree
							//top surface partial derivative conversion
							xiaoI=idx*(ipam[5]-ipam[4]+1+2*LenFD)*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;//valid Y
							
							if(idz == ipam[8]+LenFD-1)//surface layer
							{
						DzVx1 = CoVx[xiaoI+0]*pd.DxVx[Gindex] + CoVx[xiaoI+1]*pd.DxVy[Gindex] + CoVx[xiaoI+2]*pd.DxVz[Gindex];
						DzVx2 = CoVy[xiaoI+0]*pd.DyVx[Gindex] + CoVy[xiaoI+1]*pd.DyVy[Gindex] + CoVy[xiaoI+2]*pd.DyVz[Gindex];
						DzVy1 = CoVx[xiaoI+3]*pd.DxVx[Gindex] + CoVx[xiaoI+4]*pd.DxVy[Gindex] + CoVx[xiaoI+5]*pd.DxVz[Gindex];
						DzVy2 = CoVy[xiaoI+3]*pd.DyVx[Gindex] + CoVy[xiaoI+4]*pd.DyVy[Gindex] + CoVy[xiaoI+5]*pd.DyVz[Gindex];
						DzVz1 = CoVx[xiaoI+6]*pd.DxVx[Gindex] + CoVx[xiaoI+7]*pd.DxVy[Gindex] + CoVx[xiaoI+8]*pd.DxVz[Gindex];
						DzVz2 = CoVy[xiaoI+6]*pd.DyVx[Gindex] + CoVy[xiaoI+7]*pd.DyVy[Gindex] + CoVy[xiaoI+8]*pd.DyVz[Gindex];
							}
#endif							
							tempIdx = idx+(ipam[2]-LenFD)+ipam[9];//idx+ipam[9]
							if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])//X-dir
							{
						tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Ax.Vx[Pidx]/Bx;  
						hW.Vy[Gindex]  -= Ax.Vy[Pidx]/Bx;  
						hW.Vz[Gindex]  -= Ax.Vz[Pidx]/Bx;
						hW.Txx[Gindex] -= Ax.Txx[Pidx]/Bx;
						hW.Tyy[Gindex] -= Ax.Tyy[Pidx]/Bx;
						hW.Tzz[Gindex] -= Ax.Tzz[Pidx]/Bx;
						hW.Txy[Gindex] -= Ax.Txy[Pidx]/Bx;
						hW.Txz[Gindex] -= Ax.Txz[Pidx]/Bx;
						hW.Tyz[Gindex] -= Ax.Tyz[Pidx]/Bx;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAx.Vx[Pidx]  = DxiVx*DBx - APDx*Ax.Vx[Pidx];
						hAx.Vy[Pidx]  = DxiVy*DBx - APDx*Ax.Vy[Pidx];
						hAx.Vz[Pidx]  = DxiVz*DBx - APDx*Ax.Vz[Pidx];
						hAx.Txx[Pidx] = DxiTxx*DBx - APDx*Ax.Txx[Pidx];
						hAx.Tyy[Pidx] = DxiTyy*DBx - APDx*Ax.Tyy[Pidx];
						hAx.Tzz[Pidx] = DxiTzz*DBx - APDx*Ax.Tzz[Pidx];
						hAx.Txy[Pidx] = DxiTxy*DBx - APDx*Ax.Txy[Pidx];
						hAx.Txz[Pidx] = DxiTxz*DBx - APDx*Ax.Txz[Pidx];
						hAx.Tyz[Pidx] = DxiTyz*DBx - APDx*Ax.Tyz[Pidx];

#ifdef CondFree
						//top surface 
								if(idz == ipam[8]+LenFD-1)
								{
							hAx.Txx[Pidx] += DBx*Bx*( lam2mu*ztx*DzVx1 + lambda*zty*DzVy1 + lambda*ztz*DzVz1);
							hAx.Tyy[Pidx] += DBx*Bx*( lambda*ztx*DzVx1 + lam2mu*zty*DzVy1 + lambda*ztz*DzVz1);
							hAx.Tzz[Pidx] += DBx*Bx*( lambda*ztx*DzVx1 + lambda*zty*DzVy1 + lam2mu*ztz*DzVz1);
							hAx.Txy[Pidx] += DBx*Bx*( zty*DzVx1 + ztx*DzVy1 )*miu; 
							hAx.Txz[Pidx] += DBx*Bx*( ztz*DzVx1 + ztx*DzVz1 )*miu; 
							hAx.Tyz[Pidx] += DBx*Bx*( ztz*DzVy1 + zty*DzVz1 )*miu; 
								}
#endif						

							}
							
							tempIdx = idy + (ipam[4]-LenFD);//idy
							if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])//Y-dir
							{
						tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];		
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;		
						
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Ay.Vx[Pidx]/By;  
						hW.Vy[Gindex]  -= Ay.Vy[Pidx]/By;  
						hW.Vz[Gindex]  -= Ay.Vz[Pidx]/By;
						hW.Txx[Gindex] -= Ay.Txx[Pidx]/By;
						hW.Tyy[Gindex] -= Ay.Tyy[Pidx]/By;
						hW.Tzz[Gindex] -= Ay.Tzz[Pidx]/By;
						hW.Txy[Gindex] -= Ay.Txy[Pidx]/By;
						hW.Txz[Gindex] -= Ay.Txz[Pidx]/By;
						hW.Tyz[Gindex] -= Ay.Tyz[Pidx]/By;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAy.Vx[Pidx]  = DetVx*DBy - APDy*Ay.Vx[Pidx];
						hAy.Vy[Pidx]  = DetVy*DBy - APDy*Ay.Vy[Pidx];
						hAy.Vz[Pidx]  = DetVz*DBy - APDy*Ay.Vz[Pidx];
						hAy.Txx[Pidx] = DetTxx*DBy - APDy*Ay.Txx[Pidx];
						hAy.Tyy[Pidx] = DetTyy*DBy - APDy*Ay.Tyy[Pidx];
						hAy.Tzz[Pidx] = DetTzz*DBy - APDy*Ay.Tzz[Pidx];
						hAy.Txy[Pidx] = DetTxy*DBy - APDy*Ay.Txy[Pidx];
						hAy.Txz[Pidx] = DetTxz*DBy - APDy*Ay.Txz[Pidx];
						hAy.Tyz[Pidx] = DetTyz*DBy - APDy*Ay.Tyz[Pidx];

#ifdef CondFree
						//top surface 
								if(idz == ipam[8]+LenFD-1)
								{
							hAy.Txx[Pidx] += DBy*By*( lam2mu*ztx*DzVx2 + lambda*zty*DzVy2 + lambda*ztz*DzVz2);
							hAy.Tyy[Pidx] += DBy*By*( lambda*ztx*DzVx2 + lam2mu*zty*DzVy2 + lambda*ztz*DzVz2);
							hAy.Tzz[Pidx] += DBy*By*( lambda*ztx*DzVx2 + lambda*zty*DzVy2 + lam2mu*ztz*DzVz2);
							hAy.Txy[Pidx] += DBy*By*( zty*DzVx2 + ztx*DzVy2 )*miu; 
							hAy.Txz[Pidx] += DBy*By*( ztz*DzVx2 + ztx*DzVz2 )*miu; 
							hAy.Tyz[Pidx] += DBy*By*( ztz*DzVy2 + zty*DzVz2 )*miu; 
								}
#endif						
							
							}


							if(idz<=apr.nabs[4]+LenFD-1 || idz>=ipam[8]+LenFD-apr.nabs[5])//Z-dir
							{
						idz<apr.nabs[4]+LenFD ? Pidx=idz-LenFD : Pidx=idz-(ipam[8]+LenFD-apr.nabs[5])+apr.nabs[4];		
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) + idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;		
							
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Az.Vx[Pidx]/Bz;  
						hW.Vy[Gindex]  -= Az.Vy[Pidx]/Bz;  
						hW.Vz[Gindex]  -= Az.Vz[Pidx]/Bz;
						hW.Txx[Gindex] -= Az.Txx[Pidx]/Bz;
						hW.Tyy[Gindex] -= Az.Tyy[Pidx]/Bz;
						hW.Tzz[Gindex] -= Az.Tzz[Pidx]/Bz;
						hW.Txy[Gindex] -= Az.Txy[Pidx]/Bz;
						hW.Txz[Gindex] -= Az.Txz[Pidx]/Bz;
						hW.Tyz[Gindex] -= Az.Tyz[Pidx]/Bz;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAz.Vx[Pidx]  = DztVx*DBz - APDz*Az.Vx[Pidx];
						hAz.Vy[Pidx]  = DztVy*DBz - APDz*Az.Vy[Pidx];
						hAz.Vz[Pidx]  = DztVz*DBz - APDz*Az.Vz[Pidx];
						hAz.Txx[Pidx] = DztTxx*DBz - APDz*Az.Txx[Pidx];
						hAz.Tyy[Pidx] = DztTyy*DBz - APDz*Az.Tyy[Pidx];
						hAz.Tzz[Pidx] = DztTzz*DBz - APDz*Az.Tzz[Pidx];
						hAz.Txy[Pidx] = DztTxy*DBz - APDz*Az.Txy[Pidx];
						hAz.Txz[Pidx] = DztTxz*DBz - APDz*Az.Txz[Pidx];
						hAz.Tyz[Pidx] = DztTyz*DBz - APDz*Az.Tyz[Pidx];
							
							}

#endif

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void CalDiffSL(int Xvec, int Yvec, int Zvec, int ConIndex, Real steph, Real *CoVx, Real* CoVy, wfield W, PartialD pd)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int Zclamp;

	Real xstep, ystep, zstep;
	int xinc, yinc, zinc;
	
	xstep = steph*Xvec;
	ystep = steph*Yvec;
	zstep = steph*Zvec;
	xinc = Xvec*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD);//skip cdx.ny*cdx.nz
	yinc = Yvec*(ipam[8]+2*LenFD);//skip cdx.nz
	zinc = Zvec*1;//skip 1

	ConIndex > ipam[8]+LenFD ? Zclamp = ipam[8]+LenFD : Zclamp = ConIndex;

	//generally use DRP/opt MacCormack scheme to get derivative, as Equation 2.23 and coefficients is Equation 2.24 in Thesis.
	//for the top layer transfrom the derivative of xi and eta to get zeta direction derivative, as Equation 3.4 in Thesis.

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current device compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<Zclamp)//vaild point with one virtual bounds
						{

							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

							pd.DxTxx[Gindex] = DRPFD( W.Txx, Gindex, xstep, xinc);//311
							pd.DxTxy[Gindex] = DRPFD( W.Txy, Gindex, xstep, xinc);//312
							pd.DxTxz[Gindex] = DRPFD( W.Txz, Gindex, xstep, xinc);//313
							pd.DxVx[Gindex] = DRPFD( W.Vx, Gindex, xstep, xinc);//31-456
							pd.DxVy[Gindex] = DRPFD( W.Vy, Gindex, xstep, xinc);//317
							pd.DxVz[Gindex] = DRPFD( W.Vz, Gindex, xstep, xinc);//318

							pd.DyTyy[Gindex] = DRPFD( W.Tyy, Gindex, ystep, yinc);//312
							pd.DyTxy[Gindex] = DRPFD( W.Txy, Gindex, ystep, yinc);//311
							pd.DyTyz[Gindex] = DRPFD( W.Tyz, Gindex, ystep, yinc);//313
							pd.DyVx[Gindex] = DRPFD( W.Vx, Gindex, ystep, yinc);//317
							pd.DyVy[Gindex] = DRPFD( W.Vy, Gindex, ystep, yinc);//31-456
							pd.DyVz[Gindex] = DRPFD( W.Vz, Gindex, ystep, yinc);//319

							//pd.DzTxx[Gindex] = DRPFD( W.Txx, Gindex, zstep, zinc);//esp
							//pd.DzTyy[Gindex] = DRPFD( W.Tyy, Gindex, zstep, zinc);//esp
							pd.DzTzz[Gindex] = DRPFD( W.Tzz, Gindex, zstep, zinc);//313
							//pd.DzTxy[Gindex] = DRPFD( W.Txy, Gindex, zstep, zinc);//esp
							pd.DzTxz[Gindex] = DRPFD( W.Txz, Gindex, zstep, zinc);//311
							pd.DzTyz[Gindex] = DRPFD( W.Tyz, Gindex, zstep, zinc);//312
							pd.DzVx[Gindex] = DRPFD( W.Vx, Gindex, zstep, zinc);//318
							pd.DzVy[Gindex] = DRPFD( W.Vy, Gindex, zstep, zinc);//319
							pd.DzVz[Gindex] = DRPFD( W.Vz, Gindex, zstep, zinc);//31-456

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}


__global__ void CalWaveSL(int ConIndex, derivF drv, mdparF mpa, apara apr, PartialD pd, wfield hW,
			  wfield Ax, wfield hAx, wfield Ay, wfield hAy, wfield Az, wfield hAz)
{
	//int i,j,k;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int Zclamp;

	Real lambda,miu,rho,lam2mu;
	Real xix,ety,ztz;
	//Real ztx,zty;
	
	Real DxiVx,DetVx,DztVx, DxiVy,DetVy,DztVy, DxiVz,DetVz,DztVz;
	Real DxiTxx,DetTxx,DztTxx, DxiTyy,DetTyy,DztTyy, DxiTzz,DetTzz,DztTzz;
	Real DxiTxy,DetTxy,DztTxy, DxiTxz,DetTxz,DztTxz, DxiTyz,DetTyz,DztTyz;//T6V3
	Real Bx,By,Bz;//absorb boundary pars

#ifdef CFSPML
	Real APDx,APDy,APDz, DBx,DBy,DBz;
	int Pidx,tempIdx;
#endif

	ConIndex > ipam[8]+LenFD ? Zclamp = ipam[8]+LenFD : Zclamp = ConIndex;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current device compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<Zclamp)//ecept convers interface
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							
							rho = mpa.rho[Gindex];
							miu = rho*mpa.beta[Gindex]*mpa.beta[Gindex];
							lam2mu = rho*mpa.alpha[Gindex]*mpa.alpha[Gindex];
							lambda = lam2mu - 2.0*miu;
							rho = 1.0/rho;
							
							xix = drv.xix[Gindex];
							ety = drv.etay[Gindex];
							//ztx = drv.zetax[Gindex];
							//zty = drv.zetay[Gindex];
							ztz = drv.zetaz[Gindex];
							//for hy1, for hy2 should add ztx and zty
							//xiy,xiz,etax,etaz,should be 0 for straight line
							//and xix,etay,zetax,zetay,zetaz has NO rotation property, only scaling property
							//especailly in topo area Z varies in three direction, so zetax,zetay,zetaz all have scaling property

							//hVx
							DxiVx = rho*xix*pd.DxTxx[Gindex];
							DetVx = rho*ety*pd.DyTxy[Gindex];
							DztVx = rho*ztz*pd.DzTxz[Gindex];
							//DztVx = rho*(ztx*pd.DzTxx[Gindex] + zty*pd.DzTxy[Gindex] + ztz*pd.DzTxz[Gindex]);

							//hVy
							DxiVy = rho*xix*pd.DxTxy[Gindex];
							DetVy = rho*ety*pd.DyTyy[Gindex];
							DztVy = rho*ztz*pd.DzTyz[Gindex];
							//DztVy = rho*(ztx*pd.DzTxy[Gindex] + zty*pd.DzTyy[Gindex] + ztz*pd.DzTyz[Gindex]);

							//hVz
							DxiVz = rho*xix*pd.DxTxz[Gindex];
							DetVz = rho*ety*pd.DyTyz[Gindex];
							DztVz = rho*ztz*pd.DzTzz[Gindex];
							//DztVz = rho*(ztx*pd.DzTxz[Gindex] + zty*pd.DzTyz[Gindex] + ztz*pd.DzTzz[Gindex]);

							//hTxx
							DxiTxx = lam2mu*xix*pd.DxVx[Gindex];
							DetTxx = lambda*ety*pd.DyVy[Gindex];
							DztTxx = lambda*ztz*pd.DzVz[Gindex];
							//DztTxx = lam2mu*ztx*pd.DzVx[Gindex] + lambda*zty*pd.DzVy[Gindex] + lambda*ztz*pd.DzVz[Gindex];

							//hTyy
							DxiTyy = lambda*xix*pd.DxVx[Gindex];
							DetTyy = lam2mu*ety*pd.DyVy[Gindex];
							DztTyy = lambda*ztz*pd.DzVz[Gindex];
							//DztTyy = lambda*ztx*pd.DzVx[Gindex] + lam2mu*zty*pd.DzVy[Gindex] + lambda*ztz*pd.DzVz[Gindex];

							//hTzz
							DxiTzz = lambda*xix*pd.DxVx[Gindex];
							DetTzz = lambda*ety*pd.DyVy[Gindex];
							DztTzz = lam2mu*ztz*pd.DzVz[Gindex];
							//DztTzz = lambda*ztx*pd.DzVx[Gindex] + lambda*zty*pd.DzVy[Gindex] + lam2mu*ztz*pd.DzVz[Gindex];

							//hTxy
							DxiTxy = miu*xix*pd.DxVy[Gindex];
							DetTxy = miu*ety*pd.DyVx[Gindex];
							DztTxy = 0.0;
							//DztTxy = miu*(zty*pd.DzVx[Gindex] + ztx*pd.DzVy[Gindex]);

							//hTxz
							DxiTxz = miu*xix*pd.DxVz[Gindex];
							DetTxz = 0.0;
							DztTxz = miu*ztz*pd.DzVx[Gindex];
							//DztTxz = miu*(ztz*pd.DzVx[Gindex] + ztx*pd.DzVz[Gindex]);

							//hTyz
							DxiTyz = 0.0;
							DetTyz = miu*ety*pd.DyVz[Gindex];
							DztTyz = miu*ztz*pd.DzVy[Gindex];
							//DztTyz = miu*(ztz*pd.DzVy[Gindex] + zty*pd.DzVz[Gindex]);
	
#ifdef CFSPML
	APDx = apr.APDx[idx];	APDy = apr.APDy[idy];	APDz = apr.APDz[idz];
	Bx = apr.Bx[idx];	By = apr.By[idy];	Bz = apr.Bz[idz];
	DBx = apr.DBx[idx];	DBy = apr.DBy[idy];	DBz = apr.DBz[idz];
#else	
	Bx = 1.0;	By = 1.0;	Bz = 1.0;
#endif

							//time domain partial derivative--->wave field
							hW.Txx[Gindex] = DxiTxx/Bx + DetTxx/By + DztTxx/Bz;
							hW.Tyy[Gindex] = DxiTyy/Bx + DetTyy/By + DztTyy/Bz;
							hW.Tzz[Gindex] = DxiTzz/Bx + DetTzz/By + DztTzz/Bz;
							hW.Txy[Gindex] = DxiTxy/Bx + DetTxy/By + DztTxy/Bz;
							hW.Txz[Gindex] = DxiTxz/Bx + DetTxz/By + DztTxz/Bz;
							hW.Tyz[Gindex] = DxiTyz/Bx + DetTyz/By + DztTyz/Bz;
							hW.Vx[Gindex] = DxiVx/Bx + DetVx/By + DztVx/Bz;
							hW.Vy[Gindex] = DxiVy/Bx + DetVy/By + DztVy/Bz;
							hW.Vz[Gindex] = DxiVz/Bx + DetVz/By + DztVz/Bz;
							
#ifdef CFSPML
							tempIdx = idx + (ipam[2]-LenFD) + ipam[9];//idx+ipam[9]
							if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])//X-dir
							{
						tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Ax.Vx[Pidx]/Bx;  
						hW.Vy[Gindex]  -= Ax.Vy[Pidx]/Bx;  
						hW.Vz[Gindex]  -= Ax.Vz[Pidx]/Bx;
						hW.Txx[Gindex] -= Ax.Txx[Pidx]/Bx;
						hW.Tyy[Gindex] -= Ax.Tyy[Pidx]/Bx;
						hW.Tzz[Gindex] -= Ax.Tzz[Pidx]/Bx;
						hW.Txy[Gindex] -= Ax.Txy[Pidx]/Bx;
						hW.Txz[Gindex] -= Ax.Txz[Pidx]/Bx;
						hW.Tyz[Gindex] -= Ax.Tyz[Pidx]/Bx;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAx.Vx[Pidx]  = DxiVx*DBx - APDx*Ax.Vx[Pidx];
						hAx.Vy[Pidx]  = DxiVy*DBx - APDx*Ax.Vy[Pidx];
						hAx.Vz[Pidx]  = DxiVz*DBx - APDx*Ax.Vz[Pidx];
						hAx.Txx[Pidx] = DxiTxx*DBx - APDx*Ax.Txx[Pidx];
						hAx.Tyy[Pidx] = DxiTyy*DBx - APDx*Ax.Tyy[Pidx];
						hAx.Tzz[Pidx] = DxiTzz*DBx - APDx*Ax.Tzz[Pidx];
						hAx.Txy[Pidx] = DxiTxy*DBx - APDx*Ax.Txy[Pidx];
						hAx.Txz[Pidx] = DxiTxz*DBx - APDx*Ax.Txz[Pidx];
						hAx.Tyz[Pidx] = DxiTyz*DBx - APDx*Ax.Tyz[Pidx];

							}
							
							tempIdx = idy + (ipam[4]-LenFD);//idy
							if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])//Y-dir
							{
						tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];		
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;		
						
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Ay.Vx[Pidx]/By;  
						hW.Vy[Gindex]  -= Ay.Vy[Pidx]/By;  
						hW.Vz[Gindex]  -= Ay.Vz[Pidx]/By;
						hW.Txx[Gindex] -= Ay.Txx[Pidx]/By;
						hW.Tyy[Gindex] -= Ay.Tyy[Pidx]/By;
						hW.Tzz[Gindex] -= Ay.Tzz[Pidx]/By;
						hW.Txy[Gindex] -= Ay.Txy[Pidx]/By;
						hW.Txz[Gindex] -= Ay.Txz[Pidx]/By;
						hW.Tyz[Gindex] -= Ay.Tyz[Pidx]/By;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAy.Vx[Pidx]  = DetVx*DBy - APDy*Ay.Vx[Pidx];
						hAy.Vy[Pidx]  = DetVy*DBy - APDy*Ay.Vy[Pidx];
						hAy.Vz[Pidx]  = DetVz*DBy - APDy*Ay.Vz[Pidx];
						hAy.Txx[Pidx] = DetTxx*DBy - APDy*Ay.Txx[Pidx];
						hAy.Tyy[Pidx] = DetTyy*DBy - APDy*Ay.Tyy[Pidx];
						hAy.Tzz[Pidx] = DetTzz*DBy - APDy*Ay.Tzz[Pidx];
						hAy.Txy[Pidx] = DetTxy*DBy - APDy*Ay.Txy[Pidx];
						hAy.Txz[Pidx] = DetTxz*DBy - APDy*Ay.Txz[Pidx];
						hAy.Tyz[Pidx] = DetTyz*DBy - APDy*Ay.Tyz[Pidx];
							
							}


							if(idz<=apr.nabs[4]+LenFD-1 || idz>=ipam[8]+LenFD-apr.nabs[5])//Z1
							{
						idz<apr.nabs[4]+LenFD ? Pidx=idz-LenFD : Pidx=idz-(ipam[8]+LenFD-apr.nabs[5])+apr.nabs[4];		
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) + idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;		
							
						//wavefield attenuation (Equation 14 of Zhang 2010)
						hW.Vx[Gindex]  -= Az.Vx[Pidx]/Bz;  
						hW.Vy[Gindex]  -= Az.Vy[Pidx]/Bz;  
						hW.Vz[Gindex]  -= Az.Vz[Pidx]/Bz;
						hW.Txx[Gindex] -= Az.Txx[Pidx]/Bz;
						hW.Tyy[Gindex] -= Az.Tyy[Pidx]/Bz;
						hW.Tzz[Gindex] -= Az.Tzz[Pidx]/Bz;
						hW.Txy[Gindex] -= Az.Txy[Pidx]/Bz;
						hW.Txz[Gindex] -= Az.Txz[Pidx]/Bz;
						hW.Tyz[Gindex] -= Az.Tyz[Pidx]/Bz;
						
						//ADE update (Equation A10 of Zhang 2010)
						hAz.Vx[Pidx]  = DztVx*DBz - APDz*Az.Vx[Pidx];
						hAz.Vy[Pidx]  = DztVy*DBz - APDz*Az.Vy[Pidx];
						hAz.Vz[Pidx]  = DztVz*DBz - APDz*Az.Vz[Pidx];
						hAz.Txx[Pidx] = DztTxx*DBz - APDz*Az.Txx[Pidx];
						hAz.Tyy[Pidx] = DztTyy*DBz - APDz*Az.Tyy[Pidx];
						hAz.Tzz[Pidx] = DztTzz*DBz - APDz*Az.Tzz[Pidx];
						hAz.Txy[Pidx] = DztTxy*DBz - APDz*Az.Txy[Pidx];
						hAz.Txz[Pidx] = DztTxz*DBz - APDz*Az.Txz[Pidx];
						hAz.Tyz[Pidx] = DztTyz*DBz - APDz*Az.Tyz[Pidx];
							
							}

#endif
						
						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void CalTIMG(int Xvec, int Yvec, int Zvec, Real steph, Real *rho, derivF drv, wfield W, wfield hW,
			wfield Ax, wfield hAx, wfield Ay, wfield hAy, apara apr)
{
	//  <<<BPG.y, BPG.x>>>
	//gridDim.x<=cdx.nx  blockDim.x<=cdx.ny
	int i,j;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int Relidx;//relative index

	Real VecTx[LenFD*2+1], VecTy[LenFD*2+1], VecTz[LenFD*2+1];
	Real DxTx,DyTy,DzTz;
	Real Bx,By;//absorb boundary pars
	Real rhojac;
	Real xstep, ystep, zstep;
	int xinc, yinc;
	Real T3Src=0.0;//initial value

#ifdef CFSPML
	Real APDx,DBx, APDy,DBy;
	int Pidx,tempIdx;
#endif
	
	xstep = steph*Xvec;
	ystep = steph*Yvec;
	zstep = steph*Zvec;
	//none direction
	xinc = (ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD);//skip cdx.ny*cdx.nz
	yinc = ipam[8]+2*LenFD;//skip cdx.nz

	//the Traction Image method for free surface condition, use the conservative form momentum equation as Equation 3.11 in Thesis
	
	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.x)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.x + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=blockDim.x)
			{
				idy = countY + threadIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{  
					for(countZ=0; countZ<LenFD; countZ++)
					{
						idz = ipam[8] + countZ;//valid point number + LenFD = last location of valid point

						Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

						rhojac = 1.0/rho[Gindex]/drv.jac[Gindex];

#ifdef CFSPML
	APDx = apr.APDx[idx];	APDy = apr.APDy[idy];
	Bx = apr.Bx[idx];	By = apr.By[idy];
	DBx = apr.DBx[idx];	DBy = apr.DBy[idy];
#else	
	Bx = 1.0;	By = 1.0;
#endif

						//X-dir
						//vector of covariant pars multiplied directional stress
						for(i=0;i<LenFD*2+1;i++)
						{
							Relidx = Gindex - LenFD*xinc;//X-dir
							VecTx[i] = drv.jac[Relidx+i*xinc]*(drv.xix[Relidx+i*xinc]*W.Txx[Relidx+i*xinc]+
											   drv.xiy[Relidx+i*xinc]*W.Txy[Relidx+i*xinc]+
											   drv.xiz[Relidx+i*xinc]*W.Txz[Relidx+i*xinc]);

							Relidx = Gindex - LenFD*yinc;//Y-dir
							VecTy[i] = drv.jac[Relidx+i*yinc]*(drv.etax[Relidx+i*yinc]*W.Txx[Relidx+i*yinc]+
											   drv.etay[Relidx+i*yinc]*W.Txy[Relidx+i*yinc]+
											   drv.etaz[Relidx+i*yinc]*W.Txz[Relidx+i*yinc]);

							Relidx = Gindex - LenFD;//Z-dir
							VecTz[i] = drv.jac[Relidx+i]*(drv.zetax[Relidx+i]*W.Txx[Relidx+i]+
										      drv.zetay[Relidx+i]*W.Txy[Relidx+i]+
										      drv.zetaz[Relidx+i]*W.Txz[Relidx+i]);
						}

						//traction image
						for(j=1;j<=LenFD-(2-countZ);j++)
							VecTz[LenFD + (2-countZ) + j] = 2.0*T3Src - VecTz[LenFD + (2-countZ) - j];//TxSrc
						VecTz[LenFD + (2-countZ)] = T3Src;

						//partial derivative vector
						//the data has been already extracted from orignal array and put into new array point by point,
						//so it doesn't need big step skip when do differential work, and only need direction information.
						//the differential center is vector center.
						DxTx = rhojac*strF( VecTx, LenFD, xstep, Xvec);
						DyTy = rhojac*strF( VecTy, LenFD, ystep, Yvec);
						DzTz = rhojac*strF( VecTz, LenFD, zstep, Zvec);

						//time domain partial derivative--->wave field
						hW.Vx[Gindex] = DxTx/Bx + DyTy/By + DzTz;

#ifdef CFSPML
				//if apply free surface condition, in top area of Z-dir will not apply absorbtion
				//so nabs[5] equals 0, and related ADE wavefield(Az,hAz) will be none
				//absorbing pars will be default APDz=0, DBz=0, Bz=1
				//so the Z-top absorption is eliminated
						//X dir absorption
						tempIdx = idx + (ipam[2]-LenFD) + ipam[9];
						if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])
						{
						tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						hW.Vx[Gindex] -= Ax.Vx[Pidx]/Bx;
						hAx.Vx[Pidx] = DBx*DxTx - APDx*Ax.Vx[Pidx];
						}


						//Y dir absorption
						tempIdx = idy + (ipam[4]-LenFD);
						if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])
						{
						tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

						hW.Vx[Gindex] -= Ay.Vx[Pidx]/By;
						hAy.Vx[Pidx] = DBy*DyTy - APDy*Ay.Vx[Pidx];
						}


#endif
						
						//Y-dir
						for(i=0;i<LenFD*2+1;i++)
						{
							Relidx = Gindex - LenFD*xinc;
							VecTx[i] = drv.jac[Relidx+i*xinc]*(drv.xix[Relidx+i*xinc]*W.Txy[Relidx+i*xinc]+
											   drv.xiy[Relidx+i*xinc]*W.Tyy[Relidx+i*xinc]+
											   drv.xiz[Relidx+i*xinc]*W.Tyz[Relidx+i*xinc]);
							
							Relidx = Gindex - LenFD*yinc;
							VecTy[i] = drv.jac[Relidx+i*yinc]*(drv.etax[Relidx+i*yinc]*W.Txy[Relidx+i*yinc]+
											   drv.etay[Relidx+i*yinc]*W.Tyy[Relidx+i*yinc]+
											   drv.etaz[Relidx+i*yinc]*W.Tyz[Relidx+i*yinc]);
							
							Relidx = Gindex - LenFD;
							VecTz[i] = drv.jac[Relidx+i]*(drv.zetax[Relidx+i]*W.Txy[Relidx+i]+
										      drv.zetay[Relidx+i]*W.Tyy[Relidx+i]+
										      drv.zetaz[Relidx+i]*W.Tyz[Relidx+i]);
						}

						for(j=1;j<=LenFD-(2-countZ);j++)
							VecTz[LenFD + (2-countZ) + j] = 2.0*T3Src - VecTz[LenFD + (2-countZ) - j];//TySrc
						VecTz[LenFD + (2-countZ)] = T3Src;

						DxTx = rhojac*strF( VecTx, LenFD, xstep, Xvec);//LenFD means center
						DyTy = rhojac*strF( VecTy, LenFD, ystep, Yvec);
						DzTz = rhojac*strF( VecTz, LenFD, zstep, Zvec);

						hW.Vy[Gindex] = DxTx/Bx + DyTy/By + DzTz;
					
#ifdef CFSPML
						//X dir absorption
						tempIdx = idx + (ipam[2]-LenFD) + ipam[9];
						if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])
						{
						tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						hW.Vy[Gindex] -= Ax.Vy[Pidx]/Bx;
						hAx.Vy[Pidx] = DBx*DxTx - APDx*Ax.Vy[Pidx];
						}
						//Y dir absorption
						tempIdx = idy + (ipam[4]-LenFD);
						if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])
						{
						tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

						hW.Vy[Gindex] -= Ay.Vy[Pidx]/By;
						hAy.Vy[Pidx] = DBy*DyTy - APDy*Ay.Vy[Pidx];
						}

#endif
						//Z-dir
						for(i=0;i<LenFD*2+1;i++)
						{
							Relidx = Gindex - LenFD*xinc;
							VecTx[i] = drv.jac[Relidx+i*xinc]*(drv.xix[Relidx+i*xinc]*W.Txz[Relidx+i*xinc]+
											   drv.xiy[Relidx+i*xinc]*W.Tyz[Relidx+i*xinc]+
											   drv.xiz[Relidx+i*xinc]*W.Tzz[Relidx+i*xinc]);
							
							Relidx = Gindex - LenFD*yinc;
							VecTy[i] = drv.jac[Relidx+i*yinc]*(drv.etax[Relidx+i*yinc]*W.Txz[Relidx+i*yinc]+
											   drv.etay[Relidx+i*yinc]*W.Tyz[Relidx+i*yinc]+
											   drv.etaz[Relidx+i*yinc]*W.Tzz[Relidx+i*yinc]);
							
							Relidx = Gindex - LenFD;
							VecTz[i] = drv.jac[Relidx+i]*(drv.zetax[Relidx+i]*W.Txz[Relidx+i]+
										      drv.zetay[Relidx+i]*W.Tyz[Relidx+i]+
										      drv.zetaz[Relidx+i]*W.Tzz[Relidx+i]);
						}

						for(j=1;j<=LenFD-(2-countZ);j++)
							VecTz[LenFD + (2-countZ) + j] = 2.0*T3Src - VecTz[LenFD + (2-countZ) - j];//TzSrc
						VecTz[LenFD +(2-countZ)] = T3Src;

						DxTx = rhojac*strF( VecTx, LenFD, xstep, Xvec);
						DyTy = rhojac*strF( VecTy, LenFD, ystep, Yvec);
						DzTz = rhojac*strF( VecTz, LenFD, zstep, Zvec);

						hW.Vz[Gindex] = DxTx/Bx + DyTy/By + DzTz;

#ifdef CFSPML
						//X dir absorption
						tempIdx = idx+(ipam[2]-LenFD)+ipam[9];
						if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])
						{
						tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						hW.Vz[Gindex] -= Ax.Vz[Pidx]/Bx;
						hAx.Vz[Pidx] = DBx*DxTx - APDx*Ax.Vz[Pidx];
						}
						//Y dir absorption
						tempIdx = idy+(ipam[4]-LenFD);
						if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])
						{
						tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];
						Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

						hW.Vz[Gindex] -= Ay.Vz[Pidx]/By;
						hAy.Vz[Pidx] = DBy*DyTy - APDy*Ay.Vz[Pidx];
						}

#endif


					}//loop countZ(LenFD)
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void CalVUCD(int Xvec, int Yvec, int Zvec, Real steph, Real *CoVx, Real* CoVy, mdparF mpa, derivF drv, wfield W, wfield hW,
			wfield Ax, wfield hAx, wfield Ay, wfield hAy, apara apr)
{//wrong
	//  <<<BPG.y, BPG.x>>>
	//gridDim.x<=cdx.nx  blockDim.x<=cdx.ny
	int n;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int xiaoI;

	Real lambda,miu,rho,lam2mu;
	Real xix,xiy,xiz, etx,ety,etz, ztx,zty,ztz;//covariants
	Real DxVx[LenFD+1],DxVy[LenFD+1],DxVz[LenFD+1];
	Real DyVx[LenFD+1],DyVy[LenFD+1],DyVz[LenFD+1];
	Real DzVx[LenFD+1],DzVy[LenFD+1],DzVz[LenFD+1];
	Real DxiTxx,DetTxx,DztTxx, DxiTyy,DetTyy,DztTyy, DxiTzz,DetTzz,DztTzz;//vector of covariants mutiply space derivative in xi,eta and zeta direction, respectively.
	Real DxiTxy,DetTxy,DztTxy, DxiTxz,DetTxz,DztTxz, DxiTyz,DetTyz,DztTyz;//T6V3
	Real Bx,By;//absorb boundary pars
	Real xstep, ystep, zstep;
	int xinc, yinc, zinc;
	Real V3Src=0.0;//initial value
	
	xstep = steph*Xvec;
	ystep = steph*Yvec;
	zstep = steph*Zvec;
	//none direction
	xinc = Xvec*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD);//skip cdx.ny*cdx.nz
	yinc = Yvec*(ipam[8]+2*LenFD);//skip cdx.nz
	zinc = Zvec*1;

#ifdef CFSPML
	Real APDx,DBx, APDy,DBy;
	int Pidx,tempIdx;
#endif
	
	//This is computing for the velocity free surface condition by the unilateral compact MacCormack type difference scheme
	//Hixion & Turkel, 2000. And in Thesis is Equation 2.25

	//computational sequence
	//			FORTRAN			|	GPU
	//		SN	index	direction
	//  DzV3	4	nk2	F	-	|	3	nk2-1
	//		3	nk2-1	F	B	|	2	nk2-2
	//		2	nk2-2	F	B	|	1	nk2-3
	//		1	nk2-3	-	B	|	0	nk2-4


	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.x)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.x + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=blockDim.x)
			{
				idy = countY + threadIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{  
					//case 1: nk2-4 layer, needed for UCDFD
					n = 0; countZ = -4;
					idz = ipam[8] + countZ;//valid point number + LenFD = last location of valid point
					Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

					DzVx[n] = DRPFD( W.Vx, Gindex, zstep, zinc );
					DzVy[n] = DRPFD( W.Vy, Gindex, zstep, zinc );
					DzVz[n] = DRPFD( W.Vz, Gindex, zstep, zinc );


					//case 2: nk2-1 layer, top layer, needed for UCDFD
					n = 3; countZ = -1;
					idz = ipam[8] + countZ;//valid point number + LenFD = last location of valid point
					Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
					xiaoI = idx*(ipam[5]-ipam[4]+1+2*LenFD)*SeisGeo*SeisGeo + idy*SeisGeo*SeisGeo;//valid Y

					DxVx[n] = DRPFD( W.Vx, Gindex, xstep, xinc );
					DxVy[n] = DRPFD( W.Vy, Gindex, xstep, xinc );
					DxVz[n] = DRPFD( W.Vz, Gindex, xstep, xinc );

					DyVx[n] = DRPFD( W.Vx, Gindex, ystep, yinc );
					DyVy[n] = DRPFD( W.Vy, Gindex, ystep, yinc );
					DyVz[n] = DRPFD( W.Vz, Gindex, ystep, yinc );

					DzVx[n] = CoVx[xiaoI+0]*DxVx[n] + CoVx[xiaoI+1]*DxVy[n] + CoVx[xiaoI+2]*DxVz[n]
						+ CoVy[xiaoI+0]*DyVx[n] + CoVy[xiaoI+1]*DyVy[n] + CoVy[xiaoI+2]*DyVz[n];
					DzVy[n] = CoVx[xiaoI+3]*DxVx[n] + CoVx[xiaoI+4]*DxVy[n] + CoVx[xiaoI+5]*DxVz[n]
						+ CoVy[xiaoI+3]*DyVx[n] + CoVy[xiaoI+4]*DyVy[n] + CoVy[xiaoI+5]*DyVz[n];
					DzVz[n] = CoVx[xiaoI+6]*DxVx[n] + CoVx[xiaoI+7]*DxVy[n] + CoVx[xiaoI+8]*DxVz[n]
						+ CoVy[xiaoI+6]*DyVx[n] + CoVy[xiaoI+7]*DyVy[n] + CoVy[xiaoI+8]*DyVz[n];
					
					DzVx[n] = DzVx[n] + V3Src;
					DzVy[n] = DzVy[n] + V3Src;
					DzVz[n] = DzVz[n] + V3Src;
					

					//case 3: nk2-2 and nk2-3 layer, need to apply UCDFD
					for(n=1;n<=2;n++)
					{
						countZ = n-4;//-3 -2
						idz = ipam[8] + countZ;//valid point number + LenFD = last location of valid point
						Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						
						DxVx[n] = DRPFD( W.Vx, Gindex, xstep, xinc );
						DxVy[n] = DRPFD( W.Vy, Gindex, xstep, xinc );
						DxVz[n] = DRPFD( W.Vz, Gindex, xstep, xinc );

						DyVx[n] = DRPFD( W.Vx, Gindex, ystep, yinc );
						DyVy[n] = DRPFD( W.Vy, Gindex, ystep, yinc );
						DyVz[n] = DRPFD( W.Vz, Gindex, ystep, yinc );

						DzVx[n] = UCDFD_R( W.Vx, Gindex, zstep, zinc ) - UCDFD_L( DzVx, n, zinc );
						DzVy[n] = UCDFD_R( W.Vy, Gindex, zstep, zinc ) - UCDFD_L( DzVy, n, zinc );
						DzVz[n] = UCDFD_R( W.Vz, Gindex, zstep, zinc ) - UCDFD_L( DzVz, n, zinc );


					}//loop for layer

					//compute time-domain partial derivative
					for(n=1;n<=3;n++)
					{
						countZ = n-4;
						idz = ipam[8] + countZ;//valid point number + LenFD = last location of valid point
						Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

						rho = mpa.rho[Gindex];
						miu = rho*mpa.beta[Gindex]*mpa.beta[Gindex];
						lam2mu = rho*mpa.alpha[Gindex]*mpa.alpha[Gindex];
						lambda = lam2mu - 2.0*miu;

						xix = drv.xix[Gindex];
						xiy = drv.xiy[Gindex];
						xiz = drv.xiz[Gindex];
						etx = drv.etax[Gindex];
						ety = drv.etay[Gindex];
						etz = drv.etaz[Gindex];
						ztx = drv.zetax[Gindex];
						zty = drv.zetay[Gindex];
						ztz = drv.zetaz[Gindex];

#ifdef CFSPML
	APDx = apr.APDx[idx];	APDy = apr.APDy[idy];
	Bx = apr.Bx[idx];	By = apr.By[idy];
	DBx = apr.DBx[idx];	DBy = apr.DBy[idy];
#else	
	Bx = 1.0;	By = 1.0;
#endif
						
						DxiTxx = lam2mu*xix*DxVx[n] + lambda*xiy*DxVy[n] + lambda*xiz*DxVz[n];
						DetTxx = lam2mu*etx*DyVx[n] + lambda*ety*DyVy[n] + lambda*etz*DyVz[n];
						DztTxx = lam2mu*ztx*DzVx[n] + lambda*zty*DzVy[n] + lambda*ztz*DzVz[n];

						DxiTyy = lambda*xix*DxVx[n] + lam2mu*xiy*DxVy[n] + lambda*xiz*DxVz[n];
						DetTyy = lambda*etx*DyVx[n] + lam2mu*ety*DyVy[n] + lambda*etz*DyVz[n];
						DztTyy = lambda*ztx*DzVx[n] + lam2mu*zty*DzVy[n] + lambda*ztz*DzVz[n];
						
						DxiTzz = lambda*xix*DxVx[n] + lambda*xiy*DxVy[n] + lam2mu*xiz*DxVz[n];
						DetTzz = lambda*etx*DyVx[n] + lambda*ety*DyVy[n] + lam2mu*etz*DyVz[n];
						DztTzz = lambda*ztx*DzVx[n] + lambda*zty*DzVy[n] + lam2mu*ztz*DzVz[n];
						
						DxiTxy = miu*(xiy*DxVx[n] + xix*DxVy[n]);
						DetTxy = miu*(ety*DyVx[n] + etx*DyVy[n]);
						DztTxy = miu*(zty*DzVx[n] + ztx*DzVy[n]);

						DxiTxz = miu*(xiz*DxVx[n] + xix*DxVz[n]);
						DetTxz = miu*(etz*DyVx[n] + etx*DyVz[n]);
						DztTxz = miu*(ztz*DzVx[n] + ztx*DzVz[n]);
						
						DxiTyz = miu*(xiz*DxVy[n] + xiy*DxVz[n]);
						DetTyz = miu*(etz*DyVy[n] + ety*DyVz[n]);
						DztTyz = miu*(ztz*DzVy[n] + zty*DzVz[n]);
						
						hW.Txx[Gindex] = DxiTxx/Bx + DetTxx/By + DztTxx;
						hW.Tyy[Gindex] = DxiTyy/Bx + DetTyy/By + DztTyy;
						hW.Tzz[Gindex] = DxiTzz/Bx + DetTzz/By + DztTzz;
						hW.Txy[Gindex] = DxiTxy/Bx + DetTxy/By + DztTxy;
						hW.Txz[Gindex] = DxiTxz/Bx + DetTxz/By + DztTxz;
						hW.Tyz[Gindex] = DxiTyz/Bx + DetTyz/By + DztTyz;

#ifdef CFSPML
						tempIdx = idx+(ipam[2]-LenFD)+ipam[9];//idx+ipam[9]
						if(tempIdx<=apr.nabs[0]+LenFD-1 || tempIdx>=ipam[10]+LenFD-apr.nabs[1])//X-dir
						{
							tempIdx<apr.nabs[0]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[10]+LenFD-apr.nabs[1])+apr.nabs[0];
						        Pidx = Pidx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

							hW.Txx[Gindex] -= Ax.Txx[Pidx]/Bx;
							hW.Tyy[Gindex] -= Ax.Tyy[Pidx]/Bx;
							hW.Tzz[Gindex] -= Ax.Tzz[Pidx]/Bx;
							hW.Txy[Gindex] -= Ax.Txy[Pidx]/Bx;
							hW.Txz[Gindex] -= Ax.Txz[Pidx]/Bx;
							hW.Tyz[Gindex] -= Ax.Tyz[Pidx]/Bx;

							hAx.Txx[Pidx] = DxiTxx*DBx - APDx*Ax.Txx[Pidx];
							hAx.Tyy[Pidx] = DxiTyy*DBx - APDx*Ax.Tyy[Pidx];
							hAx.Tzz[Pidx] = DxiTzz*DBx - APDx*Ax.Tzz[Pidx];
							hAx.Txy[Pidx] = DxiTxy*DBx - APDx*Ax.Txy[Pidx];
							hAx.Txz[Pidx] = DxiTxz*DBx - APDx*Ax.Txz[Pidx];
							hAx.Tyz[Pidx] = DxiTyz*DBx - APDx*Ax.Tyz[Pidx];
						}

						tempIdx = idy + (ipam[4]-LenFD);//idy
						if(tempIdx<=apr.nabs[2]+LenFD-1 || tempIdx>=ipam[7]+LenFD-apr.nabs[3])//Y-dir
						{
							tempIdx<apr.nabs[2]+LenFD ? Pidx=tempIdx-LenFD : Pidx=tempIdx-(ipam[7]+LenFD-apr.nabs[3])+apr.nabs[2];
							Pidx = Pidx*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

							hW.Txx[Gindex] -= Ay.Txx[Pidx]/By;
							hW.Tyy[Gindex] -= Ay.Tyy[Pidx]/By;
							hW.Tzz[Gindex] -= Ay.Tzz[Pidx]/By;
							hW.Txy[Gindex] -= Ay.Txy[Pidx]/By;
							hW.Txz[Gindex] -= Ay.Txz[Pidx]/By;
							hW.Tyz[Gindex] -= Ay.Tyz[Pidx]/By;

							hAy.Txx[Pidx] = DetTxx*DBy - APDy*Ay.Txx[Pidx];
							hAy.Tyy[Pidx] = DetTyy*DBy - APDy*Ay.Tyy[Pidx];
							hAy.Tzz[Pidx] = DetTzz*DBy - APDy*Ay.Tzz[Pidx];
							hAy.Txy[Pidx] = DetTxy*DBy - APDy*Ay.Txy[Pidx];
							hAy.Txz[Pidx] = DetTxz*DBy - APDy*Ay.Txz[Pidx];
							hAy.Tyz[Pidx] = DetTyz*DBy - APDy*Ay.Tyz[Pidx];
						}

#endif

					}

				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void LoadForce(int Tindex, cindx cdx, Real steph, int nfrc, int nstf, forceF frc, Real *jac, Real *rho, wfield hW)
{
	int i;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int modX,modY,modZ;//modify to global index

	Real stf,A,d;
	int mid;
	for(i=0;i<nfrc;i+=gridDim.y*gridDim.x*blockDim.x)
	{
		mid = i + blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
		if( mid<nfrc && frc.locx[i]-ipam[9] >= ipam[2] && frc.locx[i]-ipam[9] <=ipam[3] && frc.locy[i] >=ipam[4] && frc.locy[i] <=ipam[5] )
		{
			modX = frc.locx[mid]-ipam[9]-(ipam[2]-LenFD);
			modY = frc.locy[mid]-(ipam[4]-LenFD);
			modZ = frc.locz[mid];
#ifdef SrcSmooth
			for(idx = modX-LenFD; idx<=modX+LenFD; idx++)
				for(idy = modY-LenFD; idy<=modY+LenFD; idy++)
					for(idz = modZ-LenFD; idz<=modZ+LenFD; idz++)
					{
						Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						if( idx>=cdx.ni1 && idx<cdx.ni2 && idy>=cdx.nj1 && idy<cdx.nj2 && idz>=cdx.nk1 && idz<cdx.nk2 )
						{
							stf = frc.stf[mid*nstf + Tindex];
							d = frc.dnorm[mid*LenNorm*LenNorm*LenNorm + (idx - (modX-LenFD))*LenNorm*LenNorm +
								(idy - (modY-LenFD))*LenNorm + (idz - (modZ-LenFD))];
							
							A = steph*steph*steph*jac[Gindex];
							A = 1/(A*rho[Gindex])*d;
							
							if(idz == ipam[8]+LenFD-1)
								A = A*2.0;

							hW.Vx[Gindex] += stf*frc.fx[mid]*A;
							hW.Vy[Gindex] += stf*frc.fy[mid]*A;
							hW.Vz[Gindex] += stf*frc.fz[mid]*A;
						}
					}
#else
			idx = modX;	idy = modY;	idz = modZ;
			Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
			if( idx>=cdx.ni1 && idx<cdx.ni2 && idy>=cdx.nj1 && idy<cdx.nj2 && idz>=cdx.nk1 && idz<cdx.nk2 )
			{
				stf = frc.stf[mid*nstf + Tindex];
				d = 1.0;

				A = steph*steph*steph*jac[Gindex];
				A = 1/(A*rho[Gindex])*d;

				if(idz == ipam[8]+LenFD-1)
					A = A*2.0;

				hW.Vx[Gindex] += stf*frc.fx[mid]*A;
				hW.Vy[Gindex] += stf*frc.fy[mid]*A;
				hW.Vz[Gindex] += stf*frc.fz[mid]*A;
			}
#endif
		}//loop modXYZ
	}//loop i

}

__global__ void LoadMoment(int Tindex, cindx cdx, Real steph, int nmnt, int nstf, momentF mnt, Real *jac, wfield hW)
{
	int i;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int modX,modY,modZ;//modify to global index
	Real stf,A;

	int mid;
	for(i=0;i<nmnt;i+=gridDim.y*gridDim.x*blockDim.x)
	{
		mid = i + blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
		if( mid<nmnt && mnt.locx[i]-ipam[9] >= ipam[2] && mnt.locx[i]-ipam[9] <=ipam[3] && mnt.locy[i] >=ipam[4] && mnt.locy[i] <=ipam[5] )
		{
			modX = mnt.locx[mid]-ipam[9]-(ipam[2]-LenFD);
			modY = mnt.locy[mid]-(ipam[4]-LenFD);
			modZ = mnt.locz[mid];
#ifdef SrcSmooth
			for(idx = modX-LenFD; idx<=modX+LenFD; idx++)
				for(idy = modY-LenFD; idy<=modY+LenFD; idy++)
					for(idz = modZ-LenFD; idz<=modZ+LenFD; idz++)
					{
			//printf("PCS[%d]DEV[%d]loc(%d,%d,%d),modx=%d, mody=%d, modzz=%d, idx=%d, idy=%d, idz=%d\n",
			//	ipam[1],ipam[0], mnt.locx[mid],mnt.locy[mid],mnt.locz[mid], modX, modY, modZ, idx,idy,idz);
						Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						if( idx>=cdx.ni1 && idx<cdx.ni2 && idy>=cdx.nj1 && idy<cdx.nj2 && idz>=cdx.nk1 && idz<cdx.nk2 )
						{
							stf = mnt.stf[mid*nstf + Tindex];
							A = mnt.dnorm[mid*LenNorm*LenNorm*LenNorm + (idx - (modX-LenFD))*LenNorm*LenNorm +
								(idy - (modY-LenFD))*LenNorm + (idz - (modZ-LenFD))];
							
							A = stf*A/(steph*steph*steph*jac[Gindex]);

							hW.Txx[Gindex] -= mnt.mxx[mid]*A;
							hW.Tyy[Gindex] -= mnt.myy[mid]*A;
							hW.Tzz[Gindex] -= mnt.mzz[mid]*A;
							hW.Txy[Gindex] -= mnt.mxy[mid]*A;
							hW.Txz[Gindex] -= mnt.mxz[mid]*A;
							hW.Tyz[Gindex] -= mnt.myz[mid]*A;
						}
					}
#else
			idx = modX;	idy = modY;	idz = modZ;
			Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
			if( idx>=cdx.ni1 && idx<cdx.ni2 && idy>=cdx.nj1 && idy<cdx.nj2 && idz>=cdx.nk1 && idz<cdx.nk2 )
			{
				stf = mnt.stf[mid*nstf + Tindex];
				A = 1.0;

				A = stf*A/(steph*steph*steph*jac[Gindex]);

				hW.Txx[Gindex] -= mnt.mxx[mid]*A;
				hW.Tyy[Gindex] -= mnt.myy[mid]*A;
				hW.Tzz[Gindex] -= mnt.mzz[mid]*A;
				hW.Txy[Gindex] -= mnt.mxy[mid]*A;
				hW.Txz[Gindex] -= mnt.mxz[mid]*A;
				hW.Tyz[Gindex] -= mnt.myz[mid]*A;
			}
#endif
		}//loop modXYZ
	}//loop i

}

__global__ void LoadRmom(cindx cdx, Real steph, int Dfpn, RmomF mnt, Real *jac, wfield hW)
{
	int i;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int modX,modY,modZ;//modify to global index

	Real A;
	int mid;

	for(i=0;i<Dfpn;i+=gridDim.y*gridDim.x*blockDim.x)
	{
		mid = i + blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
		if( mid<Dfpn && mnt.locx[i]-ipam[9] >= ipam[2] && mnt.locx[i]-ipam[9] <=ipam[3] && mnt.locy[i] >=ipam[4] && mnt.locy[i] <=ipam[5] )
		{
			modX = mnt.locx[mid]-ipam[9]-(ipam[2]-LenFD);
			modY = mnt.locy[mid]-(ipam[4]-LenFD);
			modZ = mnt.locz[mid];
#ifdef SrcSmooth
			for(idx = modX-LenFD; idx<=modX+LenFD; idx++)
				for(idy = modY-LenFD; idy<=modY+LenFD; idy++)
					for(idz = modZ-LenFD; idz<=modZ+LenFD; idz++)
					{
						Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
						if( idx>=cdx.ni1 && idx<cdx.ni2 && idy>=cdx.nj1 && idy<cdx.nj2 && idz>=cdx.nk1 && idz<cdx.nk2 )
						{
							A = mnt.dnorm[mid*LenNorm*LenNorm*LenNorm + (idx - (modX-LenFD))*LenNorm*LenNorm +
								(idy - (modY-LenFD))*LenNorm + (idz - (modZ-LenFD))];
							
							A = A/(steph*steph*steph*jac[Gindex]);

							hW.Txx[Gindex] -= mnt.mxx[mid]*A;
							hW.Tyy[Gindex] -= mnt.myy[mid]*A;
							hW.Tzz[Gindex] -= mnt.mzz[mid]*A;
							hW.Txy[Gindex] -= mnt.mxy[mid]*A;
							hW.Txz[Gindex] -= mnt.mxz[mid]*A;
							hW.Tyz[Gindex] -= mnt.myz[mid]*A;
						}
					}
#else
			idx = modX;	idy = modY;	idz = modZ;
			Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
			//printf("PCS[%d]DEV[%d]loc(%d,%d,%d),modx=%d, mody=%d, modzz=%d, idx=%d, idy=%d, idz=%d\n",
			//	ipam[1],ipam[0], mnt.locx[mid],mnt.locy[mid],mnt.locz[mid], modX, modY, modZ, idx,idy,idz);
			if( idx>=cdx.ni1 && idx<cdx.ni2 && idy>=cdx.nj1 && idy<cdx.nj2 && idz>=cdx.nk1 && idz<cdx.nk2 )
			{
				A = 1.0;
				
				A = A/(steph*steph*steph*jac[Gindex]);

				hW.Txx[Gindex] -= mnt.mxx[mid]*A;
				hW.Tyy[Gindex] -= mnt.myy[mid]*A;
				hW.Tzz[Gindex] -= mnt.mzz[mid]*A;
				hW.Txy[Gindex] -= mnt.mxy[mid]*A;
				hW.Txz[Gindex] -= mnt.mxz[mid]*A;
				hW.Tyz[Gindex] -= mnt.myz[mid]*A;
			}
#endif
		}//loop modXYZ
	}//loop i

}

__global__ void IterationBegin(Real stept, Real alpha, Real beta, wfield FW, wfield hW, wfield tW, wfield W,
		    int *nabs, wfield FAx, wfield hAx, wfield tAx, wfield Ax,	wfield FAy, wfield hAy, wfield tAy, wfield Ay,
			       wfield FAz, wfield hAz, wfield tAz, wfield Az)
{
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int Pidx;

	alpha *= stept;
	beta *= stept;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

							W.Txx[Gindex] = FW.Txx[Gindex] + alpha*hW.Txx[Gindex];
							W.Tyy[Gindex] = FW.Tyy[Gindex] + alpha*hW.Tyy[Gindex];
							W.Tzz[Gindex] = FW.Tzz[Gindex] + alpha*hW.Tzz[Gindex];
							W.Txy[Gindex] = FW.Txy[Gindex] + alpha*hW.Txy[Gindex];
							W.Txz[Gindex] = FW.Txz[Gindex] + alpha*hW.Txz[Gindex];
							W.Tyz[Gindex] = FW.Tyz[Gindex] + alpha*hW.Tyz[Gindex];
							W.Vx[Gindex] = FW.Vx[Gindex] + alpha*hW.Vx[Gindex];
							W.Vy[Gindex] = FW.Vy[Gindex] + alpha*hW.Vy[Gindex];
							W.Vz[Gindex] = FW.Vz[Gindex] + alpha*hW.Vz[Gindex];

							tW.Txx[Gindex] = FW.Txx[Gindex] + beta*hW.Txx[Gindex];
							tW.Tyy[Gindex] = FW.Tyy[Gindex] + beta*hW.Tyy[Gindex];
							tW.Tzz[Gindex] = FW.Tzz[Gindex] + beta*hW.Tzz[Gindex];
							tW.Txy[Gindex] = FW.Txy[Gindex] + beta*hW.Txy[Gindex];
							tW.Txz[Gindex] = FW.Txz[Gindex] + beta*hW.Txz[Gindex];
							tW.Tyz[Gindex] = FW.Tyz[Gindex] + beta*hW.Tyz[Gindex];
							tW.Vx[Gindex] = FW.Vx[Gindex] + beta*hW.Vx[Gindex];
							tW.Vy[Gindex] = FW.Vy[Gindex] + beta*hW.Vy[Gindex];
							tW.Vz[Gindex] = FW.Vz[Gindex] + beta*hW.Vz[Gindex];

#ifdef DisBug
if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf(" IteBeg W-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,W.Txx[Gindex],W.Tyy[Gindex],W.Tzz[Gindex],W.Txy[Gindex],W.Txz[Gindex],W.Tyz[Gindex],W.Vx[Gindex],W.Vy[Gindex],W.Vz[Gindex]);
	printf(" IteBeg tW-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,tW.Txx[Gindex],tW.Tyy[Gindex],tW.Tzz[Gindex],tW.Txy[Gindex],tW.Txz[Gindex],tW.Tyz[Gindex],tW.Vx[Gindex],tW.Vy[Gindex],tW.Vz[Gindex]);
}
#endif

#ifdef CFSPML
							//X dir absorption
							if(idx <= nabs[0] + nabs[1] + LenFD -1 )
							{
								Pidx = (idx - LenFD)*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
								
								Ax.Txx[Pidx] = FAx.Txx[Pidx] + alpha*hAx.Txx[Pidx];
								Ax.Tyy[Pidx] = FAx.Tyy[Pidx] + alpha*hAx.Tyy[Pidx];
								Ax.Tzz[Pidx] = FAx.Tzz[Pidx] + alpha*hAx.Tzz[Pidx];
								Ax.Txy[Pidx] = FAx.Txy[Pidx] + alpha*hAx.Txy[Pidx];
								Ax.Txz[Pidx] = FAx.Txz[Pidx] + alpha*hAx.Txz[Pidx];
								Ax.Tyz[Pidx] = FAx.Tyz[Pidx] + alpha*hAx.Tyz[Pidx];
								Ax.Vx[Pidx] = FAx.Vx[Pidx] + alpha*hAx.Vx[Pidx];
								Ax.Vy[Pidx] = FAx.Vy[Pidx] + alpha*hAx.Vy[Pidx];
								Ax.Vz[Pidx] = FAx.Vz[Pidx] + alpha*hAx.Vz[Pidx];

								tAx.Txx[Pidx] = FAx.Txx[Pidx] + beta*hAx.Txx[Pidx];
								tAx.Tyy[Pidx] = FAx.Tyy[Pidx] + beta*hAx.Tyy[Pidx];
								tAx.Tzz[Pidx] = FAx.Tzz[Pidx] + beta*hAx.Tzz[Pidx];
								tAx.Txy[Pidx] = FAx.Txy[Pidx] + beta*hAx.Txy[Pidx];
								tAx.Txz[Pidx] = FAx.Txz[Pidx] + beta*hAx.Txz[Pidx];
								tAx.Tyz[Pidx] = FAx.Tyz[Pidx] + beta*hAx.Tyz[Pidx];
								tAx.Vx[Pidx] = FAx.Vx[Pidx] + beta*hAx.Vx[Pidx];
								tAx.Vy[Pidx] = FAx.Vy[Pidx] + beta*hAx.Vy[Pidx];
								tAx.Vz[Pidx] = FAx.Vz[Pidx] + beta*hAx.Vz[Pidx];
							}

							//Y dir absorption
							if(idy <= nabs[2] + nabs[3] +LenFD-1)
							{
								Pidx = (idy-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

								Ay.Txx[Pidx] = FAy.Txx[Pidx] + alpha*hAy.Txx[Pidx];
								Ay.Tyy[Pidx] = FAy.Tyy[Pidx] + alpha*hAy.Tyy[Pidx];
								Ay.Tzz[Pidx] = FAy.Tzz[Pidx] + alpha*hAy.Tzz[Pidx];
								Ay.Txy[Pidx] = FAy.Txy[Pidx] + alpha*hAy.Txy[Pidx];
								Ay.Txz[Pidx] = FAy.Txz[Pidx] + alpha*hAy.Txz[Pidx];
								Ay.Tyz[Pidx] = FAy.Tyz[Pidx] + alpha*hAy.Tyz[Pidx];
								Ay.Vx[Pidx] = FAy.Vx[Pidx] + alpha*hAy.Vx[Pidx];
								Ay.Vy[Pidx] = FAy.Vy[Pidx] + alpha*hAy.Vy[Pidx];
								Ay.Vz[Pidx] = FAy.Vz[Pidx] + alpha*hAy.Vz[Pidx];

								tAy.Txx[Pidx] = FAy.Txx[Pidx] + beta*hAy.Txx[Pidx];
								tAy.Tyy[Pidx] = FAy.Tyy[Pidx] + beta*hAy.Tyy[Pidx];
								tAy.Tzz[Pidx] = FAy.Tzz[Pidx] + beta*hAy.Tzz[Pidx];
								tAy.Txy[Pidx] = FAy.Txy[Pidx] + beta*hAy.Txy[Pidx];
								tAy.Txz[Pidx] = FAy.Txz[Pidx] + beta*hAy.Txz[Pidx];
								tAy.Tyz[Pidx] = FAy.Tyz[Pidx] + beta*hAy.Tyz[Pidx];
								tAy.Vx[Pidx] = FAy.Vx[Pidx] + beta*hAy.Vx[Pidx];
								tAy.Vy[Pidx] = FAy.Vy[Pidx] + beta*hAy.Vy[Pidx];
								tAy.Vz[Pidx] = FAy.Vz[Pidx] + beta*hAy.Vz[Pidx];
							}
							
							//Z dir absorption
							if(idz <= nabs[4] + nabs[5] +LenFD-1)
							{
								Pidx = (idz-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) 
									+ idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;

								Az.Txx[Pidx] = FAz.Txx[Pidx] + alpha*hAz.Txx[Pidx];
								Az.Tyy[Pidx] = FAz.Tyy[Pidx] + alpha*hAz.Tyy[Pidx];
								Az.Tzz[Pidx] = FAz.Tzz[Pidx] + alpha*hAz.Tzz[Pidx];
								Az.Txy[Pidx] = FAz.Txy[Pidx] + alpha*hAz.Txy[Pidx];
								Az.Txz[Pidx] = FAz.Txz[Pidx] + alpha*hAz.Txz[Pidx];
								Az.Tyz[Pidx] = FAz.Tyz[Pidx] + alpha*hAz.Tyz[Pidx];
								Az.Vx[Pidx] = FAz.Vx[Pidx] + alpha*hAz.Vx[Pidx];
								Az.Vy[Pidx] = FAz.Vy[Pidx] + alpha*hAz.Vy[Pidx];
								Az.Vz[Pidx] = FAz.Vz[Pidx] + alpha*hAz.Vz[Pidx];

								tAz.Txx[Pidx] = FAz.Txx[Pidx] + beta*hAz.Txx[Pidx];
								tAz.Tyy[Pidx] = FAz.Tyy[Pidx] + beta*hAz.Tyy[Pidx];
								tAz.Tzz[Pidx] = FAz.Tzz[Pidx] + beta*hAz.Tzz[Pidx];
								tAz.Txy[Pidx] = FAz.Txy[Pidx] + beta*hAz.Txy[Pidx];
								tAz.Txz[Pidx] = FAz.Txz[Pidx] + beta*hAz.Txz[Pidx];
								tAz.Tyz[Pidx] = FAz.Tyz[Pidx] + beta*hAz.Tyz[Pidx];
								tAz.Vx[Pidx] = FAz.Vx[Pidx] + beta*hAz.Vx[Pidx];
								tAz.Vy[Pidx] = FAz.Vy[Pidx] + beta*hAz.Vy[Pidx];
								tAz.Vz[Pidx] = FAz.Vz[Pidx] + beta*hAz.Vz[Pidx];
							}
#endif

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void IterationInner(Real stept, Real alpha, Real beta, wfield FW, wfield hW, wfield tW, wfield W,
		    int *nabs, wfield FAx, wfield hAx, wfield tAx, wfield Ax,	wfield FAy, wfield hAy, wfield tAy, wfield Ay,
			       wfield FAz, wfield hAz, wfield tAz, wfield Az)
{
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int Pidx;

	alpha *= stept;
	beta *= stept;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

							W.Txx[Gindex] = FW.Txx[Gindex] + alpha*hW.Txx[Gindex];
							W.Tyy[Gindex] = FW.Tyy[Gindex] + alpha*hW.Tyy[Gindex];
							W.Tzz[Gindex] = FW.Tzz[Gindex] + alpha*hW.Tzz[Gindex];
							W.Txy[Gindex] = FW.Txy[Gindex] + alpha*hW.Txy[Gindex];
							W.Txz[Gindex] = FW.Txz[Gindex] + alpha*hW.Txz[Gindex];
							W.Tyz[Gindex] = FW.Tyz[Gindex] + alpha*hW.Tyz[Gindex];
							W.Vx[Gindex] = FW.Vx[Gindex] + alpha*hW.Vx[Gindex];
							W.Vy[Gindex] = FW.Vy[Gindex] + alpha*hW.Vy[Gindex];
							W.Vz[Gindex] = FW.Vz[Gindex] + alpha*hW.Vz[Gindex];

							tW.Txx[Gindex] = tW.Txx[Gindex] + beta*hW.Txx[Gindex];
							tW.Tyy[Gindex] = tW.Tyy[Gindex] + beta*hW.Tyy[Gindex];
							tW.Tzz[Gindex] = tW.Tzz[Gindex] + beta*hW.Tzz[Gindex];
							tW.Txy[Gindex] = tW.Txy[Gindex] + beta*hW.Txy[Gindex];
							tW.Txz[Gindex] = tW.Txz[Gindex] + beta*hW.Txz[Gindex];
							tW.Tyz[Gindex] = tW.Tyz[Gindex] + beta*hW.Tyz[Gindex];
							tW.Vx[Gindex] = tW.Vx[Gindex] + beta*hW.Vx[Gindex];
							tW.Vy[Gindex] = tW.Vy[Gindex] + beta*hW.Vy[Gindex];
							tW.Vz[Gindex] = tW.Vz[Gindex] + beta*hW.Vz[Gindex];

#ifdef DisBug
if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf(" IteInn W-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,W.Txx[Gindex],W.Tyy[Gindex],W.Tzz[Gindex],W.Txy[Gindex],W.Txz[Gindex],W.Tyz[Gindex],W.Vx[Gindex],W.Vy[Gindex],W.Vz[Gindex]);
	printf(" IteInn tW-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,tW.Txx[Gindex],tW.Tyy[Gindex],tW.Tzz[Gindex],tW.Txy[Gindex],tW.Txz[Gindex],tW.Tyz[Gindex],tW.Vx[Gindex],tW.Vy[Gindex],tW.Vz[Gindex]);
}
#endif

#ifdef CFSPML
							//X dir absorption
							if(idx <= nabs[0] + nabs[1] + LenFD -1 )
							{
								Pidx = (idx - LenFD)*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
								
								Ax.Txx[Pidx] = FAx.Txx[Pidx] + alpha*hAx.Txx[Pidx];
								Ax.Tyy[Pidx] = FAx.Tyy[Pidx] + alpha*hAx.Tyy[Pidx];
								Ax.Tzz[Pidx] = FAx.Tzz[Pidx] + alpha*hAx.Tzz[Pidx];
								Ax.Txy[Pidx] = FAx.Txy[Pidx] + alpha*hAx.Txy[Pidx];
								Ax.Txz[Pidx] = FAx.Txz[Pidx] + alpha*hAx.Txz[Pidx];
								Ax.Tyz[Pidx] = FAx.Tyz[Pidx] + alpha*hAx.Tyz[Pidx];
								Ax.Vx[Pidx] = FAx.Vx[Pidx] + alpha*hAx.Vx[Pidx];
								Ax.Vy[Pidx] = FAx.Vy[Pidx] + alpha*hAx.Vy[Pidx];
								Ax.Vz[Pidx] = FAx.Vz[Pidx] + alpha*hAx.Vz[Pidx];

								tAx.Txx[Pidx] = tAx.Txx[Pidx] + beta*hAx.Txx[Pidx];
								tAx.Tyy[Pidx] = tAx.Tyy[Pidx] + beta*hAx.Tyy[Pidx];
								tAx.Tzz[Pidx] = tAx.Tzz[Pidx] + beta*hAx.Tzz[Pidx];
								tAx.Txy[Pidx] = tAx.Txy[Pidx] + beta*hAx.Txy[Pidx];
								tAx.Txz[Pidx] = tAx.Txz[Pidx] + beta*hAx.Txz[Pidx];
								tAx.Tyz[Pidx] = tAx.Tyz[Pidx] + beta*hAx.Tyz[Pidx];
								tAx.Vx[Pidx] = tAx.Vx[Pidx] + beta*hAx.Vx[Pidx];
								tAx.Vy[Pidx] = tAx.Vy[Pidx] + beta*hAx.Vy[Pidx];
								tAx.Vz[Pidx] = tAx.Vz[Pidx] + beta*hAx.Vz[Pidx];
							}

							//Y dir absorption
							if(idy <= nabs[2] + nabs[3] +LenFD-1)
							{
								Pidx = (idy-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

								Ay.Txx[Pidx] = FAy.Txx[Pidx] + alpha*hAy.Txx[Pidx];
								Ay.Tyy[Pidx] = FAy.Tyy[Pidx] + alpha*hAy.Tyy[Pidx];
								Ay.Tzz[Pidx] = FAy.Tzz[Pidx] + alpha*hAy.Tzz[Pidx];
								Ay.Txy[Pidx] = FAy.Txy[Pidx] + alpha*hAy.Txy[Pidx];
								Ay.Txz[Pidx] = FAy.Txz[Pidx] + alpha*hAy.Txz[Pidx];
								Ay.Tyz[Pidx] = FAy.Tyz[Pidx] + alpha*hAy.Tyz[Pidx];
								Ay.Vx[Pidx] = FAy.Vx[Pidx] + alpha*hAy.Vx[Pidx];
								Ay.Vy[Pidx] = FAy.Vy[Pidx] + alpha*hAy.Vy[Pidx];
								Ay.Vz[Pidx] = FAy.Vz[Pidx] + alpha*hAy.Vz[Pidx];

								tAy.Txx[Pidx] = tAy.Txx[Pidx] + beta*hAy.Txx[Pidx];
								tAy.Tyy[Pidx] = tAy.Tyy[Pidx] + beta*hAy.Tyy[Pidx];
								tAy.Tzz[Pidx] = tAy.Tzz[Pidx] + beta*hAy.Tzz[Pidx];
								tAy.Txy[Pidx] = tAy.Txy[Pidx] + beta*hAy.Txy[Pidx];
								tAy.Txz[Pidx] = tAy.Txz[Pidx] + beta*hAy.Txz[Pidx];
								tAy.Tyz[Pidx] = tAy.Tyz[Pidx] + beta*hAy.Tyz[Pidx];
								tAy.Vx[Pidx] = tAy.Vx[Pidx] + beta*hAy.Vx[Pidx];
								tAy.Vy[Pidx] = tAy.Vy[Pidx] + beta*hAy.Vy[Pidx];
								tAy.Vz[Pidx] = tAy.Vz[Pidx] + beta*hAy.Vz[Pidx];
							}
							
							//Z dir absorption
							if(idz <= nabs[4] + nabs[5] +LenFD-1)
							{
								Pidx = (idz-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) 
									+ idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;

								Az.Txx[Pidx] = FAz.Txx[Pidx] + alpha*hAz.Txx[Pidx];
								Az.Tyy[Pidx] = FAz.Tyy[Pidx] + alpha*hAz.Tyy[Pidx];
								Az.Tzz[Pidx] = FAz.Tzz[Pidx] + alpha*hAz.Tzz[Pidx];
								Az.Txy[Pidx] = FAz.Txy[Pidx] + alpha*hAz.Txy[Pidx];
								Az.Txz[Pidx] = FAz.Txz[Pidx] + alpha*hAz.Txz[Pidx];
								Az.Tyz[Pidx] = FAz.Tyz[Pidx] + alpha*hAz.Tyz[Pidx];
								Az.Vx[Pidx] = FAz.Vx[Pidx] + alpha*hAz.Vx[Pidx];
								Az.Vy[Pidx] = FAz.Vy[Pidx] + alpha*hAz.Vy[Pidx];
								Az.Vz[Pidx] = FAz.Vz[Pidx] + alpha*hAz.Vz[Pidx];

								tAz.Txx[Pidx] = tAz.Txx[Pidx] + beta*hAz.Txx[Pidx];
								tAz.Tyy[Pidx] = tAz.Tyy[Pidx] + beta*hAz.Tyy[Pidx];
								tAz.Tzz[Pidx] = tAz.Tzz[Pidx] + beta*hAz.Tzz[Pidx];
								tAz.Txy[Pidx] = tAz.Txy[Pidx] + beta*hAz.Txy[Pidx];
								tAz.Txz[Pidx] = tAz.Txz[Pidx] + beta*hAz.Txz[Pidx];
								tAz.Tyz[Pidx] = tAz.Tyz[Pidx] + beta*hAz.Tyz[Pidx];
								tAz.Vx[Pidx] = tAz.Vx[Pidx] + beta*hAz.Vx[Pidx];
								tAz.Vy[Pidx] = tAz.Vy[Pidx] + beta*hAz.Vy[Pidx];
								tAz.Vz[Pidx] = tAz.Vz[Pidx] + beta*hAz.Vz[Pidx];
							}
#endif

							
						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void IterationFinal(Real stept, Real beta, wfield hW, wfield tW, wfield W,
		    int *nabs, wfield hAx, wfield tAx, wfield Ax,	wfield hAy, wfield tAy, wfield Ay,
			       wfield hAz, wfield tAz, wfield Az)
{
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int Pidx;

	beta *= stept;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;

							W.Txx[Gindex] = tW.Txx[Gindex] + beta*hW.Txx[Gindex];
							W.Tyy[Gindex] = tW.Tyy[Gindex] + beta*hW.Tyy[Gindex];
							W.Tzz[Gindex] = tW.Tzz[Gindex] + beta*hW.Tzz[Gindex];
							W.Txy[Gindex] = tW.Txy[Gindex] + beta*hW.Txy[Gindex];
							W.Txz[Gindex] = tW.Txz[Gindex] + beta*hW.Txz[Gindex];
							W.Tyz[Gindex] = tW.Tyz[Gindex] + beta*hW.Tyz[Gindex];
							W.Vx[Gindex] = tW.Vx[Gindex] + beta*hW.Vx[Gindex];
							W.Vy[Gindex] = tW.Vy[Gindex] + beta*hW.Vy[Gindex];
							W.Vz[Gindex] = tW.Vz[Gindex] + beta*hW.Vz[Gindex];

#ifdef DisBug
if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf(" IteFin W-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,W.Txx[Gindex],W.Tyy[Gindex],W.Tzz[Gindex],W.Txy[Gindex],W.Txz[Gindex],W.Tyz[Gindex],W.Vx[Gindex],W.Vy[Gindex],W.Vz[Gindex]);
}
#endif

#ifdef CFSPML
							//X dir absorption
							if(idx <= nabs[0] + nabs[1] + LenFD -1 )
							{
								Pidx = (idx - LenFD)*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
								
								Ax.Txx[Pidx] = tAx.Txx[Pidx] + beta*hAx.Txx[Pidx];
								Ax.Tyy[Pidx] = tAx.Tyy[Pidx] + beta*hAx.Tyy[Pidx];
								Ax.Tzz[Pidx] = tAx.Tzz[Pidx] + beta*hAx.Tzz[Pidx];
								Ax.Txy[Pidx] = tAx.Txy[Pidx] + beta*hAx.Txy[Pidx];
								Ax.Txz[Pidx] = tAx.Txz[Pidx] + beta*hAx.Txz[Pidx];
								Ax.Tyz[Pidx] = tAx.Tyz[Pidx] + beta*hAx.Tyz[Pidx];
								Ax.Vx[Pidx] = tAx.Vx[Pidx] + beta*hAx.Vx[Pidx];
								Ax.Vy[Pidx] = tAx.Vy[Pidx] + beta*hAx.Vy[Pidx];
								Ax.Vz[Pidx] = tAx.Vz[Pidx] + beta*hAx.Vz[Pidx];
							}

							//Y dir absorption
							if(idy <= nabs[2] + nabs[3] +LenFD-1)
							{
								Pidx = (idy-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

								Ay.Txx[Pidx] = tAy.Txx[Pidx] + beta*hAy.Txx[Pidx];
								Ay.Tyy[Pidx] = tAy.Tyy[Pidx] + beta*hAy.Tyy[Pidx];
								Ay.Tzz[Pidx] = tAy.Tzz[Pidx] + beta*hAy.Tzz[Pidx];
								Ay.Txy[Pidx] = tAy.Txy[Pidx] + beta*hAy.Txy[Pidx];
								Ay.Txz[Pidx] = tAy.Txz[Pidx] + beta*hAy.Txz[Pidx];
								Ay.Tyz[Pidx] = tAy.Tyz[Pidx] + beta*hAy.Tyz[Pidx];
								Ay.Vx[Pidx] = tAy.Vx[Pidx] + beta*hAy.Vx[Pidx];
								Ay.Vy[Pidx] = tAy.Vy[Pidx] + beta*hAy.Vy[Pidx];
								Ay.Vz[Pidx] = tAy.Vz[Pidx] + beta*hAy.Vz[Pidx];
							}
							
							//Z dir absorption
							if(idz <= nabs[4] + nabs[5] +LenFD-1)
							{
								Pidx = (idz-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) 
									+ idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;

								Az.Txx[Pidx] = tAz.Txx[Pidx] + beta*hAz.Txx[Pidx];
								Az.Tyy[Pidx] = tAz.Tyy[Pidx] + beta*hAz.Tyy[Pidx];
								Az.Tzz[Pidx] = tAz.Tzz[Pidx] + beta*hAz.Tzz[Pidx];
								Az.Txy[Pidx] = tAz.Txy[Pidx] + beta*hAz.Txy[Pidx];
								Az.Txz[Pidx] = tAz.Txz[Pidx] + beta*hAz.Txz[Pidx];
								Az.Tyz[Pidx] = tAz.Tyz[Pidx] + beta*hAz.Tyz[Pidx];
								Az.Vx[Pidx] = tAz.Vx[Pidx] + beta*hAz.Vx[Pidx];
								Az.Vy[Pidx] = tAz.Vy[Pidx] + beta*hAz.Vy[Pidx];
								Az.Vz[Pidx] = tAz.Vz[Pidx] + beta*hAz.Vz[Pidx];
							}
#endif


						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void IterationFinalPV(Real stept, Real beta, PeakVel Dpv, wfield hW, wfield tW, wfield W,
		    int *nabs, wfield hAx, wfield tAx, wfield Ax,	wfield hAy, wfield tAy, wfield Ay,
			       wfield hAz, wfield tAz, wfield Az)
{
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int xiaoI;
	int Pidx;

	beta *= stept;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							xiaoI = idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;

							W.Txx[Gindex] = tW.Txx[Gindex] + beta*hW.Txx[Gindex];
							W.Tyy[Gindex] = tW.Tyy[Gindex] + beta*hW.Tyy[Gindex];
							W.Tzz[Gindex] = tW.Tzz[Gindex] + beta*hW.Tzz[Gindex];
							W.Txy[Gindex] = tW.Txy[Gindex] + beta*hW.Txy[Gindex];
							W.Txz[Gindex] = tW.Txz[Gindex] + beta*hW.Txz[Gindex];
							W.Tyz[Gindex] = tW.Tyz[Gindex] + beta*hW.Tyz[Gindex];
							W.Vx[Gindex] = tW.Vx[Gindex] + beta*hW.Vx[Gindex];
							W.Vy[Gindex] = tW.Vy[Gindex] + beta*hW.Vy[Gindex];
							W.Vz[Gindex] = tW.Vz[Gindex] + beta*hW.Vz[Gindex];
							
							if(idz == ipam[8]+LenFD-1)
							{
								Dpv.Vx[xiaoI] = MAX( ABS(W.Vx[Gindex]), ABS(Dpv.Vx[xiaoI]) );
								Dpv.Vy[xiaoI] = MAX( ABS(W.Vy[Gindex]), ABS(Dpv.Vy[xiaoI]) );
								Dpv.Vz[xiaoI] = MAX( ABS(W.Vz[Gindex]), ABS(Dpv.Vz[xiaoI]) );
							}


#ifdef DisBug
if(zbx == idx+(ipam[2]-LenFD)+ipam[9] && zby == idy+(ipam[4]-LenFD) && zbz == idz)
{
	printf(" IteFin W-->(%d,%d,%d) Txx=%e, Tyy=%e, Tzz=%e\n\tTxy=%e, Txz=%e,Tzz=%e\n\tVx=%e Vy=%e Vz=%e\n",
		zbx,zby,idz,W.Txx[Gindex],W.Tyy[Gindex],W.Tzz[Gindex],W.Txy[Gindex],W.Txz[Gindex],W.Tyz[Gindex],W.Vx[Gindex],W.Vy[Gindex],W.Vz[Gindex]);
}
#endif

#ifdef CFSPML
							//X dir absorption
							if(idx <= nabs[0] + nabs[1] + LenFD -1 )
							{
								Pidx = (idx - LenFD)*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
								
								Ax.Txx[Pidx] = tAx.Txx[Pidx] + beta*hAx.Txx[Pidx];
								Ax.Tyy[Pidx] = tAx.Tyy[Pidx] + beta*hAx.Tyy[Pidx];
								Ax.Tzz[Pidx] = tAx.Tzz[Pidx] + beta*hAx.Tzz[Pidx];
								Ax.Txy[Pidx] = tAx.Txy[Pidx] + beta*hAx.Txy[Pidx];
								Ax.Txz[Pidx] = tAx.Txz[Pidx] + beta*hAx.Txz[Pidx];
								Ax.Tyz[Pidx] = tAx.Tyz[Pidx] + beta*hAx.Tyz[Pidx];
								Ax.Vx[Pidx] = tAx.Vx[Pidx] + beta*hAx.Vx[Pidx];
								Ax.Vy[Pidx] = tAx.Vy[Pidx] + beta*hAx.Vy[Pidx];
								Ax.Vz[Pidx] = tAx.Vz[Pidx] + beta*hAx.Vz[Pidx];
							}

							//Y dir absorption
							if(idy <= nabs[2] + nabs[3] +LenFD-1)
							{
								Pidx = (idy-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[8]+2*LenFD) + idx*(ipam[8]+2*LenFD) + idz;

								Ay.Txx[Pidx] = tAy.Txx[Pidx] + beta*hAy.Txx[Pidx];
								Ay.Tyy[Pidx] = tAy.Tyy[Pidx] + beta*hAy.Tyy[Pidx];
								Ay.Tzz[Pidx] = tAy.Tzz[Pidx] + beta*hAy.Tzz[Pidx];
								Ay.Txy[Pidx] = tAy.Txy[Pidx] + beta*hAy.Txy[Pidx];
								Ay.Txz[Pidx] = tAy.Txz[Pidx] + beta*hAy.Txz[Pidx];
								Ay.Tyz[Pidx] = tAy.Tyz[Pidx] + beta*hAy.Tyz[Pidx];
								Ay.Vx[Pidx] = tAy.Vx[Pidx] + beta*hAy.Vx[Pidx];
								Ay.Vy[Pidx] = tAy.Vy[Pidx] + beta*hAy.Vy[Pidx];
								Ay.Vz[Pidx] = tAy.Vz[Pidx] + beta*hAy.Vz[Pidx];
							}
							
							//Z dir absorption
							if(idz <= nabs[4] + nabs[5] +LenFD-1)
							{
								Pidx = (idz-LenFD)*(ipam[3]-ipam[2]+1+2*LenFD)*(ipam[5]-ipam[4]+1+2*LenFD) 
									+ idx*(ipam[5]-ipam[4]+1+2*LenFD) + idy;

								Az.Txx[Pidx] = tAz.Txx[Pidx] + beta*hAz.Txx[Pidx];
								Az.Tyy[Pidx] = tAz.Tyy[Pidx] + beta*hAz.Tyy[Pidx];
								Az.Tzz[Pidx] = tAz.Tzz[Pidx] + beta*hAz.Tzz[Pidx];
								Az.Txy[Pidx] = tAz.Txy[Pidx] + beta*hAz.Txy[Pidx];
								Az.Txz[Pidx] = tAz.Txz[Pidx] + beta*hAz.Txz[Pidx];
								Az.Tyz[Pidx] = tAz.Tyz[Pidx] + beta*hAz.Tyz[Pidx];
								Az.Vx[Pidx] = tAz.Vx[Pidx] + beta*hAz.Vx[Pidx];
								Az.Vy[Pidx] = tAz.Vy[Pidx] + beta*hAz.Vy[Pidx];
								Az.Vz[Pidx] = tAz.Vz[Pidx] + beta*hAz.Vz[Pidx];
							}
#endif


						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

__global__ void ErrorSta(wfield W, int *flag)
{
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index


	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;


							if( W.Txx[Gindex]!=0 && ABS( W.Txx[Gindex] ) < 1E-10 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Txx[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Txx[Gindex]);	
							}

							if( W.Tyy[Gindex]!=0 && ABS( W.Tyy[Gindex] ) < 1E-10 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Tyy[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Tyy[Gindex]);	
							}

							if( W.Tzz[Gindex]!=0 && ABS( W.Tzz[Gindex] ) < 1E-10 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Tzz[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Txx[Gindex]);	
							}

							if( W.Txy[Gindex]!=0 && ABS( W.Txy[Gindex] ) < 1E-10 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Txy[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Txy[Gindex]);	
							}

							if( W.Txz[Gindex]!=0 && ABS( W.Txz[Gindex] ) < 1E-10 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Txz[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Txz[Gindex]);	
							}

							if( W.Tyz[Gindex]!=0 && ABS( W.Tyz[Gindex] ) < 1E-10 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Tyz[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Tyz[Gindex]);	
							}

							if( W.Vx[Gindex]!=0 && ABS( W.Vx[Gindex] ) < 1E-15 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Vx[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Vx[Gindex]);	
							}

							if( W.Vy[Gindex]!=0 && ABS( W.Vy[Gindex] ) < 1E-15 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Vy[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Vy[Gindex]);	
							}

							if( W.Vz[Gindex]!=0 && ABS( W.Vz[Gindex] ) < 1E-15 ) 
							{
								atomicAdd_system(flag,1);
								printf("\t\tat PCS[%d]DEV[%d] RELpoint(%d,%d,%d) ABSpoint(%d,%d,%d),W.Vz[Gindex]=%e\n",
									ipam[1],ipam[0], idx,idy,idz, 
									idx+(ipam[2]-LenFD)+ipam[9], idy+(ipam[4]-LenFD), idz,
									W.Vz[Gindex]);	
							}

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}


__global__ void AbsExp(Real *Ex, Real *Ey, Real *Ez, int *nabs, wfield W)
{
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point indexa
	Real D=1.0;

	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							
							//X1 && X2
							//if(idx+Cstart<=AbsLoc[0*6+1] || idx+Cstart>=AbsLoc[1*6+0])
							if(idx+(ipam[2]-LenFD)+ipam[9]<=nabs[0]+LenFD-1 || idx+(ipam[2]-LenFD)+ipam[9]>=ipam[10]+LenFD-nabs[1])
							{
								D = MIN( MIN(Ex[idx],Ey[idy]), Ez[idz]);

								W.Txx[Gindex] = D*W.Txx[Gindex];
								W.Tyy[Gindex] = D*W.Tyy[Gindex];
								W.Tzz[Gindex] = D*W.Tzz[Gindex];
								W.Txy[Gindex] = D*W.Txy[Gindex];
								W.Txz[Gindex] = D*W.Txz[Gindex];
								W.Tyz[Gindex] = D*W.Tyz[Gindex];
								W.Vx[Gindex]  = D*W.Vx[Gindex];
								W.Vy[Gindex]  = D*W.Vy[Gindex];
								W.Vz[Gindex]  = D*W.Vz[Gindex];
							}
							else
							{
								//Y1 && Y2
								//if(idy<=AbsLoc[2*6+3] || idy>=AbsLoc[3*6+2] )
								if(idy+(ipam[4]-LenFD)<=nabs[2]+LenFD-1 || idy+(ipam[4]-LenFD)>=ipam[7]+LenFD-nabs[3])
								{
									D = MIN( MIN(Ex[idx],Ey[idy]), Ez[idz]);

									W.Txx[Gindex] = D*W.Txx[Gindex];
									W.Tyy[Gindex] = D*W.Tyy[Gindex];
									W.Tzz[Gindex] = D*W.Tzz[Gindex];
									W.Txy[Gindex] = D*W.Txy[Gindex];
									W.Txz[Gindex] = D*W.Txz[Gindex];
									W.Tyz[Gindex] = D*W.Tyz[Gindex];
									W.Vx[Gindex]  = D*W.Vx[Gindex];
									W.Vy[Gindex]  = D*W.Vy[Gindex];
									W.Vz[Gindex]  = D*W.Vz[Gindex];
								}
								else
								{
									//Z1 && Z2
									//if(idz<=AbsLoc[4*6+5] || idz>=AbsLoc[5*6+4] )
									if(idz<=nabs[4]+LenFD-1 || idz>=ipam[8]+LenFD-nabs[5])
									{
										D = MIN( MIN(Ex[idx],Ey[idy]), Ez[idz]);

										W.Txx[Gindex] = D*W.Txx[Gindex];
										W.Tyy[Gindex] = D*W.Tyy[Gindex];
										W.Tzz[Gindex] = D*W.Tzz[Gindex];
										W.Txy[Gindex] = D*W.Txy[Gindex];
										W.Txz[Gindex] = D*W.Txz[Gindex];
										W.Tyz[Gindex] = D*W.Tyz[Gindex];
										W.Vx[Gindex]  = D*W.Vx[Gindex];
										W.Vy[Gindex]  = D*W.Vy[Gindex];
										W.Vz[Gindex]  = D*W.Vz[Gindex];
									}//end Z1
								}//end Y1 
							}//end X1


						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}


__global__ void WavefieldPick(wfield W, wfield DPW, PointIndexBufferF Dpt, int currT, int np, int nt)
{
	int i;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int dst;
	int modX,modY;
	
	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							//idx = idx + Cstart;//wrong, it could chang the absolute access index
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							modX = idx+(ipam[2]-LenFD)+ipam[9];//modify to absolute index
							modY = idy+(ipam[4]-LenFD);
							
							for(i=0;i<np;i++)
							{
								dst = Dpt.Rsn[i]*nt+currT;
								
								if( modX == Dpt.locx[i] && modY == Dpt.locy[i] && idz == Dpt.locz[i] )
								{
									
									DPW.Vx[dst] = W.Vx[Gindex];
									DPW.Vy[dst] = W.Vy[Gindex];
									DPW.Vz[dst] = W.Vz[Gindex];
									DPW.Txx[dst] = W.Txx[Gindex];
									DPW.Tyy[dst] = W.Tyy[Gindex];
									DPW.Tzz[dst] = W.Tzz[Gindex];
									DPW.Txy[dst] = W.Txy[Gindex];
									DPW.Txz[dst] = W.Txz[Gindex];
									DPW.Tyz[dst] = W.Tyz[Gindex];

								}//restrict source location
							
							}//loop point in device
						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}
		
__global__ void SnapWavefieldPick(wfield W, wfield DSW, SnapIndexBufferF DSpt, int currT, int np, int nTime)
{
	//wave pick by kernel, Abandoned
	int i;
	int countX,countY,countZ;
	int idx,idy,idz;
	int Gindex;//valid physical point index
	int dst;
	int modX,modY;
	
	for(countX=0; countX<=ipam[3]-ipam[2]; countX+=gridDim.y)//loop in current compute range with step of Launch Par
	{
		idx = countX + blockIdx.y + LenFD;
		if(idx<=ipam[3]-ipam[2]+LenFD)//restrict to last index
		{
			for(countY=0; countY<=ipam[5]-ipam[4]; countY+=gridDim.x)
			{
				idy = countY + blockIdx.x + LenFD;
				if(idy<=ipam[5]-ipam[4]+LenFD)
				{
					for(countZ=0; countZ<ipam[8]; countZ+=blockDim.x)
					{
						idz = countZ + threadIdx.x + LenFD;
						if(idz<ipam[8]+LenFD)//vaild point with one virtual bounds
						{
							//idx = idx + Cstart;//wrong, it could chang the absolute access index
							Gindex = idx*(ipam[5]-ipam[4]+1+2*LenFD)*(ipam[8]+2*LenFD) + idy*(ipam[8]+2*LenFD) + idz;
							modX = idx+(ipam[2]-LenFD)+ipam[9];//modify to absolute index
							modY = idy+(ipam[4]-LenFD);
							
							for(i=0;i<np;i++)
							{
								dst = DSpt.Rsn[i]*nTime+currT;
								
								if( modX == DSpt.locx[i] && modY == DSpt.locy[i] && idz == DSpt.locz[i] )
								{
									//printf("at PCS=%d DEV=%d,Rsn=%d,Gsn=%d,(%d,%d,%d)\n",ipam[1],ipam[0],
									//	DSpt.Rsn[i],DSpt.Gsn[i],DSpt.locx[i],DSpt.locy[i],DSpt.locz[i]);
									if(DSpt.cmp==1 || DSpt.cmp==3)
									{
										DSW.Vx[dst] = W.Vx[Gindex];
										DSW.Vy[dst] = W.Vy[Gindex];
										DSW.Vz[dst] = W.Vz[Gindex];
									}
									if(DSpt.cmp==2 || DSpt.cmp==3)
									{
										DSW.Txx[dst] = W.Txx[Gindex];
										DSW.Tyy[dst] = W.Tyy[Gindex];
										DSW.Tzz[dst] = W.Tzz[Gindex];
										DSW.Txy[dst] = W.Txy[Gindex];
										DSW.Txz[dst] = W.Txz[Gindex];
										DSW.Tyz[dst] = W.Tyz[Gindex];
									}
									
								}//restrict source location
								
							}//loop point in device

						}//restrict idz;
					}//loop countZ
				}//restrict idy
			}//loop countY
		}//restrict idx
	}//loop countX

}

